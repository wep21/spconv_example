#include "hip/hip_runtime.h"
#include <spconvlib/cumm/conv/main/Ampere_s8s8s8s32f16tnt_m64n32k32m32n32k32A1T16816_200_C301LLL_SK_S8/ConvKernel.h>
namespace spconvlib {
namespace cumm {
namespace conv {
namespace main {
namespace Ampere_s8s8s8s32f16tnt_m64n32k32m32n32k32A1T16816_200_C301LLL_SK_S8 {
using TensorViewNVRTCKernel = spconvlib::cumm::common::TensorViewNVRTCKernel;
using RowMajor = spconvlib::cumm::gemm::layout::RowMajor;
using ColumnMajor = spconvlib::cumm::gemm::layout::ColumnMajor;
using GemmBasicKernel = spconvlib::cumm::common::GemmBasicKernel;
using GemmBasic = spconvlib::cumm::common::GemmBasic;
using GemmKernelFlags = spconvlib::cumm::common::GemmKernelFlags;
using InputIteratorA = spconvlib::cumm::conv::main::Ampere_s8s8s8s32f16tnt_m64n32k32m32n32k32A1T16816_200_C301LLL_SK_S8::inpitera::ForwardDgradSparseIOIterator;
using InputIteratorB = spconvlib::cumm::conv::main::Ampere_s8s8s8s32f16tnt_m64n32k32m32n32k32A1T16816_200_C301LLL_SK_S8::inpiterb::WeightIteratorDP4A;
using LayoutA = spconvlib::cumm::conv::main::cpAmpere_s8s8s8s32f16tnt_m64n32k32m32n32k32A1T16816_200_C301LLL_SK_S8::la::TensorGeneric;
using LayoutB = spconvlib::cumm::conv::main::cpAmpere_s8s8s8s32f16tnt_m64n32k32m32n32k32A1T16816_200_C301LLL_SK_S8::lb::TensorGeneric;
using LayoutC = spconvlib::cumm::conv::main::cpAmpere_s8s8s8s32f16tnt_m64n32k32m32n32k32A1T16816_200_C301LLL_SK_S8::lc::TensorGeneric;
using BlockMmaStorage = spconvlib::cumm::conv::main::Ampere_s8s8s8s32f16tnt_m64n32k32m32n32k32A1T16816_200_C301LLL_SK_S8::gemm_smem_storage::BlockMmaStorage;
using OutputSmemStorage = spconvlib::cumm::conv::main::Ampere_s8s8s8s32f16tnt_m64n32k32m32n32k32A1T16816_200_C301LLL_SK_S8::out_smem_storage::OutputSmemStorage;
using ConvParams = spconvlib::cumm::conv::main::cpAmpere_s8s8s8s32f16tnt_m64n32k32m32n32k32A1T16816_200_C301LLL_SK_S8::ConvParams;
using ConvProblem = spconvlib::cumm::conv::main::cpAmpere_s8s8s8s32f16tnt_m64n32k32m32n32k32A1T16816_200_C301LLL_SK_S8::cp::ConvProblem;
using OutIter = spconvlib::cumm::conv::main::Ampere_s8s8s8s32f16tnt_m64n32k32m32n32k32A1T16816_200_C301LLL_SK_S8::out_iter::OutIterator;
using ConstOutIter = spconvlib::cumm::conv::main::Ampere_s8s8s8s32f16tnt_m64n32k32m32n32k32A1T16816_200_C301LLL_SK_S8::out_iter_const::OutIterator;
using ConstScaleOutIter = spconvlib::cumm::conv::main::Ampere_s8s8s8s32f16tnt_m64n32k32m32n32k32A1T16816_200_C301LLL_SK_S8::scale_out_iter_const::OutIterator;
using OutputOp = spconvlib::cumm::conv::main::Ampere_s8s8s8s32f16tnt_m64n32k32m32n32k32A1T16816_200_C301LLL_SK_S8::out_op::Int8Inference;
using Mma = spconvlib::cumm::conv::main::Ampere_s8s8s8s32f16tnt_m64n32k32m32n32k32A1T16816_200_C301LLL_SK_S8::mma::MmaMultiStage;
using MaskIGemmIteratorDynamic = spconvlib::cumm::conv::main::Ampere_s8s8s8s32f16tnt_m64n32k32m32n32k32A1T16816_200_C301LLL_SK_S8::mma_miterd::MaskIGemmIteratorMaskLoaderDynamic;
using Output = spconvlib::cumm::conv::main::Ampere_s8s8s8s32f16tnt_m64n32k32m32n32k32A1T16816_200_C301LLL_SK_S8::output::Output;
__global__ void conv_kernel(ConvParams params)   {
  
  #if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 800))
    constexpr bool kSplitKSerial = false;
    extern __shared__ uint8_t SharedStorage[];
    auto gemm_shared_mem =
        reinterpret_cast<BlockMmaStorage *>(SharedStorage);
    auto out_shared_mem =
        reinterpret_cast<OutputSmemStorage *>(SharedStorage);
    int tile_offset_m = blockIdx.x;
    int tile_offset_n = blockIdx.y;
    int tile_offset_k = blockIdx.z;
    if (tile_offset_m >= params.grid_dims.x ||
        tile_offset_n >= params.grid_dims.y) {
        return;
    }
    tv::array<int, 2> block_offset_A{tile_offset_m * 64, tile_offset_k * 32};
    tv::array<int, 2> block_offset_B{tile_offset_n * 32, tile_offset_k * 32};
    int thread_idx = threadIdx.x;
    InputIteratorA input_iter_A(
        params.itera_params_, params.problem, params.ptr_A,
        thread_idx,
        block_offset_A);
    InputIteratorB input_iter_B(
        params.iterb_params_, params.problem, params.ptr_B,
        thread_idx,
        block_offset_B);
    int warp_idx = __shfl_sync(0xffffffff, threadIdx.x / 32, 0);
    int lane_idx = threadIdx.x % 32;
    int warp_mn =
        warp_idx % (2 * 1);
    int warp_idx_k =
        warp_idx / (2 * 1);
    int warp_m = warp_mn % 2;
    int warp_n = warp_mn / 2;
    uint32_t kmask = 0;
    tv::array<uint32_t, 2> masks;
    masks.clear();
    TV_PRAGMA_UNROLL
    for (int i = 0; i < 2; ++i){
        if (tile_offset_m * 64 + i * 32 + lane_idx < params.m){
            masks[i] = params.mask_ptr[tile_offset_m * 64 + i * 32 + lane_idx];
        }
    }
    TV_PRAGMA_UNROLL
    for (int i = 0; i < 2; ++i){
        kmask |= masks[i];
    }
    // perform a warp reduce to get block mask
    TV_PRAGMA_UNROLL
    for (int mask = 16; mask > 0; mask /= 2) {
        kmask |= __shfl_xor_sync(0xffffffff, kmask, mask, 32);
    }
    kmask &= params.mask_filter;
    if (params.mask_out_ptr != nullptr){
        params.mask_out_ptr[tile_offset_m] = kmask;
    }
    Mma mma(gemm_shared_mem, thread_idx, warp_idx_k, warp_m, warp_n, lane_idx);
    tv::array<int32_t, 32, 0> accumulators;
    accumulators.clear();
    if (!kSplitKSerial || params.gemm_k_iterations > 0){
      if (kmask != 0){
          mma(params.gemm_k_iterations, accumulators, input_iter_A, input_iter_B, accumulators, kmask, params.problem.kernel_volume);
      }
    }
    // // C = alpha * A@B + beta * D, D can be C
    OutputOp output_op(params.alpha, params.beta, params.act_alpha, params.act_beta, params.act_type);
    tv::array<int, 2> block_offset_C{tile_offset_m * 64,
                                    tile_offset_n * 32};
    tv::array<int, 2> block_extent_C{params.m, params.n};
    OutIter out_iter_C(params.out_params_, params.ptr_C, params.ptr_D, block_extent_C,
                            block_offset_C,
                            thread_idx);
    ConstScaleOutIter out_iter_bias(params.out_params_scalebias_, params.bias_pointer, block_extent_C,
                            block_offset_C,
                            thread_idx);
    ConstScaleOutIter out_iter_scale(params.out_params_scalebias_, params.scale_pointer, block_extent_C,
                            block_offset_C,
                            thread_idx);
    Output out(out_shared_mem, thread_idx, warp_idx_k, warp_m, warp_n, lane_idx);
    out.run(output_op, accumulators, out_iter_C, out_iter_bias, out_iter_scale);
  #else
    tv::printf2_once("this arch isn't supported!");
    assert(0);
  #endif
}
} // namespace Ampere_s8s8s8s32f16tnt_m64n32k32m32n32k32A1T16816_200_C301LLL_SK_S8
} // namespace main
} // namespace conv
} // namespace cumm
} // namespace spconvlib