#include "hip/hip_runtime.h"
#include <spconvlib/cumm/conv/main/ConvMainUnitTest.h>
#include <spconvlib/cumm/common/TensorViewKernel.h>
#include <spconvlib/cumm/conv/main/cpAmpere_f32f32f32f32f32tnt_m32n32k16m16n16k16A0T1688_200_C301LLL_SK/ConvParams.h>
#include <spconvlib/cumm/conv/main/Ampere_f32f32f32f32f32tnt_m32n32k16m16n16k16A0T1688_200_C301LLL_SK/ConvKernel.h>
#include <spconvlib/cumm/conv/main/cpAmpere_f32f32f32f32f32tnt_m32n32k16m16n16k16A1T1688_200_C301LLL_SK/ConvParams.h>
#include <spconvlib/cumm/conv/main/Ampere_f32f32f32f32f32tnt_m32n32k16m16n16k16A1T1688_200_C301LLL_SK/ConvKernel.h>
#include <spconvlib/cumm/conv/main/cpAmpere_f32f32f32f32f32tnt_m32n32k16m16n16k16A1T1688_300_C301LLL_SK/ConvParams.h>
#include <spconvlib/cumm/conv/main/Ampere_f32f32f32f32f32tnt_m32n32k16m16n16k16A1T1688_300_C301LLL_SK/ConvKernel.h>
#include <spconvlib/cumm/conv/main/cpAmpere_f32f32f32f32f32tnt_m32n32k16m16n16k16A1T1688_400_C301LLL_SK/ConvParams.h>
#include <spconvlib/cumm/conv/main/Ampere_f32f32f32f32f32tnt_m32n32k16m16n16k16A1T1688_400_C301LLL_SK/ConvKernel.h>
#include <spconvlib/cumm/conv/main/cpAmpere_f32f32f32f32f32tnt_m64n64k32m32n32k32A1T1688_200_C301LLL_SK/ConvParams.h>
#include <spconvlib/cumm/conv/main/Ampere_f32f32f32f32f32tnt_m64n64k32m32n32k32A1T1688_200_C301LLL_SK/ConvKernel.h>
#include <spconvlib/cumm/conv/main/cpAmpere_f32f32f32f32f32tnt_m64n64k32m32n32k32A1T1688_300_C301LLL_SK/ConvParams.h>
#include <spconvlib/cumm/conv/main/Ampere_f32f32f32f32f32tnt_m64n64k32m32n32k32A1T1688_300_C301LLL_SK/ConvKernel.h>
#include <spconvlib/cumm/conv/main/cpAmpere_f32f32f32f32f32tnt_m64n64k32m32n32k32A1T1688_400_C301LLL_SK/ConvParams.h>
#include <spconvlib/cumm/conv/main/Ampere_f32f32f32f32f32tnt_m64n64k32m32n32k32A1T1688_400_C301LLL_SK/ConvKernel.h>
namespace spconvlib {
namespace cumm {
namespace conv {
namespace main {
using TensorView = spconvlib::cumm::common::TensorView;
using GemmBasic = spconvlib::cumm::common::GemmBasic;
using GemmBasicHost = spconvlib::cumm::common::GemmBasicHost;
using ConvNVRTCParams = spconvlib::cumm::conv::kernel::ConvNVRTCParams;
using CummNVRTCLib = spconvlib::cumm::common::CummNVRTCLib;
using TensorViewKernel = spconvlib::cumm::common::TensorViewKernel;
using ConvParamsAmpere_f32f32f32f32f32tnt_m32n32k16m16n16k16A0T1688_200_C301LLL_SK = spconvlib::cumm::conv::main::cpAmpere_f32f32f32f32f32tnt_m32n32k16m16n16k16A0T1688_200_C301LLL_SK::ConvParams;
using ConvAmpere_f32f32f32f32f32tnt_m32n32k16m16n16k16A0T1688_200_C301LLL_SK = spconvlib::cumm::conv::main::Ampere_f32f32f32f32f32tnt_m32n32k16m16n16k16A0T1688_200_C301LLL_SK::ConvKernel;
using ConvParamsAmpere_f32f32f32f32f32tnt_m32n32k16m16n16k16A1T1688_200_C301LLL_SK = spconvlib::cumm::conv::main::cpAmpere_f32f32f32f32f32tnt_m32n32k16m16n16k16A1T1688_200_C301LLL_SK::ConvParams;
using ConvAmpere_f32f32f32f32f32tnt_m32n32k16m16n16k16A1T1688_200_C301LLL_SK = spconvlib::cumm::conv::main::Ampere_f32f32f32f32f32tnt_m32n32k16m16n16k16A1T1688_200_C301LLL_SK::ConvKernel;
using ConvParamsAmpere_f32f32f32f32f32tnt_m32n32k16m16n16k16A1T1688_300_C301LLL_SK = spconvlib::cumm::conv::main::cpAmpere_f32f32f32f32f32tnt_m32n32k16m16n16k16A1T1688_300_C301LLL_SK::ConvParams;
using ConvAmpere_f32f32f32f32f32tnt_m32n32k16m16n16k16A1T1688_300_C301LLL_SK = spconvlib::cumm::conv::main::Ampere_f32f32f32f32f32tnt_m32n32k16m16n16k16A1T1688_300_C301LLL_SK::ConvKernel;
using ConvParamsAmpere_f32f32f32f32f32tnt_m32n32k16m16n16k16A1T1688_400_C301LLL_SK = spconvlib::cumm::conv::main::cpAmpere_f32f32f32f32f32tnt_m32n32k16m16n16k16A1T1688_400_C301LLL_SK::ConvParams;
using ConvAmpere_f32f32f32f32f32tnt_m32n32k16m16n16k16A1T1688_400_C301LLL_SK = spconvlib::cumm::conv::main::Ampere_f32f32f32f32f32tnt_m32n32k16m16n16k16A1T1688_400_C301LLL_SK::ConvKernel;
using ConvParamsAmpere_f32f32f32f32f32tnt_m64n64k32m32n32k32A1T1688_200_C301LLL_SK = spconvlib::cumm::conv::main::cpAmpere_f32f32f32f32f32tnt_m64n64k32m32n32k32A1T1688_200_C301LLL_SK::ConvParams;
using ConvAmpere_f32f32f32f32f32tnt_m64n64k32m32n32k32A1T1688_200_C301LLL_SK = spconvlib::cumm::conv::main::Ampere_f32f32f32f32f32tnt_m64n64k32m32n32k32A1T1688_200_C301LLL_SK::ConvKernel;
using ConvParamsAmpere_f32f32f32f32f32tnt_m64n64k32m32n32k32A1T1688_300_C301LLL_SK = spconvlib::cumm::conv::main::cpAmpere_f32f32f32f32f32tnt_m64n64k32m32n32k32A1T1688_300_C301LLL_SK::ConvParams;
using ConvAmpere_f32f32f32f32f32tnt_m64n64k32m32n32k32A1T1688_300_C301LLL_SK = spconvlib::cumm::conv::main::Ampere_f32f32f32f32f32tnt_m64n64k32m32n32k32A1T1688_300_C301LLL_SK::ConvKernel;
using ConvParamsAmpere_f32f32f32f32f32tnt_m64n64k32m32n32k32A1T1688_400_C301LLL_SK = spconvlib::cumm::conv::main::cpAmpere_f32f32f32f32f32tnt_m64n64k32m32n32k32A1T1688_400_C301LLL_SK::ConvParams;
using ConvAmpere_f32f32f32f32f32tnt_m64n64k32m32n32k32A1T1688_400_C301LLL_SK = spconvlib::cumm::conv::main::Ampere_f32f32f32f32f32tnt_m64n64k32m32n32k32A1T1688_400_C301LLL_SK::ConvKernel;
void ConvMainUnitTest::matmul_split_Ampere_f32f32f32_0(tv::gemm::ConvParams params)   {
  
  // auto rtxtimer = tv::CPUTimer<>();
  // auto ev1 = tv::CUDAEvent("wtf").record();
  static_assert(3 == CUMM_MAXIMUM_NVRTC_CONV_NDIM, "error");
  int groups = 1;
  bool found = false;
  auto& algo_desp = params.conv_algo_desp;
  auto dacc = tv::DType(algo_desp.dacc);
  auto dcomp = tv::DType(algo_desp.dcomp);
  tv::gemm::ConvOpType op_type = static_cast<tv::gemm::ConvOpType>(algo_desp.op_type);
  int split_k_slices = params.split_k_slices;
  auto& workspace = params.workspace;
  auto& input = params.input;
  auto& weight = params.weight;
  auto& output = params.output;
  auto& output_add = params.output_add;
  auto& bias = params.bias;
  auto& scale = params.scale;
  auto& indices = params.indices;
  auto& mask = params.mask;
  auto& mask_argsort = params.mask_argsort;
  auto& mask_output = params.mask_output;
  auto& padding = params.padding;
  auto& stride = params.stride;
  auto& dilation = params.dilation;
  auto& mask_width = params.mask_width;
  auto& evtimer = params.timer;
  int io_dim = algo_desp.mask_sparse ? 2 : algo_desp.ndim + 2;
  int weight_ndim = algo_desp.mask_sparse ? 3 : algo_desp.ndim + 2;
  int dim_start =  algo_desp.layout_w == tv::gemm::ConvLayoutType::kChannelFirst ? 2 : 1;
  int ndim = algo_desp.ndim;
  TV_ASSERT_RT_ERR(input.ndim() == io_dim, "error");
  TV_ASSERT_RT_ERR(weight.ndim() == weight_ndim, "error");
  TV_ASSERT_RT_ERR(output.ndim() == io_dim, "error");
  if (!(algo_desp.split_k_serial() || algo_desp.split_k_parallel()) && split_k_slices > 1){
      TV_ASSERT_RT_ERR("algo don't support splitk but you provide split_k_slices > 1.", split_k_slices);
  }
  int kernel_volume = 1;
  int N = input.dim(0);
  int K = weight.dim(0);
  int C = algo_desp.layout_i == tv::gemm::ConvLayoutType::kChannelFirst ? input.dim(1) : input.dim(io_dim - 1);
  int K2 = algo_desp.layout_o == tv::gemm::ConvLayoutType::kChannelFirst ? output.dim(1) : output.dim(io_dim - 1);
  TV_ASSERT_RT_ERR(K2 == K, "error");
  tv::array<int, 3> mnk;
  auto inv_indices = tv::gemm::gemm_abc_012_to_iwo(tv::gemm::ConvOpType(algo_desp.op_type));
  std::array<tv::Tensor, 3> conv_inputs{input, weight, output};
  auto& a_ten = conv_inputs[inv_indices[0]];
  auto& b_ten = conv_inputs[inv_indices[1]];
  auto& c_ten = conv_inputs[inv_indices[2]];
  auto& nvrtc_params = params.nvrtc_params;
  tv::gemm::ConvNVRTCParams kernel_params;
  tv::gemm::SparseConvNVRTCParams sp_kernel_params;
  kernel_params.ptr_A = a_ten.const_raw_data();
  kernel_params.ptr_B = b_ten.const_raw_data();
  kernel_params.ptr_C = c_ten.raw_data();
  if (!algo_desp.is_int8_inference){
      TV_ASSERT_INVALID_ARG(output_add.empty(), "only int8 inference support output_add not empty ")
  }else{
      TV_ASSERT_RT_ERR(!bias.empty() && !scale.empty(), "int8 inference must have both scale and bias");
  }
  if (output_add.empty()){
      kernel_params.ptr_D = algo_desp.is_int8_inference ? c_ten.const_raw_data() : (bias.empty() ? c_ten.const_raw_data() : bias.const_raw_data());
  }else{
      TV_ASSERT_RT_ERR(output_add.dtype() == output.dtype() && output_add.shape() == output.shape(),
          "output and output_add must have same dtype and shape", output_add.dtype(), output.dtype(),
          output_add.shape(), output.shape());
      kernel_params.ptr_D = output_add.const_raw_data();
  }
  kernel_params.bias_pointer = bias.empty() ? nullptr : bias.const_raw_data();
  kernel_params.scale_pointer = scale.empty() ? nullptr : scale.const_raw_data();
  kernel_params.alpha = params.alpha;
  kernel_params.beta = params.beta;
  kernel_params.ndim = ndim;
  kernel_params.d_is_bias = !bias.empty();
  kernel_params.act_alpha = params.act_alpha;
  kernel_params.act_beta = params.act_beta;
  kernel_params.act_type = static_cast<int>(params.act_type);
  sp_kernel_params.ptr_A = kernel_params.ptr_A;
  sp_kernel_params.ptr_B = kernel_params.ptr_B;
  sp_kernel_params.ptr_C = kernel_params.ptr_C;
  sp_kernel_params.ptr_D = kernel_params.ptr_D;
  sp_kernel_params.scale_pointer = kernel_params.scale_pointer;
  sp_kernel_params.bias_pointer = kernel_params.bias_pointer;
  sp_kernel_params.alpha = kernel_params.alpha;
  sp_kernel_params.beta = kernel_params.beta;
  sp_kernel_params.ndim = kernel_params.ndim;
  sp_kernel_params.d_is_bias = !bias.empty();
  sp_kernel_params.act_alpha = kernel_params.act_alpha;
  sp_kernel_params.act_beta = kernel_params.act_beta;
  sp_kernel_params.act_type = kernel_params.act_type;
  constexpr int int_max = std::numeric_limits<int32_t>::max();
  if (algo_desp.mask_sparse){
      if (algo_desp.op_type == tv::gemm::ConvOpType::kBackwardWeight){
          TV_ASSERT_RT_ERR(mask_width > 0 && mask_width % algo_desp.tile_shape[2] == 0, "error");
      }
      TV_ASSERT_RT_ERR(!indices.empty(), "error");
      TV_ASSERT_RT_ERR(!mask.empty(), "error");
      TV_ASSERT_RT_ERR(!mask_argsort.empty(), "error");
      kernel_volume = weight.dim(dim_start);
      tv::check_shape(indices, {kernel_volume, -1});
      N = indices.dim(1);
      if (algo_desp.op_type == tv::gemm::ConvOpType::kBackwardWeight){
          TV_ASSERT_RT_ERR(N == output.dim(0), "error");
          TV_ASSERT_RT_ERR(int64_t(N) * int64_t(C) * tv::bit_size(algo_desp.dtype_b) / 8 < int_max, 
              "your data exceed int32 range. this will be fixed in cumm + nvrtc (spconv 2.2/2.3).");
          TV_ASSERT_RT_ERR(int64_t(N) * int64_t(K) * tv::bit_size(algo_desp.dtype_a) / 8 < int_max, 
              "your data exceed int32 range. this will be fixed in cumm + nvrtc (spconv 2.2/2.3).");
      }else if (algo_desp.op_type == tv::gemm::ConvOpType::kForward){
          TV_ASSERT_RT_ERR(N == output.dim(0), "error");
          TV_ASSERT_RT_ERR(int64_t(N) * int64_t(C) * tv::bit_size(algo_desp.dtype_a) / 8 < int_max, 
              "your data exceed int32 range. this will be fixed in cumm + nvrtc (spconv 2.2/2.3).");
      }else{
              TV_ASSERT_RT_ERR(int64_t(N) * int64_t(K) * tv::bit_size(algo_desp.dtype_a) / 8 < int_max, 
                  "your data exceed int32 range. this will be fixed in cumm + nvrtc (spconv 2.2/2.3).");
              TV_ASSERT_RT_ERR(N == input.dim(0), "error");
      }
      mnk = tv::gemm::implicit_gemm_mnk(tv::gemm::ConvOpType(algo_desp.op_type), N, C, K, kernel_volume, -1, -1, true);
  }else{
      TV_ASSERT_RT_ERR(algo_desp.ndim <= 3, "ndim too large for nvrtc");
      tv::array<int, 3> ksize, padding_arr, stride_arr, dilation_arr, input_dims, output_dims;
      TV_ASSERT_RT_ERR(ndim == padding.size() && ndim == stride.size() && ndim == dilation.size(), "error");
      for (int i = dim_start; i < dim_start + ndim; ++i){
          ksize[i - dim_start] = weight.dim(i);
          input_dims[i - dim_start] = input.dim(i);
          output_dims[i - dim_start] = output.dim(i);
      }
      for (int i = 0; i < ndim; ++i){
          padding_arr[i] = padding[i];
          stride_arr[i] = stride[i];
          dilation_arr[i] = dilation[i];
      }
      kernel_volume = 1;
      int in_prod = 1;
      int out_prod = 1;
      for (int i = 0; i < ndim; ++i){
          kernel_volume *= ksize[i];
          in_prod *= input_dims[i];
          out_prod *= output_dims[i];
      }
      mnk = tv::gemm::implicit_gemm_mnk(tv::gemm::ConvOpType(algo_desp.op_type), N, C, K, kernel_volume, in_prod, out_prod, false);
      kernel_params.input_dims = input_dims;
      kernel_params.output_dims = output_dims;
      kernel_params.ksize = ksize;
      kernel_params.padding = padding_arr;
      kernel_params.stride = stride_arr;
      kernel_params.dilation = dilation_arr;
  }
  TV_ASSERT_RT_ERR(algo_desp.supported(mnk[0], mnk[1], mnk[2], C, K, mask_width), "error");
  int workspace_size = algo_desp.query_conv_workspace_size(mnk[0], mnk[1], mnk[2], split_k_slices, kernel_volume);
  auto ctx = tv::Context();
  ctx.set_cuda_stream(reinterpret_cast<hipStream_t>(params.stream));
  if (workspace_size > 0){
      if (!workspace.empty()){
          workspace.zero_(ctx);
          TV_ASSERT_RT_ERR(workspace.nbytes() >= workspace_size, 
              "workspace at least", workspace_size, "bytes.");
      }else{
          workspace = tv::empty({workspace_size}, tv::uint8, 0);
          workspace.zero_(ctx);
      }
  }
  void* workspace_ptr = nullptr;
  if (!workspace.empty()){
      workspace_ptr = workspace.raw_data();
  }
  if (nvrtc_params.cumodule){
      TV_ASSERT_RT_ERR(!nvrtc_params.kernel_name.empty(), "you must provide name of your kernel");
      kernel_params.N = N;
      kernel_params.C = C;
      kernel_params.K = K;
      kernel_params.kernel_volume = kernel_volume;
      kernel_params.mode = static_cast<int>(tv::gemm::ConvMode::kCrossCorrelation);
      kernel_params.split_k_slices = split_k_slices;
      kernel_params.groups = groups;
      kernel_params.workspace = workspace_ptr;
      sp_kernel_params.N = kernel_params.N;
      sp_kernel_params.C = kernel_params.C;
      sp_kernel_params.K = kernel_params.K;
      sp_kernel_params.kernel_volume = kernel_params.kernel_volume;
      sp_kernel_params.mode = kernel_params.mode;
      sp_kernel_params.split_k_slices = kernel_params.split_k_slices;
      sp_kernel_params.groups = kernel_params.groups;
      sp_kernel_params.workspace = kernel_params.workspace;
      tv::array<int, 3> grid_dims_arr;
      if (algo_desp.mask_sparse){
          sp_kernel_params.mask_out_ptr = mask_output.empty() ? nullptr : mask_output.data_ptr<uint32_t>();
          sp_kernel_params.mask_width = mask_width;
          sp_kernel_params.mask_ptr = mask.data_ptr<const uint32_t>();
          sp_kernel_params.reverse_mask = params.reverse_mask;
          sp_kernel_params.mask_filter = params.mask_filter;
          sp_kernel_params.indice_ptr = indices.data_ptr<const int32_t>();
          sp_kernel_params.mask_argsort_ptr = mask_argsort.data_ptr<const int32_t>();
          grid_dims_arr = tv::gemm::get_spconv_logical_tile_count(mnk[0], mnk[1], mnk[2], 
                          algo_desp.tile_shape[0], algo_desp.tile_shape[1], split_k_slices, kernel_volume, algo_desp.op_type);
      }else{
          grid_dims_arr = tv::gemm::get_logical_tile_count(mnk[0], mnk[1], mnk[2], 
                          algo_desp.tile_shape[0], algo_desp.tile_shape[1], split_k_slices);
      }
      dim3 grid_dims;
      grid_dims.x = grid_dims_arr[0];
      grid_dims.y = grid_dims_arr[1];
      grid_dims.z = grid_dims_arr[2];
      if (algo_desp.op_type == tv::gemm::ConvOpType::kBackwardWeight && algo_desp.mask_sparse){
          int num_reduced_mask = tv::div_up(sp_kernel_params.N, sp_kernel_params.mask_width);
          TV_ASSERT_RT_ERR(mask.dim(0) >= num_reduced_mask, "error");
      }
      std::string algo_name;
      if (evtimer.enable()){
          algo_name = algo_desp.__repr__();
      }
      auto kernel = nvrtc_params.cumodule->kernel(nvrtc_params.kernel_name);
      auto& driver = nvrtc_params.cumodule->get_driver_wrapper();
      hipError_t result;
      if (nvrtc_params.smem_size > 0){
          if (nvrtc_params.smem_size >= (48 << 10)) {
              TV_CUDA_RESULT_CHECK(driver.cuDrvFuncSetAttribute(kernel,
                                              HIP_FUNC_ATTRIBUTE_MAX_DYNAMIC_SHARED_SIZE_BYTES,
                                              nvrtc_params.smem_size));
              TV_CUDA_RESULT_CHECK(driver.cuDrvFuncSetAttribute(
                  kernel,
                  HIP_FUNC_ATTRIBUTE_PREFERRED_SHARED_MEMORY_CARVEOUT, 100));
          }
      }
      hipStream_t stream = reinterpret_cast<hipStream_t>(params.stream);
      void* kernel_params_ptr;
      if (algo_desp.mask_sparse){
          kernel_params_ptr = &sp_kernel_params;
      }else{
          kernel_params_ptr = &kernel_params;
      }
      // auto ev2 = tv::CUDAEvent("wtf").record();
      // ev1.sync();
      // ev2.sync();
      // tv::ssprint("prep time", tv::CUDAEvent::duration(ev1, ev2));
      if (nvrtc_params.mode == 2){
          tv::CUDAKernelTimerGuard timerguard(algo_name, evtimer, stream);
              std::vector<void*> args{kernel_params_ptr, &grid_dims, &params.stream};
          TV_CUDA_RESULT_CHECK(nvrtc_params.cumodule->cuDrvLaunchKernel(kernel, 1, 1, 1, 
              1, 1, 1, 0, stream, args.data(), 0));
      }else if (nvrtc_params.mode == 3){
          // use kernel-cpu-kernel
          auto init_kernel = nvrtc_params.cumodule->kernel(nvrtc_params.init_kernel_name);
          tv::Tensor temp_data = nvrtc_params.param_storage;
          if (nvrtc_params.param_storage.empty()){
              temp_data = tv::empty({nvrtc_params.param_size}, tv::uint8, 0);
          }else{
              TV_ASSERT_RT_ERR(temp_data.nbytes() >= nvrtc_params.param_size, "your params storage too small");
          }
          void* raw_data_ptr;
          void* temp_data_ptr = temp_data.raw_data();
          tv::Tensor temp_data_cpu = nvrtc_params.param_storage_cpu;
          {
              tv::CUDAKernelTimerGuard timerguard(algo_name + "/init", evtimer, stream);
              std::vector<void*> args{kernel_params_ptr, &temp_data_ptr};
              TV_CUDA_RESULT_CHECK(driver.cuDrvLaunchKernel(init_kernel, 1, 1, 1, 32, 1, 1, 0, stream, args.data(), 0));
              if (nvrtc_params.param_storage_cpu.empty()){
                  temp_data_cpu = temp_data.cpu(ctx);
              }else{
                  temp_data_cpu.copy_(temp_data, ctx);
              }
              // we must sync here because following kernel launch requires cpu data.
              checkCudaErrors(hipStreamSynchronize(stream));
              raw_data_ptr = temp_data_cpu.raw_data();
          }
          {
              tv::CUDAKernelTimerGuard timerguard(algo_name, evtimer, stream);
              std::vector<void*> args{raw_data_ptr};
              TV_CUDA_RESULT_CHECK(driver.cuDrvLaunchKernel(kernel, grid_dims.x, grid_dims.y, grid_dims.z, 
                  nvrtc_params.num_threads, 1, 1, nvrtc_params.smem_size, stream, args.data(), 0));
          }
      }else if (nvrtc_params.mode == 1){
          tv::CUDAKernelTimerGuard timerguard(algo_name, evtimer, stream);
          std::vector<void*> args{kernel_params_ptr};
          TV_CUDA_RESULT_CHECK(driver.cuDrvLaunchKernel(kernel, grid_dims.x, grid_dims.y, grid_dims.z, 
              nvrtc_params.num_threads, 1, 1, nvrtc_params.smem_size, stream, args.data(), 0));
      }else if (nvrtc_params.mode == 4){
          auto init_kernel = nvrtc_params.cumodule->kernel(nvrtc_params.init_kernel_name);
          tv::Tensor temp_data = nvrtc_params.param_storage;
          if (nvrtc_params.param_storage.empty()){
              temp_data = tv::empty({nvrtc_params.param_size}, tv::uint8, 0);
          }else{
              TV_ASSERT_RT_ERR(temp_data.nbytes() >= nvrtc_params.param_size, "your params storage too small");
          }
          void* temp_data_ptr = temp_data.raw_data();
          {
              tv::CUDAKernelTimerGuard timerguard(algo_name + "/init", evtimer, stream);
              std::vector<void*> args{kernel_params_ptr, &temp_data_ptr};
              TV_CUDA_RESULT_CHECK(nvrtc_params.cumodule->cuDrvLaunchKernel(init_kernel, 1, 1, 1, 32, 1, 1, 0, stream, args.data(), 0));
          }
          {
              tv::CUDAKernelTimerGuard timerguard(algo_name, evtimer, stream);
              auto ptr = nvrtc_params.cumodule->get_global_ptr(nvrtc_params.constant_name);
              auto constant_ten = tv::from_blob(ptr, {nvrtc_params.param_size}, tv::uint8, 0);
              constant_ten.copy_(temp_data, ctx);
              std::vector<void*> args{};
              TV_CUDA_RESULT_CHECK(nvrtc_params.cumodule->cuDrvLaunchKernel(kernel, grid_dims.x, grid_dims.y, grid_dims.z, 
                  nvrtc_params.num_threads, 1, 1, nvrtc_params.smem_size, stream, nullptr, 0));
          }
      }else{
          TV_THROW_RT_ERR("not implemented");
      }
      TV_CHECK_CUDA_ERR_V2(algo_desp.__repr__(), "error with params", input.shape(), output.shape(), weight.shape());
      return;
  }
  if (algo_desp.trans_a() == false && algo_desp.trans_b() == true && algo_desp.trans_c() == false){
    if (algo_desp.tile_shape == std::array<int, 3>{32, 32, 16}){
      if (algo_desp.warp_tile_shape == std::array<int, 3>{16, 16, 16}){
        if (algo_desp.num_stage == 2 && algo_desp.dacc == 0 && algo_desp.dcomp == 0){
          if ((params.split_k_slices == 1)){
            if (algo_desp.access_per_vector == 0){
              if (algo_desp.tensorop == std::array<int, 3>{16, 8, 8}){
                if (algo_desp.ndim == 3 && static_cast<int>(algo_desp.op_type) == 0 && static_cast<int>(algo_desp.iter_algo) == 1){
                  if (static_cast<int>(algo_desp.layout_i) == 1 && static_cast<int>(algo_desp.layout_w) == 1 && static_cast<int>(algo_desp.layout_o) == 1){
                    if (algo_desp.interleave_i == 1 && algo_desp.interleave_w == 1 && algo_desp.interleave_o == 1){
                      if (algo_desp.mask_sparse == true && algo_desp.increment_k_first == true && algo_desp.is_int8_inference == false && algo_desp.dynamic_mask == false){
                        TV_ASSERT_RT_ERR("algo don't support splitk but you provide split_k_slices > 1.", split_k_slices);
                        // Ampere_f32f32f32f32f32tnt_m32n32k16m16n16k16A0T1688_200_C301LLL_SK
                        found = true;
                        bool d_is_bias = !bias.empty();
                        TV_ASSERT_RT_ERR(!indices.empty(), "error");
                        TV_ASSERT_RT_ERR(!mask.empty(), "error");
                        TV_ASSERT_RT_ERR(!mask_argsort.empty(), "error");
                        int kernel_volume = weight.dim(1);
                        tv::check_shape(indices, {kernel_volume, -1});
                        N = indices.dim(1);
                        cpAmpere_f32f32f32f32f32tnt_m32n32k16m16n16k16A0T1688_200_C301LLL_SK::ConvProblem problem(N, C, K, kernel_volume, 
                            tv::gemm::ConvMode::kCrossCorrelation, split_k_slices, groups);
                        TV_ASSERT_RT_ERR(N == output.dim(0), "error");
                        TV_ASSERT_RT_ERR(int64_t(N) * int64_t(C) * 32 / 8 < std::numeric_limits<int32_t>::max(), 
                            "your data exceed int32 range. this will be fixed in cumm + nvrtc (spconv 2.2/2.3).");
                        if (!algo_desp.is_int8_inference){
                            TV_ASSERT_INVALID_ARG(output_add.empty(), "only int8 inference support output_add not empty ")
                        }
                        auto source_ptr = algo_desp.is_int8_inference ? c_ten.data_ptr<const float>() : (bias.empty() ? c_ten.data_ptr<const float>() : bias.data_ptr<const float>());
                        if (!output_add.empty()){
                            source_ptr = output_add.data_ptr<const float>();
                        }
                        ConvParamsAmpere_f32f32f32f32f32tnt_m32n32k16m16n16k16A0T1688_200_C301LLL_SK ker_params(problem, a_ten.data_ptr<const float>(), b_ten.data_ptr<const float>(), c_ten.data_ptr<float>(), source_ptr, mask.data_ptr<const uint32_t>(), mask_argsort.data_ptr<const int32_t>(), indices.data_ptr<const int32_t>(), mask_output.empty() ? nullptr : mask_output.data_ptr<uint32_t>(), params.mask_filter, params.reverse_mask, float(params.alpha), float(params.beta), float(params.act_alpha), float(params.act_beta), params.act_type, 1, d_is_bias);
                        tv::cuda::Launch launcher(ker_params.grid_dims, dim3(128),
                                                    8192, reinterpret_cast<hipStream_t>(params.stream));
                        hipError_t result;
                        if (8192 >= (48 << 10)) {
                            result = hipFuncSetAttribute(reinterpret_cast<const void*>(Ampere_f32f32f32f32f32tnt_m32n32k16m16n16k16A0T1688_200_C301LLL_SK::conv_kernel),
                                                            hipFuncAttributeMaxDynamicSharedMemorySize,
                                                            8192);
                            TV_ASSERT_RT_ERR(result == hipSuccess, "error");
                            result = hipFuncSetAttribute(reinterpret_cast<const void*>(
                                Ampere_f32f32f32f32f32tnt_m32n32k16m16n16k16A0T1688_200_C301LLL_SK::conv_kernel),
                                hipFuncAttributePreferredSharedMemoryCarveout, 100);
                            TV_ASSERT_RT_ERR(result == hipSuccess, "error");
                        }
                        auto timer = tv::CUDATimer(params.verbose);
                        // tv::ssprint("CPU Time", rtxtimer.report() / 1000.0);
                        {
                            tv::CUDAKernelTimerGuard timerguard("Ampere_f32f32f32f32f32tnt_m32n32k16m16n16k16A0T1688_200_C301LLL_SK", evtimer, reinterpret_cast<hipStream_t>(params.stream));
                            launcher(Ampere_f32f32f32f32f32tnt_m32n32k16m16n16k16A0T1688_200_C301LLL_SK::conv_kernel, ker_params);
                        }
                        TV_CHECK_CUDA_ERR_V2("Ampere_f32f32f32f32f32tnt_m32n32k16m16n16k16A0T1688_200_C301LLL_SK", "error with params", input.shape(), weight.shape(), output.shape(), 
                            indices.shape(), mask.shape(), mask_argsort.shape(), mask_output.shape(), mask_width);
                        if (params.verbose){
                            hipFuncAttributes attr;
                            checkCudaErrors(
                                hipFuncGetAttributes(&attr, reinterpret_cast<const void*>(Ampere_f32f32f32f32f32tnt_m32n32k16m16n16k16A0T1688_200_C301LLL_SK::conv_kernel)));
                            tv::ssprint("Ampere_f32f32f32f32f32tnt_m32n32k16m16n16k16A0T1688_200_C301LLL_SK kernel num regs:", attr.numRegs, "time:", timer.report() / 1000.0);
                        }
                        return;
                      }
                    }
                  }
                }
              }
            }
            if (algo_desp.access_per_vector == 1){
              if (algo_desp.tensorop == std::array<int, 3>{16, 8, 8}){
                if (algo_desp.ndim == 3 && static_cast<int>(algo_desp.op_type) == 0 && static_cast<int>(algo_desp.iter_algo) == 1){
                  if (static_cast<int>(algo_desp.layout_i) == 1 && static_cast<int>(algo_desp.layout_w) == 1 && static_cast<int>(algo_desp.layout_o) == 1){
                    if (algo_desp.interleave_i == 1 && algo_desp.interleave_w == 1 && algo_desp.interleave_o == 1){
                      if (algo_desp.mask_sparse == true && algo_desp.increment_k_first == true && algo_desp.is_int8_inference == false && algo_desp.dynamic_mask == false){
                        TV_ASSERT_RT_ERR("algo don't support splitk but you provide split_k_slices > 1.", split_k_slices);
                        // Ampere_f32f32f32f32f32tnt_m32n32k16m16n16k16A1T1688_200_C301LLL_SK
                        found = true;
                        bool d_is_bias = !bias.empty();
                        TV_ASSERT_RT_ERR(!indices.empty(), "error");
                        TV_ASSERT_RT_ERR(!mask.empty(), "error");
                        TV_ASSERT_RT_ERR(!mask_argsort.empty(), "error");
                        int kernel_volume = weight.dim(1);
                        tv::check_shape(indices, {kernel_volume, -1});
                        N = indices.dim(1);
                        cpAmpere_f32f32f32f32f32tnt_m32n32k16m16n16k16A1T1688_200_C301LLL_SK::ConvProblem problem(N, C, K, kernel_volume, 
                            tv::gemm::ConvMode::kCrossCorrelation, split_k_slices, groups);
                        TV_ASSERT_RT_ERR(N == output.dim(0), "error");
                        TV_ASSERT_RT_ERR(int64_t(N) * int64_t(C) * 32 / 8 < std::numeric_limits<int32_t>::max(), 
                            "your data exceed int32 range. this will be fixed in cumm + nvrtc (spconv 2.2/2.3).");
                        if (!algo_desp.is_int8_inference){
                            TV_ASSERT_INVALID_ARG(output_add.empty(), "only int8 inference support output_add not empty ")
                        }
                        auto source_ptr = algo_desp.is_int8_inference ? c_ten.data_ptr<const float>() : (bias.empty() ? c_ten.data_ptr<const float>() : bias.data_ptr<const float>());
                        if (!output_add.empty()){
                            source_ptr = output_add.data_ptr<const float>();
                        }
                        ConvParamsAmpere_f32f32f32f32f32tnt_m32n32k16m16n16k16A1T1688_200_C301LLL_SK ker_params(problem, a_ten.data_ptr<const float>(), b_ten.data_ptr<const float>(), c_ten.data_ptr<float>(), source_ptr, mask.data_ptr<const uint32_t>(), mask_argsort.data_ptr<const int32_t>(), indices.data_ptr<const int32_t>(), mask_output.empty() ? nullptr : mask_output.data_ptr<uint32_t>(), params.mask_filter, params.reverse_mask, float(params.alpha), float(params.beta), float(params.act_alpha), float(params.act_beta), params.act_type, 1, d_is_bias);
                        tv::cuda::Launch launcher(ker_params.grid_dims, dim3(128),
                                                    8192, reinterpret_cast<hipStream_t>(params.stream));
                        hipError_t result;
                        if (8192 >= (48 << 10)) {
                            result = hipFuncSetAttribute(reinterpret_cast<const void*>(Ampere_f32f32f32f32f32tnt_m32n32k16m16n16k16A1T1688_200_C301LLL_SK::conv_kernel),
                                                            hipFuncAttributeMaxDynamicSharedMemorySize,
                                                            8192);
                            TV_ASSERT_RT_ERR(result == hipSuccess, "error");
                            result = hipFuncSetAttribute(reinterpret_cast<const void*>(
                                Ampere_f32f32f32f32f32tnt_m32n32k16m16n16k16A1T1688_200_C301LLL_SK::conv_kernel),
                                hipFuncAttributePreferredSharedMemoryCarveout, 100);
                            TV_ASSERT_RT_ERR(result == hipSuccess, "error");
                        }
                        auto timer = tv::CUDATimer(params.verbose);
                        // tv::ssprint("CPU Time", rtxtimer.report() / 1000.0);
                        {
                            tv::CUDAKernelTimerGuard timerguard("Ampere_f32f32f32f32f32tnt_m32n32k16m16n16k16A1T1688_200_C301LLL_SK", evtimer, reinterpret_cast<hipStream_t>(params.stream));
                            launcher(Ampere_f32f32f32f32f32tnt_m32n32k16m16n16k16A1T1688_200_C301LLL_SK::conv_kernel, ker_params);
                        }
                        TV_CHECK_CUDA_ERR_V2("Ampere_f32f32f32f32f32tnt_m32n32k16m16n16k16A1T1688_200_C301LLL_SK", "error with params", input.shape(), weight.shape(), output.shape(), 
                            indices.shape(), mask.shape(), mask_argsort.shape(), mask_output.shape(), mask_width);
                        if (params.verbose){
                            hipFuncAttributes attr;
                            checkCudaErrors(
                                hipFuncGetAttributes(&attr, reinterpret_cast<const void*>(Ampere_f32f32f32f32f32tnt_m32n32k16m16n16k16A1T1688_200_C301LLL_SK::conv_kernel)));
                            tv::ssprint("Ampere_f32f32f32f32f32tnt_m32n32k16m16n16k16A1T1688_200_C301LLL_SK kernel num regs:", attr.numRegs, "time:", timer.report() / 1000.0);
                        }
                        return;
                      }
                    }
                  }
                }
              }
            }
          }
        }
        if (algo_desp.num_stage == 3 && algo_desp.dacc == 0 && algo_desp.dcomp == 0){
          if ((params.split_k_slices == 1)){
            if (algo_desp.access_per_vector == 1){
              if (algo_desp.tensorop == std::array<int, 3>{16, 8, 8}){
                if (algo_desp.ndim == 3 && static_cast<int>(algo_desp.op_type) == 0 && static_cast<int>(algo_desp.iter_algo) == 1){
                  if (static_cast<int>(algo_desp.layout_i) == 1 && static_cast<int>(algo_desp.layout_w) == 1 && static_cast<int>(algo_desp.layout_o) == 1){
                    if (algo_desp.interleave_i == 1 && algo_desp.interleave_w == 1 && algo_desp.interleave_o == 1){
                      if (algo_desp.mask_sparse == true && algo_desp.increment_k_first == true && algo_desp.is_int8_inference == false && algo_desp.dynamic_mask == false){
                        TV_ASSERT_RT_ERR("algo don't support splitk but you provide split_k_slices > 1.", split_k_slices);
                        // Ampere_f32f32f32f32f32tnt_m32n32k16m16n16k16A1T1688_300_C301LLL_SK
                        found = true;
                        bool d_is_bias = !bias.empty();
                        TV_ASSERT_RT_ERR(!indices.empty(), "error");
                        TV_ASSERT_RT_ERR(!mask.empty(), "error");
                        TV_ASSERT_RT_ERR(!mask_argsort.empty(), "error");
                        int kernel_volume = weight.dim(1);
                        tv::check_shape(indices, {kernel_volume, -1});
                        N = indices.dim(1);
                        cpAmpere_f32f32f32f32f32tnt_m32n32k16m16n16k16A1T1688_300_C301LLL_SK::ConvProblem problem(N, C, K, kernel_volume, 
                            tv::gemm::ConvMode::kCrossCorrelation, split_k_slices, groups);
                        TV_ASSERT_RT_ERR(N == output.dim(0), "error");
                        TV_ASSERT_RT_ERR(int64_t(N) * int64_t(C) * 32 / 8 < std::numeric_limits<int32_t>::max(), 
                            "your data exceed int32 range. this will be fixed in cumm + nvrtc (spconv 2.2/2.3).");
                        if (!algo_desp.is_int8_inference){
                            TV_ASSERT_INVALID_ARG(output_add.empty(), "only int8 inference support output_add not empty ")
                        }
                        auto source_ptr = algo_desp.is_int8_inference ? c_ten.data_ptr<const float>() : (bias.empty() ? c_ten.data_ptr<const float>() : bias.data_ptr<const float>());
                        if (!output_add.empty()){
                            source_ptr = output_add.data_ptr<const float>();
                        }
                        ConvParamsAmpere_f32f32f32f32f32tnt_m32n32k16m16n16k16A1T1688_300_C301LLL_SK ker_params(problem, a_ten.data_ptr<const float>(), b_ten.data_ptr<const float>(), c_ten.data_ptr<float>(), source_ptr, mask.data_ptr<const uint32_t>(), mask_argsort.data_ptr<const int32_t>(), indices.data_ptr<const int32_t>(), mask_output.empty() ? nullptr : mask_output.data_ptr<uint32_t>(), params.mask_filter, params.reverse_mask, float(params.alpha), float(params.beta), float(params.act_alpha), float(params.act_beta), params.act_type, 1, d_is_bias);
                        tv::cuda::Launch launcher(ker_params.grid_dims, dim3(128),
                                                    12288, reinterpret_cast<hipStream_t>(params.stream));
                        hipError_t result;
                        if (12288 >= (48 << 10)) {
                            result = hipFuncSetAttribute(reinterpret_cast<const void*>(Ampere_f32f32f32f32f32tnt_m32n32k16m16n16k16A1T1688_300_C301LLL_SK::conv_kernel),
                                                            hipFuncAttributeMaxDynamicSharedMemorySize,
                                                            12288);
                            TV_ASSERT_RT_ERR(result == hipSuccess, "error");
                            result = hipFuncSetAttribute(reinterpret_cast<const void*>(
                                Ampere_f32f32f32f32f32tnt_m32n32k16m16n16k16A1T1688_300_C301LLL_SK::conv_kernel),
                                hipFuncAttributePreferredSharedMemoryCarveout, 100);
                            TV_ASSERT_RT_ERR(result == hipSuccess, "error");
                        }
                        auto timer = tv::CUDATimer(params.verbose);
                        // tv::ssprint("CPU Time", rtxtimer.report() / 1000.0);
                        {
                            tv::CUDAKernelTimerGuard timerguard("Ampere_f32f32f32f32f32tnt_m32n32k16m16n16k16A1T1688_300_C301LLL_SK", evtimer, reinterpret_cast<hipStream_t>(params.stream));
                            launcher(Ampere_f32f32f32f32f32tnt_m32n32k16m16n16k16A1T1688_300_C301LLL_SK::conv_kernel, ker_params);
                        }
                        TV_CHECK_CUDA_ERR_V2("Ampere_f32f32f32f32f32tnt_m32n32k16m16n16k16A1T1688_300_C301LLL_SK", "error with params", input.shape(), weight.shape(), output.shape(), 
                            indices.shape(), mask.shape(), mask_argsort.shape(), mask_output.shape(), mask_width);
                        if (params.verbose){
                            hipFuncAttributes attr;
                            checkCudaErrors(
                                hipFuncGetAttributes(&attr, reinterpret_cast<const void*>(Ampere_f32f32f32f32f32tnt_m32n32k16m16n16k16A1T1688_300_C301LLL_SK::conv_kernel)));
                            tv::ssprint("Ampere_f32f32f32f32f32tnt_m32n32k16m16n16k16A1T1688_300_C301LLL_SK kernel num regs:", attr.numRegs, "time:", timer.report() / 1000.0);
                        }
                        return;
                      }
                    }
                  }
                }
              }
            }
          }
        }
        if (algo_desp.num_stage == 4 && algo_desp.dacc == 0 && algo_desp.dcomp == 0){
          if ((params.split_k_slices == 1)){
            if (algo_desp.access_per_vector == 1){
              if (algo_desp.tensorop == std::array<int, 3>{16, 8, 8}){
                if (algo_desp.ndim == 3 && static_cast<int>(algo_desp.op_type) == 0 && static_cast<int>(algo_desp.iter_algo) == 1){
                  if (static_cast<int>(algo_desp.layout_i) == 1 && static_cast<int>(algo_desp.layout_w) == 1 && static_cast<int>(algo_desp.layout_o) == 1){
                    if (algo_desp.interleave_i == 1 && algo_desp.interleave_w == 1 && algo_desp.interleave_o == 1){
                      if (algo_desp.mask_sparse == true && algo_desp.increment_k_first == true && algo_desp.is_int8_inference == false && algo_desp.dynamic_mask == false){
                        TV_ASSERT_RT_ERR("algo don't support splitk but you provide split_k_slices > 1.", split_k_slices);
                        // Ampere_f32f32f32f32f32tnt_m32n32k16m16n16k16A1T1688_400_C301LLL_SK
                        found = true;
                        bool d_is_bias = !bias.empty();
                        TV_ASSERT_RT_ERR(!indices.empty(), "error");
                        TV_ASSERT_RT_ERR(!mask.empty(), "error");
                        TV_ASSERT_RT_ERR(!mask_argsort.empty(), "error");
                        int kernel_volume = weight.dim(1);
                        tv::check_shape(indices, {kernel_volume, -1});
                        N = indices.dim(1);
                        cpAmpere_f32f32f32f32f32tnt_m32n32k16m16n16k16A1T1688_400_C301LLL_SK::ConvProblem problem(N, C, K, kernel_volume, 
                            tv::gemm::ConvMode::kCrossCorrelation, split_k_slices, groups);
                        TV_ASSERT_RT_ERR(N == output.dim(0), "error");
                        TV_ASSERT_RT_ERR(int64_t(N) * int64_t(C) * 32 / 8 < std::numeric_limits<int32_t>::max(), 
                            "your data exceed int32 range. this will be fixed in cumm + nvrtc (spconv 2.2/2.3).");
                        if (!algo_desp.is_int8_inference){
                            TV_ASSERT_INVALID_ARG(output_add.empty(), "only int8 inference support output_add not empty ")
                        }
                        auto source_ptr = algo_desp.is_int8_inference ? c_ten.data_ptr<const float>() : (bias.empty() ? c_ten.data_ptr<const float>() : bias.data_ptr<const float>());
                        if (!output_add.empty()){
                            source_ptr = output_add.data_ptr<const float>();
                        }
                        ConvParamsAmpere_f32f32f32f32f32tnt_m32n32k16m16n16k16A1T1688_400_C301LLL_SK ker_params(problem, a_ten.data_ptr<const float>(), b_ten.data_ptr<const float>(), c_ten.data_ptr<float>(), source_ptr, mask.data_ptr<const uint32_t>(), mask_argsort.data_ptr<const int32_t>(), indices.data_ptr<const int32_t>(), mask_output.empty() ? nullptr : mask_output.data_ptr<uint32_t>(), params.mask_filter, params.reverse_mask, float(params.alpha), float(params.beta), float(params.act_alpha), float(params.act_beta), params.act_type, 1, d_is_bias);
                        tv::cuda::Launch launcher(ker_params.grid_dims, dim3(128),
                                                    16384, reinterpret_cast<hipStream_t>(params.stream));
                        hipError_t result;
                        if (16384 >= (48 << 10)) {
                            result = hipFuncSetAttribute(reinterpret_cast<const void*>(Ampere_f32f32f32f32f32tnt_m32n32k16m16n16k16A1T1688_400_C301LLL_SK::conv_kernel),
                                                            hipFuncAttributeMaxDynamicSharedMemorySize,
                                                            16384);
                            TV_ASSERT_RT_ERR(result == hipSuccess, "error");
                            result = hipFuncSetAttribute(reinterpret_cast<const void*>(
                                Ampere_f32f32f32f32f32tnt_m32n32k16m16n16k16A1T1688_400_C301LLL_SK::conv_kernel),
                                hipFuncAttributePreferredSharedMemoryCarveout, 100);
                            TV_ASSERT_RT_ERR(result == hipSuccess, "error");
                        }
                        auto timer = tv::CUDATimer(params.verbose);
                        // tv::ssprint("CPU Time", rtxtimer.report() / 1000.0);
                        {
                            tv::CUDAKernelTimerGuard timerguard("Ampere_f32f32f32f32f32tnt_m32n32k16m16n16k16A1T1688_400_C301LLL_SK", evtimer, reinterpret_cast<hipStream_t>(params.stream));
                            launcher(Ampere_f32f32f32f32f32tnt_m32n32k16m16n16k16A1T1688_400_C301LLL_SK::conv_kernel, ker_params);
                        }
                        TV_CHECK_CUDA_ERR_V2("Ampere_f32f32f32f32f32tnt_m32n32k16m16n16k16A1T1688_400_C301LLL_SK", "error with params", input.shape(), weight.shape(), output.shape(), 
                            indices.shape(), mask.shape(), mask_argsort.shape(), mask_output.shape(), mask_width);
                        if (params.verbose){
                            hipFuncAttributes attr;
                            checkCudaErrors(
                                hipFuncGetAttributes(&attr, reinterpret_cast<const void*>(Ampere_f32f32f32f32f32tnt_m32n32k16m16n16k16A1T1688_400_C301LLL_SK::conv_kernel)));
                            tv::ssprint("Ampere_f32f32f32f32f32tnt_m32n32k16m16n16k16A1T1688_400_C301LLL_SK kernel num regs:", attr.numRegs, "time:", timer.report() / 1000.0);
                        }
                        return;
                      }
                    }
                  }
                }
              }
            }
          }
        }
      }
    }
    if (algo_desp.tile_shape == std::array<int, 3>{64, 64, 32}){
      if (algo_desp.warp_tile_shape == std::array<int, 3>{32, 32, 32}){
        if (algo_desp.num_stage == 2 && algo_desp.dacc == 0 && algo_desp.dcomp == 0){
          if ((params.split_k_slices == 1)){
            if (algo_desp.access_per_vector == 1){
              if (algo_desp.tensorop == std::array<int, 3>{16, 8, 8}){
                if (algo_desp.ndim == 3 && static_cast<int>(algo_desp.op_type) == 0 && static_cast<int>(algo_desp.iter_algo) == 1){
                  if (static_cast<int>(algo_desp.layout_i) == 1 && static_cast<int>(algo_desp.layout_w) == 1 && static_cast<int>(algo_desp.layout_o) == 1){
                    if (algo_desp.interleave_i == 1 && algo_desp.interleave_w == 1 && algo_desp.interleave_o == 1){
                      if (algo_desp.mask_sparse == true && algo_desp.increment_k_first == true && algo_desp.is_int8_inference == false && algo_desp.dynamic_mask == false){
                        TV_ASSERT_RT_ERR("algo don't support splitk but you provide split_k_slices > 1.", split_k_slices);
                        // Ampere_f32f32f32f32f32tnt_m64n64k32m32n32k32A1T1688_200_C301LLL_SK
                        found = true;
                        bool d_is_bias = !bias.empty();
                        TV_ASSERT_RT_ERR(!indices.empty(), "error");
                        TV_ASSERT_RT_ERR(!mask.empty(), "error");
                        TV_ASSERT_RT_ERR(!mask_argsort.empty(), "error");
                        int kernel_volume = weight.dim(1);
                        tv::check_shape(indices, {kernel_volume, -1});
                        N = indices.dim(1);
                        cpAmpere_f32f32f32f32f32tnt_m64n64k32m32n32k32A1T1688_200_C301LLL_SK::ConvProblem problem(N, C, K, kernel_volume, 
                            tv::gemm::ConvMode::kCrossCorrelation, split_k_slices, groups);
                        TV_ASSERT_RT_ERR(N == output.dim(0), "error");
                        TV_ASSERT_RT_ERR(int64_t(N) * int64_t(C) * 32 / 8 < std::numeric_limits<int32_t>::max(), 
                            "your data exceed int32 range. this will be fixed in cumm + nvrtc (spconv 2.2/2.3).");
                        if (!algo_desp.is_int8_inference){
                            TV_ASSERT_INVALID_ARG(output_add.empty(), "only int8 inference support output_add not empty ")
                        }
                        auto source_ptr = algo_desp.is_int8_inference ? c_ten.data_ptr<const float>() : (bias.empty() ? c_ten.data_ptr<const float>() : bias.data_ptr<const float>());
                        if (!output_add.empty()){
                            source_ptr = output_add.data_ptr<const float>();
                        }
                        ConvParamsAmpere_f32f32f32f32f32tnt_m64n64k32m32n32k32A1T1688_200_C301LLL_SK ker_params(problem, a_ten.data_ptr<const float>(), b_ten.data_ptr<const float>(), c_ten.data_ptr<float>(), source_ptr, mask.data_ptr<const uint32_t>(), mask_argsort.data_ptr<const int32_t>(), indices.data_ptr<const int32_t>(), mask_output.empty() ? nullptr : mask_output.data_ptr<uint32_t>(), params.mask_filter, params.reverse_mask, float(params.alpha), float(params.beta), float(params.act_alpha), float(params.act_beta), params.act_type, 1, d_is_bias);
                        tv::cuda::Launch launcher(ker_params.grid_dims, dim3(128),
                                                    32768, reinterpret_cast<hipStream_t>(params.stream));
                        hipError_t result;
                        if (32768 >= (48 << 10)) {
                            result = hipFuncSetAttribute(reinterpret_cast<const void*>(Ampere_f32f32f32f32f32tnt_m64n64k32m32n32k32A1T1688_200_C301LLL_SK::conv_kernel),
                                                            hipFuncAttributeMaxDynamicSharedMemorySize,
                                                            32768);
                            TV_ASSERT_RT_ERR(result == hipSuccess, "error");
                            result = hipFuncSetAttribute(reinterpret_cast<const void*>(
                                Ampere_f32f32f32f32f32tnt_m64n64k32m32n32k32A1T1688_200_C301LLL_SK::conv_kernel),
                                hipFuncAttributePreferredSharedMemoryCarveout, 100);
                            TV_ASSERT_RT_ERR(result == hipSuccess, "error");
                        }
                        auto timer = tv::CUDATimer(params.verbose);
                        // tv::ssprint("CPU Time", rtxtimer.report() / 1000.0);
                        {
                            tv::CUDAKernelTimerGuard timerguard("Ampere_f32f32f32f32f32tnt_m64n64k32m32n32k32A1T1688_200_C301LLL_SK", evtimer, reinterpret_cast<hipStream_t>(params.stream));
                            launcher(Ampere_f32f32f32f32f32tnt_m64n64k32m32n32k32A1T1688_200_C301LLL_SK::conv_kernel, ker_params);
                        }
                        TV_CHECK_CUDA_ERR_V2("Ampere_f32f32f32f32f32tnt_m64n64k32m32n32k32A1T1688_200_C301LLL_SK", "error with params", input.shape(), weight.shape(), output.shape(), 
                            indices.shape(), mask.shape(), mask_argsort.shape(), mask_output.shape(), mask_width);
                        if (params.verbose){
                            hipFuncAttributes attr;
                            checkCudaErrors(
                                hipFuncGetAttributes(&attr, reinterpret_cast<const void*>(Ampere_f32f32f32f32f32tnt_m64n64k32m32n32k32A1T1688_200_C301LLL_SK::conv_kernel)));
                            tv::ssprint("Ampere_f32f32f32f32f32tnt_m64n64k32m32n32k32A1T1688_200_C301LLL_SK kernel num regs:", attr.numRegs, "time:", timer.report() / 1000.0);
                        }
                        return;
                      }
                    }
                  }
                }
              }
            }
          }
        }
        if (algo_desp.num_stage == 3 && algo_desp.dacc == 0 && algo_desp.dcomp == 0){
          if ((params.split_k_slices == 1)){
            if (algo_desp.access_per_vector == 1){
              if (algo_desp.tensorop == std::array<int, 3>{16, 8, 8}){
                if (algo_desp.ndim == 3 && static_cast<int>(algo_desp.op_type) == 0 && static_cast<int>(algo_desp.iter_algo) == 1){
                  if (static_cast<int>(algo_desp.layout_i) == 1 && static_cast<int>(algo_desp.layout_w) == 1 && static_cast<int>(algo_desp.layout_o) == 1){
                    if (algo_desp.interleave_i == 1 && algo_desp.interleave_w == 1 && algo_desp.interleave_o == 1){
                      if (algo_desp.mask_sparse == true && algo_desp.increment_k_first == true && algo_desp.is_int8_inference == false && algo_desp.dynamic_mask == false){
                        TV_ASSERT_RT_ERR("algo don't support splitk but you provide split_k_slices > 1.", split_k_slices);
                        // Ampere_f32f32f32f32f32tnt_m64n64k32m32n32k32A1T1688_300_C301LLL_SK
                        found = true;
                        bool d_is_bias = !bias.empty();
                        TV_ASSERT_RT_ERR(!indices.empty(), "error");
                        TV_ASSERT_RT_ERR(!mask.empty(), "error");
                        TV_ASSERT_RT_ERR(!mask_argsort.empty(), "error");
                        int kernel_volume = weight.dim(1);
                        tv::check_shape(indices, {kernel_volume, -1});
                        N = indices.dim(1);
                        cpAmpere_f32f32f32f32f32tnt_m64n64k32m32n32k32A1T1688_300_C301LLL_SK::ConvProblem problem(N, C, K, kernel_volume, 
                            tv::gemm::ConvMode::kCrossCorrelation, split_k_slices, groups);
                        TV_ASSERT_RT_ERR(N == output.dim(0), "error");
                        TV_ASSERT_RT_ERR(int64_t(N) * int64_t(C) * 32 / 8 < std::numeric_limits<int32_t>::max(), 
                            "your data exceed int32 range. this will be fixed in cumm + nvrtc (spconv 2.2/2.3).");
                        if (!algo_desp.is_int8_inference){
                            TV_ASSERT_INVALID_ARG(output_add.empty(), "only int8 inference support output_add not empty ")
                        }
                        auto source_ptr = algo_desp.is_int8_inference ? c_ten.data_ptr<const float>() : (bias.empty() ? c_ten.data_ptr<const float>() : bias.data_ptr<const float>());
                        if (!output_add.empty()){
                            source_ptr = output_add.data_ptr<const float>();
                        }
                        ConvParamsAmpere_f32f32f32f32f32tnt_m64n64k32m32n32k32A1T1688_300_C301LLL_SK ker_params(problem, a_ten.data_ptr<const float>(), b_ten.data_ptr<const float>(), c_ten.data_ptr<float>(), source_ptr, mask.data_ptr<const uint32_t>(), mask_argsort.data_ptr<const int32_t>(), indices.data_ptr<const int32_t>(), mask_output.empty() ? nullptr : mask_output.data_ptr<uint32_t>(), params.mask_filter, params.reverse_mask, float(params.alpha), float(params.beta), float(params.act_alpha), float(params.act_beta), params.act_type, 1, d_is_bias);
                        tv::cuda::Launch launcher(ker_params.grid_dims, dim3(128),
                                                    49152, reinterpret_cast<hipStream_t>(params.stream));
                        hipError_t result;
                        if (49152 >= (48 << 10)) {
                            result = hipFuncSetAttribute(reinterpret_cast<const void*>(Ampere_f32f32f32f32f32tnt_m64n64k32m32n32k32A1T1688_300_C301LLL_SK::conv_kernel),
                                                            hipFuncAttributeMaxDynamicSharedMemorySize,
                                                            49152);
                            TV_ASSERT_RT_ERR(result == hipSuccess, "error");
                            result = hipFuncSetAttribute(reinterpret_cast<const void*>(
                                Ampere_f32f32f32f32f32tnt_m64n64k32m32n32k32A1T1688_300_C301LLL_SK::conv_kernel),
                                hipFuncAttributePreferredSharedMemoryCarveout, 100);
                            TV_ASSERT_RT_ERR(result == hipSuccess, "error");
                        }
                        auto timer = tv::CUDATimer(params.verbose);
                        // tv::ssprint("CPU Time", rtxtimer.report() / 1000.0);
                        {
                            tv::CUDAKernelTimerGuard timerguard("Ampere_f32f32f32f32f32tnt_m64n64k32m32n32k32A1T1688_300_C301LLL_SK", evtimer, reinterpret_cast<hipStream_t>(params.stream));
                            launcher(Ampere_f32f32f32f32f32tnt_m64n64k32m32n32k32A1T1688_300_C301LLL_SK::conv_kernel, ker_params);
                        }
                        TV_CHECK_CUDA_ERR_V2("Ampere_f32f32f32f32f32tnt_m64n64k32m32n32k32A1T1688_300_C301LLL_SK", "error with params", input.shape(), weight.shape(), output.shape(), 
                            indices.shape(), mask.shape(), mask_argsort.shape(), mask_output.shape(), mask_width);
                        if (params.verbose){
                            hipFuncAttributes attr;
                            checkCudaErrors(
                                hipFuncGetAttributes(&attr, reinterpret_cast<const void*>(Ampere_f32f32f32f32f32tnt_m64n64k32m32n32k32A1T1688_300_C301LLL_SK::conv_kernel)));
                            tv::ssprint("Ampere_f32f32f32f32f32tnt_m64n64k32m32n32k32A1T1688_300_C301LLL_SK kernel num regs:", attr.numRegs, "time:", timer.report() / 1000.0);
                        }
                        return;
                      }
                    }
                  }
                }
              }
            }
          }
        }
        if (algo_desp.num_stage == 4 && algo_desp.dacc == 0 && algo_desp.dcomp == 0){
          if ((params.split_k_slices == 1)){
            if (algo_desp.access_per_vector == 1){
              if (algo_desp.tensorop == std::array<int, 3>{16, 8, 8}){
                if (algo_desp.ndim == 3 && static_cast<int>(algo_desp.op_type) == 0 && static_cast<int>(algo_desp.iter_algo) == 1){
                  if (static_cast<int>(algo_desp.layout_i) == 1 && static_cast<int>(algo_desp.layout_w) == 1 && static_cast<int>(algo_desp.layout_o) == 1){
                    if (algo_desp.interleave_i == 1 && algo_desp.interleave_w == 1 && algo_desp.interleave_o == 1){
                      if (algo_desp.mask_sparse == true && algo_desp.increment_k_first == true && algo_desp.is_int8_inference == false && algo_desp.dynamic_mask == false){
                        TV_ASSERT_RT_ERR("algo don't support splitk but you provide split_k_slices > 1.", split_k_slices);
                        // Ampere_f32f32f32f32f32tnt_m64n64k32m32n32k32A1T1688_400_C301LLL_SK
                        found = true;
                        bool d_is_bias = !bias.empty();
                        TV_ASSERT_RT_ERR(!indices.empty(), "error");
                        TV_ASSERT_RT_ERR(!mask.empty(), "error");
                        TV_ASSERT_RT_ERR(!mask_argsort.empty(), "error");
                        int kernel_volume = weight.dim(1);
                        tv::check_shape(indices, {kernel_volume, -1});
                        N = indices.dim(1);
                        cpAmpere_f32f32f32f32f32tnt_m64n64k32m32n32k32A1T1688_400_C301LLL_SK::ConvProblem problem(N, C, K, kernel_volume, 
                            tv::gemm::ConvMode::kCrossCorrelation, split_k_slices, groups);
                        TV_ASSERT_RT_ERR(N == output.dim(0), "error");
                        TV_ASSERT_RT_ERR(int64_t(N) * int64_t(C) * 32 / 8 < std::numeric_limits<int32_t>::max(), 
                            "your data exceed int32 range. this will be fixed in cumm + nvrtc (spconv 2.2/2.3).");
                        if (!algo_desp.is_int8_inference){
                            TV_ASSERT_INVALID_ARG(output_add.empty(), "only int8 inference support output_add not empty ")
                        }
                        auto source_ptr = algo_desp.is_int8_inference ? c_ten.data_ptr<const float>() : (bias.empty() ? c_ten.data_ptr<const float>() : bias.data_ptr<const float>());
                        if (!output_add.empty()){
                            source_ptr = output_add.data_ptr<const float>();
                        }
                        ConvParamsAmpere_f32f32f32f32f32tnt_m64n64k32m32n32k32A1T1688_400_C301LLL_SK ker_params(problem, a_ten.data_ptr<const float>(), b_ten.data_ptr<const float>(), c_ten.data_ptr<float>(), source_ptr, mask.data_ptr<const uint32_t>(), mask_argsort.data_ptr<const int32_t>(), indices.data_ptr<const int32_t>(), mask_output.empty() ? nullptr : mask_output.data_ptr<uint32_t>(), params.mask_filter, params.reverse_mask, float(params.alpha), float(params.beta), float(params.act_alpha), float(params.act_beta), params.act_type, 1, d_is_bias);
                        tv::cuda::Launch launcher(ker_params.grid_dims, dim3(128),
                                                    65536, reinterpret_cast<hipStream_t>(params.stream));
                        hipError_t result;
                        if (65536 >= (48 << 10)) {
                            result = hipFuncSetAttribute(reinterpret_cast<const void*>(Ampere_f32f32f32f32f32tnt_m64n64k32m32n32k32A1T1688_400_C301LLL_SK::conv_kernel),
                                                            hipFuncAttributeMaxDynamicSharedMemorySize,
                                                            65536);
                            TV_ASSERT_RT_ERR(result == hipSuccess, "error");
                            result = hipFuncSetAttribute(reinterpret_cast<const void*>(
                                Ampere_f32f32f32f32f32tnt_m64n64k32m32n32k32A1T1688_400_C301LLL_SK::conv_kernel),
                                hipFuncAttributePreferredSharedMemoryCarveout, 100);
                            TV_ASSERT_RT_ERR(result == hipSuccess, "error");
                        }
                        auto timer = tv::CUDATimer(params.verbose);
                        // tv::ssprint("CPU Time", rtxtimer.report() / 1000.0);
                        {
                            tv::CUDAKernelTimerGuard timerguard("Ampere_f32f32f32f32f32tnt_m64n64k32m32n32k32A1T1688_400_C301LLL_SK", evtimer, reinterpret_cast<hipStream_t>(params.stream));
                            launcher(Ampere_f32f32f32f32f32tnt_m64n64k32m32n32k32A1T1688_400_C301LLL_SK::conv_kernel, ker_params);
                        }
                        TV_CHECK_CUDA_ERR_V2("Ampere_f32f32f32f32f32tnt_m64n64k32m32n32k32A1T1688_400_C301LLL_SK", "error with params", input.shape(), weight.shape(), output.shape(), 
                            indices.shape(), mask.shape(), mask_argsort.shape(), mask_output.shape(), mask_width);
                        if (params.verbose){
                            hipFuncAttributes attr;
                            checkCudaErrors(
                                hipFuncGetAttributes(&attr, reinterpret_cast<const void*>(Ampere_f32f32f32f32f32tnt_m64n64k32m32n32k32A1T1688_400_C301LLL_SK::conv_kernel)));
                            tv::ssprint("Ampere_f32f32f32f32f32tnt_m64n64k32m32n32k32A1T1688_400_C301LLL_SK kernel num regs:", attr.numRegs, "time:", timer.report() / 1000.0);
                        }
                        return;
                      }
                    }
                  }
                }
              }
            }
          }
        }
      }
    }
  }
  if (!found){
      TV_THROW_INVALID_ARG("Can't Found Algorithm for params:", algo_desp.__repr__(), tv::dtype_str(input.dtype()), 
          tv::dtype_str(weight.dtype()), tv::dtype_str(output.dtype()), tv::dtype_str(dacc), 
          tv::dtype_str(dcomp));
  }
}
} // namespace main
} // namespace conv
} // namespace cumm
} // namespace spconvlib