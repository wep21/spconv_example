#include "hip/hip_runtime.h"
#include <spconvlib/cumm/conv/main/Ampere_f32f32f32f32f32tnt_m64n64k32m32n32k32A1T1688_300_C301LLL_SK/ConvKernel.h>
namespace spconvlib {
namespace cumm {
namespace conv {
namespace main {
namespace Ampere_f32f32f32f32f32tnt_m64n64k32m32n32k32A1T1688_300_C301LLL_SK {
using TensorViewNVRTCKernel = spconvlib::cumm::common::TensorViewNVRTCKernel;
using RowMajor = spconvlib::cumm::gemm::layout::RowMajor;
using ColumnMajor = spconvlib::cumm::gemm::layout::ColumnMajor;
using GemmBasicKernel = spconvlib::cumm::common::GemmBasicKernel;
using GemmBasic = spconvlib::cumm::common::GemmBasic;
using GemmKernelFlags = spconvlib::cumm::common::GemmKernelFlags;
using InputIteratorA = spconvlib::cumm::conv::main::Ampere_f32f32f32f32f32tnt_m64n64k32m32n32k32A1T1688_300_C301LLL_SK::inpitera::ForwardDgradSparseIOIterator;
using InputIteratorB = spconvlib::cumm::conv::main::Ampere_f32f32f32f32f32tnt_m64n64k32m32n32k32A1T1688_300_C301LLL_SK::inpiterb::WeightIteratorDP4A;
using LayoutA = spconvlib::cumm::conv::main::cpAmpere_f32f32f32f32f32tnt_m64n64k32m32n32k32A1T1688_300_C301LLL_SK::la::TensorGeneric;
using LayoutB = spconvlib::cumm::conv::main::cpAmpere_f32f32f32f32f32tnt_m64n64k32m32n32k32A1T1688_300_C301LLL_SK::lb::TensorGeneric;
using LayoutC = spconvlib::cumm::conv::main::cpAmpere_f32f32f32f32f32tnt_m64n64k32m32n32k32A1T1688_300_C301LLL_SK::lc::TensorGeneric;
using BlockMmaStorage = spconvlib::cumm::conv::main::Ampere_f32f32f32f32f32tnt_m64n64k32m32n32k32A1T1688_300_C301LLL_SK::gemm_smem_storage::BlockMmaStorage;
using OutputSmemStorage = spconvlib::cumm::conv::main::Ampere_f32f32f32f32f32tnt_m64n64k32m32n32k32A1T1688_300_C301LLL_SK::out_smem_storage::OutputSmemStorage;
using ConvParams = spconvlib::cumm::conv::main::cpAmpere_f32f32f32f32f32tnt_m64n64k32m32n32k32A1T1688_300_C301LLL_SK::ConvParams;
using ConvProblem = spconvlib::cumm::conv::main::cpAmpere_f32f32f32f32f32tnt_m64n64k32m32n32k32A1T1688_300_C301LLL_SK::cp::ConvProblem;
using OutIter = spconvlib::cumm::conv::main::Ampere_f32f32f32f32f32tnt_m64n64k32m32n32k32A1T1688_300_C301LLL_SK::out_iter::OutIterator;
using ConstOutIter = spconvlib::cumm::conv::main::Ampere_f32f32f32f32f32tnt_m64n64k32m32n32k32A1T1688_300_C301LLL_SK::out_iter_const::OutIterator;
using OutputOp = spconvlib::cumm::conv::main::Ampere_f32f32f32f32f32tnt_m64n64k32m32n32k32A1T1688_300_C301LLL_SK::out_op::LinearCombination;
using Mma = spconvlib::cumm::conv::main::Ampere_f32f32f32f32f32tnt_m64n64k32m32n32k32A1T1688_300_C301LLL_SK::mma::MmaMultiStage;
using MaskIGemmIteratorDynamic = spconvlib::cumm::conv::main::Ampere_f32f32f32f32f32tnt_m64n64k32m32n32k32A1T1688_300_C301LLL_SK::mma_miterd::MaskIGemmIteratorMaskLoaderDynamic;
using Output = spconvlib::cumm::conv::main::Ampere_f32f32f32f32f32tnt_m64n64k32m32n32k32A1T1688_300_C301LLL_SK::output::Output;
__global__ void conv_kernel(ConvParams params)   {
  
  #if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 800))
    constexpr bool kSplitKSerial = false;
    extern __shared__ uint8_t SharedStorage[];
    auto gemm_shared_mem =
        reinterpret_cast<BlockMmaStorage *>(SharedStorage);
    auto out_shared_mem =
        reinterpret_cast<OutputSmemStorage *>(SharedStorage);
    int tile_offset_m = blockIdx.x;
    int tile_offset_n = blockIdx.y;
    int tile_offset_k = blockIdx.z;
    if (tile_offset_m >= params.grid_dims.x ||
        tile_offset_n >= params.grid_dims.y) {
        return;
    }
    tv::array<int, 2> block_offset_A{tile_offset_m * 64, tile_offset_k * 32};
    tv::array<int, 2> block_offset_B{tile_offset_n * 64, tile_offset_k * 32};
    int thread_idx = threadIdx.x;
    InputIteratorA input_iter_A(
        params.itera_params_, params.problem, params.ptr_A,
        thread_idx,
        block_offset_A);
    InputIteratorB input_iter_B(
        params.iterb_params_, params.problem, params.ptr_B,
        thread_idx,
        block_offset_B);
    int warp_idx = __shfl_sync(0xffffffff, threadIdx.x / 32, 0);
    int lane_idx = threadIdx.x % 32;
    int warp_mn =
        warp_idx % (2 * 2);
    int warp_idx_k =
        warp_idx / (2 * 2);
    int warp_m = warp_mn % 2;
    int warp_n = warp_mn / 2;
    uint32_t kmask = 0;
    tv::array<uint32_t, 2> masks;
    masks.clear();
    TV_PRAGMA_UNROLL
    for (int i = 0; i < 2; ++i){
        if (tile_offset_m * 64 + i * 32 + lane_idx < params.m){
            masks[i] = params.mask_ptr[tile_offset_m * 64 + i * 32 + lane_idx];
        }
    }
    TV_PRAGMA_UNROLL
    for (int i = 0; i < 2; ++i){
        kmask |= masks[i];
    }
    // perform a warp reduce to get block mask
    TV_PRAGMA_UNROLL
    for (int mask = 16; mask > 0; mask /= 2) {
        kmask |= __shfl_xor_sync(0xffffffff, kmask, mask, 32);
    }
    kmask &= params.mask_filter;
    if (params.mask_out_ptr != nullptr){
        params.mask_out_ptr[tile_offset_m] = kmask;
    }
    Mma mma(gemm_shared_mem, thread_idx, warp_idx_k, warp_m, warp_n, lane_idx);
    tv::array<float, 32, 0> accumulators;
    accumulators.clear();
    if (!kSplitKSerial || params.gemm_k_iterations > 0){
      if (kmask != 0){
          mma(params.gemm_k_iterations, accumulators, input_iter_A, input_iter_B, accumulators, kmask, params.problem.kernel_volume);
      }
    }
    // // C = alpha * A@B + beta * D, D can be C
    OutputOp output_op(params.alpha, params.beta, params.act_alpha, params.act_beta, params.act_type);
    tv::array<int, 2> block_offset_C{tile_offset_m * 64,
                                    tile_offset_n * 64};
    tv::array<int, 2> block_extent_C{params.m, params.n};
    OutIter out_iter_C(params.out_params_, params.ptr_C, block_extent_C,
                            block_offset_C,
                            thread_idx);
    ConstOutIter out_iter_source(params.out_params_source_, params.ptr_D, block_extent_C,
                        block_offset_C,
                        thread_idx);
    Output out(out_shared_mem, thread_idx, warp_idx_k, warp_m, warp_n, lane_idx);
    out.run(output_op, accumulators, out_iter_C, out_iter_source);
  #else
    tv::printf2_once("this arch isn't supported!");
    assert(0);
  #endif
}
} // namespace Ampere_f32f32f32f32f32tnt_m64n64k32m32n32k32A1T1688_300_C301LLL_SK
} // namespace main
} // namespace conv
} // namespace cumm
} // namespace spconvlib