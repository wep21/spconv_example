#include "hip/hip_runtime.h"
#include <spconvlib/cumm/conv/main/Turing_s8s8f32s32f32tnt_m64n64k32m32n32k32A1T8816_200_C301LLL_SKD_S8/ConvKernel.h>
namespace spconvlib {
namespace cumm {
namespace conv {
namespace main {
namespace Turing_s8s8f32s32f32tnt_m64n64k32m32n32k32A1T8816_200_C301LLL_SKD_S8 {
using TensorViewNVRTCKernel = spconvlib::cumm::common::TensorViewNVRTCKernel;
using RowMajor = spconvlib::cumm::gemm::layout::RowMajor;
using ColumnMajor = spconvlib::cumm::gemm::layout::ColumnMajor;
using GemmBasicKernel = spconvlib::cumm::common::GemmBasicKernel;
using GemmBasic = spconvlib::cumm::common::GemmBasic;
using GemmKernelFlags = spconvlib::cumm::common::GemmKernelFlags;
using InputIteratorA = spconvlib::cumm::conv::main::Turing_s8s8f32s32f32tnt_m64n64k32m32n32k32A1T8816_200_C301LLL_SKD_S8::inpitera::ForwardDgradSparseIOIterator;
using InputIteratorB = spconvlib::cumm::conv::main::Turing_s8s8f32s32f32tnt_m64n64k32m32n32k32A1T8816_200_C301LLL_SKD_S8::inpiterb::WeightIteratorDP4A;
using LayoutA = spconvlib::cumm::conv::main::cpTuring_s8s8f32s32f32tnt_m64n64k32m32n32k32A1T8816_200_C301LLL_SKD_S8::la::TensorGeneric;
using LayoutB = spconvlib::cumm::conv::main::cpTuring_s8s8f32s32f32tnt_m64n64k32m32n32k32A1T8816_200_C301LLL_SKD_S8::lb::TensorGeneric;
using LayoutC = spconvlib::cumm::conv::main::cpTuring_s8s8f32s32f32tnt_m64n64k32m32n32k32A1T8816_200_C301LLL_SKD_S8::lc::TensorGeneric;
using BlockMmaStorage = spconvlib::cumm::conv::main::Turing_s8s8f32s32f32tnt_m64n64k32m32n32k32A1T8816_200_C301LLL_SKD_S8::gemm_smem_storage::BlockMmaStorage;
using OutputSmemStorage = spconvlib::cumm::conv::main::Turing_s8s8f32s32f32tnt_m64n64k32m32n32k32A1T8816_200_C301LLL_SKD_S8::out_smem_storage::OutputSmemStorage;
using ConvParams = spconvlib::cumm::conv::main::cpTuring_s8s8f32s32f32tnt_m64n64k32m32n32k32A1T8816_200_C301LLL_SKD_S8::ConvParams;
using ConvProblem = spconvlib::cumm::conv::main::cpTuring_s8s8f32s32f32tnt_m64n64k32m32n32k32A1T8816_200_C301LLL_SKD_S8::cp::ConvProblem;
using OutIter = spconvlib::cumm::conv::main::Turing_s8s8f32s32f32tnt_m64n64k32m32n32k32A1T8816_200_C301LLL_SKD_S8::out_iter::OutIterator;
using ConstOutIter = spconvlib::cumm::conv::main::Turing_s8s8f32s32f32tnt_m64n64k32m32n32k32A1T8816_200_C301LLL_SKD_S8::out_iter_const::OutIterator;
using ConstScaleOutIter = spconvlib::cumm::conv::main::Turing_s8s8f32s32f32tnt_m64n64k32m32n32k32A1T8816_200_C301LLL_SKD_S8::scale_out_iter_const::OutIterator;
using OutputOp = spconvlib::cumm::conv::main::Turing_s8s8f32s32f32tnt_m64n64k32m32n32k32A1T8816_200_C301LLL_SKD_S8::out_op::Int8Inference;
using Mma = spconvlib::cumm::conv::main::Turing_s8s8f32s32f32tnt_m64n64k32m32n32k32A1T8816_200_C301LLL_SKD_S8::mma::Mma;
using MaskIGemmIteratorDynamic = spconvlib::cumm::conv::main::Turing_s8s8f32s32f32tnt_m64n64k32m32n32k32A1T8816_200_C301LLL_SKD_S8::mma_miterd::MaskIGemmIteratorMaskLoaderDynamic;
using Output = spconvlib::cumm::conv::main::Turing_s8s8f32s32f32tnt_m64n64k32m32n32k32A1T8816_200_C301LLL_SKD_S8::output::Output;
__global__ void conv_kernel(ConvParams params)   {
  
  #if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 750))
    constexpr bool kSplitKSerial = false;
    extern __shared__ uint8_t SharedStorage[];
    auto gemm_shared_mem =
        reinterpret_cast<BlockMmaStorage *>(SharedStorage);
    auto out_shared_mem =
        reinterpret_cast<OutputSmemStorage *>(SharedStorage);
    int tile_offset_m = blockIdx.x;
    int tile_offset_n = blockIdx.y;
    int tile_offset_k = blockIdx.z;
    if (tile_offset_m >= params.grid_dims.x ||
        tile_offset_n >= params.grid_dims.y) {
        return;
    }
    tv::array<int, 2> block_offset_A{tile_offset_m * 64, tile_offset_k * 32};
    tv::array<int, 2> block_offset_B{tile_offset_n * 64, tile_offset_k * 32};
    int thread_idx = threadIdx.x;
    InputIteratorA input_iter_A(
        params.itera_params_, params.problem, params.ptr_A,
        thread_idx,
        block_offset_A);
    InputIteratorB input_iter_B(
        params.iterb_params_, params.problem, params.ptr_B,
        thread_idx,
        block_offset_B);
    int warp_idx = __shfl_sync(0xffffffff, threadIdx.x / 32, 0);
    int lane_idx = threadIdx.x % 32;
    int warp_mn =
        warp_idx % (2 * 2);
    int warp_idx_k =
        warp_idx / (2 * 2);
    int warp_m = warp_mn % 2;
    int warp_n = warp_mn / 2;
    int mask_int_count = tv::div_up(params.problem.kernel_volume, 32);
    MaskIGemmIteratorDynamic MaskLoader(params.mask_ptr, 
                                        params.mask_out_ptr, 
                                        mask_int_count,
                                        tile_offset_m, params.gemm_k_iterations,
                                        params.problem.kernel_volume, params.mask_filter,
                                        false, 
                                        lane_idx, params.m);
    Mma mma(gemm_shared_mem, thread_idx, warp_idx_k, warp_m, warp_n, lane_idx);
    tv::array<int32_t, 32, 0> accumulators;
    accumulators.clear();
    if (!kSplitKSerial || params.gemm_k_iterations > 0){
      if (!MaskLoader.empty()){
          mma(params.gemm_k_iterations, accumulators, input_iter_A, input_iter_B, accumulators, MaskLoader, params.problem.kernel_volume);
      }
    }
    // // C = alpha * A@B + beta * D, D can be C
    OutputOp output_op(params.alpha, params.beta, params.act_alpha, params.act_beta, params.act_type);
    tv::array<int, 2> block_offset_C{tile_offset_m * 64,
                                    tile_offset_n * 64};
    tv::array<int, 2> block_extent_C{params.m, params.n};
    OutIter out_iter_C(params.out_params_, params.ptr_C, params.ptr_D, block_extent_C,
                            block_offset_C,
                            thread_idx);
    ConstScaleOutIter out_iter_bias(params.out_params_scalebias_, params.bias_pointer, block_extent_C,
                            block_offset_C,
                            thread_idx);
    ConstScaleOutIter out_iter_scale(params.out_params_scalebias_, params.scale_pointer, block_extent_C,
                            block_offset_C,
                            thread_idx);
    Output out(out_shared_mem, thread_idx, warp_idx_k, warp_m, warp_n, lane_idx);
    out.run(output_op, accumulators, out_iter_C, out_iter_bias, out_iter_scale);
  #else
    tv::printf2_once("this arch isn't supported!");
    assert(0);
  #endif
}
} // namespace Turing_s8s8f32s32f32tnt_m64n64k32m32n32k32A1T8816_200_C301LLL_SKD_S8
} // namespace main
} // namespace conv
} // namespace cumm
} // namespace spconvlib