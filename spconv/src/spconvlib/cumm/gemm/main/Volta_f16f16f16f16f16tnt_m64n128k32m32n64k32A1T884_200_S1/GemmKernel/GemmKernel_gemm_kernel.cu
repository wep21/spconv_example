#include "hip/hip_runtime.h"
#include <spconvlib/cumm/gemm/main/Volta_f16f16f16f16f16tnt_m64n128k32m32n64k32A1T884_200_S1/GemmKernel.h>
namespace spconvlib {
namespace cumm {
namespace gemm {
namespace main {
namespace Volta_f16f16f16f16f16tnt_m64n128k32m32n64k32A1T884_200_S1 {
using TensorViewNVRTCKernel = spconvlib::cumm::common::TensorViewNVRTCKernel;
using RowMajor = spconvlib::cumm::gemm::layout::RowMajor;
using ColumnMajor = spconvlib::cumm::gemm::layout::ColumnMajor;
using GemmBasicKernel = spconvlib::cumm::common::GemmBasicKernel;
using GemmUtilsCPU = spconvlib::cumm::gemm::utils::GemmUtilsCPU;
using GemmKernelFlags = spconvlib::cumm::common::GemmKernelFlags;
using GemmUtils = spconvlib::cumm::gemm::main::Volta_f16f16f16f16f16tnt_m64n128k32m32n64k32A1T884_200_S1::gemmutils::GemmUtils;
using InputIteratorA = spconvlib::cumm::gemm::main::Volta_f16f16f16f16f16tnt_m64n128k32m32n64k32A1T884_200_S1::inpitera::MaskTileIterator;
using InputIteratorB = spconvlib::cumm::gemm::main::Volta_f16f16f16f16f16tnt_m64n128k32m32n64k32A1T884_200_S1::inpiterb::MaskTileIterator;
using BlockMmaStorage = spconvlib::cumm::gemm::main::Volta_f16f16f16f16f16tnt_m64n128k32m32n64k32A1T884_200_S1::gemm_smem_storage::BlockMmaStorage;
using OutputSmemStorage = spconvlib::cumm::gemm::main::Volta_f16f16f16f16f16tnt_m64n128k32m32n64k32A1T884_200_S1::out_smem_storage::OutputSmemStorage;
using GemmParams = spconvlib::cumm::gemm::main::gpVolta_f16f16f16f16f16tnt_m64n128k32m32n64k32A1T884_200_S1::GemmParams;
using OutIter = spconvlib::cumm::gemm::main::Volta_f16f16f16f16f16tnt_m64n128k32m32n64k32A1T884_200_S1::out_iter::OutIterator;
using ConstOutIter = spconvlib::cumm::gemm::main::Volta_f16f16f16f16f16tnt_m64n128k32m32n64k32A1T884_200_S1::out_iter_const::OutIterator;
using OutputOp = spconvlib::cumm::gemm::main::Volta_f16f16f16f16f16tnt_m64n128k32m32n64k32A1T884_200_S1::out_op::LinearCombination;
using Mma = spconvlib::cumm::gemm::main::Volta_f16f16f16f16f16tnt_m64n128k32m32n64k32A1T884_200_S1::mma::Mma;
using Output = spconvlib::cumm::gemm::main::Volta_f16f16f16f16f16tnt_m64n128k32m32n64k32A1T884_200_S1::output::Output;
__global__ void gemm_kernel(GemmParams params)   {
  
  #if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 700))
    // tv::printf2_once("?????222", params.grid_dims.x, params.grid_dims.y);
    constexpr bool kSplitKSerial = false;
    extern __shared__ uint8_t SharedStorage[];
    auto gemm_shared_mem =
        reinterpret_cast<BlockMmaStorage *>(SharedStorage);
    auto out_shared_mem =
        reinterpret_cast<OutputSmemStorage *>(SharedStorage);
    int tile_offset_m = blockIdx.x;
    int tile_offset_n = blockIdx.y;
    int tile_offset_k = blockIdx.z;
    if (tile_offset_m >= params.grid_dims.x || tile_offset_n >= params.grid_dims.y){
      return;
    }
    tv::array<int, 2> block_offset_A{tile_offset_m * 64,
                                    tile_offset_k * params.gemm_k_size_per_split};
    tv::array<int, 2> block_offset_B{tile_offset_k * params.gemm_k_size_per_split,
                                    tile_offset_n * 128};
    // Gemm::InputIteratorA::Params params_A(params.k);
    // Gemm::InputIteratorB::Params params_B(params.n);
    // refine gemm iteration for split-k
    auto problem_size_k = GemmUtils::get_gemm_k_bound(params.k, params.gemm_k_size_per_split, tile_offset_k);
    auto gemm_k_iterations = GemmUtils::get_gemm_iterations(problem_size_k, params.gemm_k_size_per_split, tile_offset_k);
    // int problem_size_k = min(params.k, (tile_offset_k + 1) * params.gemm_k_size_per_split);
    // int gemm_k_iterations =
    //     tv::div_up(problem_size_k - block_offset_A[1], 32);
    int thread_idx = threadIdx.x;
    InputIteratorA input_iter_A(
        params.itera_params_, params.ptr_A,
        tv::array<int, 2>{params.m, problem_size_k},
        thread_idx,
        block_offset_A);
    InputIteratorB input_iter_B(
        params.iterb_params_, params.ptr_B,
        tv::array<int, 2>{params.n, problem_size_k},
        thread_idx,
        tv::array<int, 2>{block_offset_B[1], block_offset_B[0]});
    int warp_idx = __shfl_sync(0xffffffff, threadIdx.x / 32, 0);
    int lane_idx = threadIdx.x % 32;
    int warp_mn =
        warp_idx % (2 * 2);
    int warp_idx_k =
        warp_idx / (2 * 2);
    int warp_m = warp_mn % 2;
    int warp_n = warp_mn / 2;
    Mma mma(gemm_shared_mem, thread_idx, warp_idx_k, warp_m, warp_n, lane_idx);
    tv::array<tv::half_t, 64, 0> accumulators;
    accumulators.clear();
    if (!kSplitKSerial || gemm_k_iterations > 0){
      mma(gemm_k_iterations, accumulators, input_iter_A, input_iter_B, accumulators);
    }
    // tv::printf2_once("HERE 0", threadIdx.x, blockIdx.x, blockIdx.y, blockIdx.z);
    // // C = alpha * A@B + beta * D, D can be C
    OutputOp output_op(params.alpha, params.beta, params.act_alpha, params.act_beta, params.act_type);
    tv::array<int, 2> block_offset_C{tile_offset_m * 64,
                                    tile_offset_n * 128};
    OutIter out_iter_C(params.out_params_, params.ptr_C, {params.m, params.n},
                            {block_offset_C[0], block_offset_C[1]},
                            thread_idx);
    ConstOutIter out_iter_D(params.out_params_scalebias_, params.ptr_D, {params.m, params.n},
                        {block_offset_C[0], block_offset_C[1]},
                        thread_idx);
    Output out(out_shared_mem, thread_idx, warp_idx_k, warp_m, warp_n, lane_idx);
    out.run(output_op, accumulators, out_iter_C, out_iter_D);
  #else
    tv::printf2_once("this arch isn't supported!");
    assert(0);
  #endif
}
} // namespace Volta_f16f16f16f16f16tnt_m64n128k32m32n64k32A1T884_200_S1
} // namespace main
} // namespace gemm
} // namespace cumm
} // namespace spconvlib