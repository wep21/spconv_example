#include "hip/hip_runtime.h"
#include <spconvlib/cumm/gemm/main/GemmMainUnitTest.h>
#include <spconvlib/cumm/common/TensorViewKernel.h>
#include <spconvlib/cumm/gemm/main/gpSimt_f16f16f16f32f32ttt_m128n128k8m32n64k8A1_200_S1/GemmParams.h>
#include <spconvlib/cumm/gemm/main/Simt_f16f16f16f32f32ttt_m128n128k8m32n64k8A1_200_S1/GemmKernel.h>
#include <spconvlib/cumm/gemm/main/gpSimt_f16f16f16f32f32tnt_m128n128k8m32n64k8A1_200_S1/GemmParams.h>
#include <spconvlib/cumm/gemm/main/Simt_f16f16f16f32f32tnt_m128n128k8m32n64k8A1_200_S1/GemmKernel.h>
#include <spconvlib/cumm/gemm/main/gpSimt_f16f16f16f32f32ttt_m32n64k32m32n32k8A1_200_S1/GemmParams.h>
#include <spconvlib/cumm/gemm/main/Simt_f16f16f16f32f32ttt_m32n64k32m32n32k8A1_200_S1/GemmKernel.h>
#include <spconvlib/cumm/gemm/main/gpSimt_f16f16f16f32f32tnt_m32n64k32m32n32k8A1_200_S1/GemmParams.h>
#include <spconvlib/cumm/gemm/main/Simt_f16f16f16f32f32tnt_m32n64k32m32n32k8A1_200_S1/GemmKernel.h>
#include <spconvlib/cumm/gemm/main/gpSimt_f16f16f16f32f32ttt_m32n32k32m32n32k8A1_200_S1/GemmParams.h>
#include <spconvlib/cumm/gemm/main/Simt_f16f16f16f32f32ttt_m32n32k32m32n32k8A1_200_S1/GemmKernel.h>
#include <spconvlib/cumm/gemm/main/gpSimt_f16f16f16f32f32tnt_m32n32k32m32n32k8A1_200_S1/GemmParams.h>
#include <spconvlib/cumm/gemm/main/Simt_f16f16f16f32f32tnt_m32n32k32m32n32k8A1_200_S1/GemmKernel.h>
#include <spconvlib/cumm/gemm/main/gpSimt_f16f16f16f32f32ttt_m64n128k16m32n64k8A1_200_S1/GemmParams.h>
#include <spconvlib/cumm/gemm/main/Simt_f16f16f16f32f32ttt_m64n128k16m32n64k8A1_200_S1/GemmKernel.h>
#include <spconvlib/cumm/gemm/main/gpSimt_f16f16f16f32f32tnt_m64n128k16m32n64k8A1_200_S1/GemmParams.h>
#include <spconvlib/cumm/gemm/main/Simt_f16f16f16f32f32tnt_m64n128k16m32n64k8A1_200_S1/GemmKernel.h>
#include <spconvlib/cumm/gemm/main/gpSimt_f16f16f16f32f32ttt_m64n64k8m32n32k8A1_200_S1/GemmParams.h>
#include <spconvlib/cumm/gemm/main/Simt_f16f16f16f32f32ttt_m64n64k8m32n32k8A1_200_S1/GemmKernel.h>
#include <spconvlib/cumm/gemm/main/gpSimt_f16f16f16f32f32tnt_m64n64k8m32n32k8A1_200_S1/GemmParams.h>
#include <spconvlib/cumm/gemm/main/Simt_f16f16f16f32f32tnt_m64n64k8m32n32k8A1_200_S1/GemmKernel.h>
namespace spconvlib {
namespace cumm {
namespace gemm {
namespace main {
using TensorView = spconvlib::cumm::common::TensorView;
using GemmBasic = spconvlib::cumm::common::GemmBasic;
using GemmBasicHost = spconvlib::cumm::common::GemmBasicHost;
using TensorViewKernel = spconvlib::cumm::common::TensorViewKernel;
using GemmParamsSimt_f16f16f16f32f32ttt_m128n128k8m32n64k8A1_200_S1 = spconvlib::cumm::gemm::main::gpSimt_f16f16f16f32f32ttt_m128n128k8m32n64k8A1_200_S1::GemmParams;
using GemmSimt_f16f16f16f32f32ttt_m128n128k8m32n64k8A1_200_S1 = spconvlib::cumm::gemm::main::Simt_f16f16f16f32f32ttt_m128n128k8m32n64k8A1_200_S1::GemmKernel;
using GemmParamsSimt_f16f16f16f32f32tnt_m128n128k8m32n64k8A1_200_S1 = spconvlib::cumm::gemm::main::gpSimt_f16f16f16f32f32tnt_m128n128k8m32n64k8A1_200_S1::GemmParams;
using GemmSimt_f16f16f16f32f32tnt_m128n128k8m32n64k8A1_200_S1 = spconvlib::cumm::gemm::main::Simt_f16f16f16f32f32tnt_m128n128k8m32n64k8A1_200_S1::GemmKernel;
using GemmParamsSimt_f16f16f16f32f32ttt_m32n64k32m32n32k8A1_200_S1 = spconvlib::cumm::gemm::main::gpSimt_f16f16f16f32f32ttt_m32n64k32m32n32k8A1_200_S1::GemmParams;
using GemmSimt_f16f16f16f32f32ttt_m32n64k32m32n32k8A1_200_S1 = spconvlib::cumm::gemm::main::Simt_f16f16f16f32f32ttt_m32n64k32m32n32k8A1_200_S1::GemmKernel;
using GemmParamsSimt_f16f16f16f32f32tnt_m32n64k32m32n32k8A1_200_S1 = spconvlib::cumm::gemm::main::gpSimt_f16f16f16f32f32tnt_m32n64k32m32n32k8A1_200_S1::GemmParams;
using GemmSimt_f16f16f16f32f32tnt_m32n64k32m32n32k8A1_200_S1 = spconvlib::cumm::gemm::main::Simt_f16f16f16f32f32tnt_m32n64k32m32n32k8A1_200_S1::GemmKernel;
using GemmParamsSimt_f16f16f16f32f32ttt_m32n32k32m32n32k8A1_200_S1 = spconvlib::cumm::gemm::main::gpSimt_f16f16f16f32f32ttt_m32n32k32m32n32k8A1_200_S1::GemmParams;
using GemmSimt_f16f16f16f32f32ttt_m32n32k32m32n32k8A1_200_S1 = spconvlib::cumm::gemm::main::Simt_f16f16f16f32f32ttt_m32n32k32m32n32k8A1_200_S1::GemmKernel;
using GemmParamsSimt_f16f16f16f32f32tnt_m32n32k32m32n32k8A1_200_S1 = spconvlib::cumm::gemm::main::gpSimt_f16f16f16f32f32tnt_m32n32k32m32n32k8A1_200_S1::GemmParams;
using GemmSimt_f16f16f16f32f32tnt_m32n32k32m32n32k8A1_200_S1 = spconvlib::cumm::gemm::main::Simt_f16f16f16f32f32tnt_m32n32k32m32n32k8A1_200_S1::GemmKernel;
using GemmParamsSimt_f16f16f16f32f32ttt_m64n128k16m32n64k8A1_200_S1 = spconvlib::cumm::gemm::main::gpSimt_f16f16f16f32f32ttt_m64n128k16m32n64k8A1_200_S1::GemmParams;
using GemmSimt_f16f16f16f32f32ttt_m64n128k16m32n64k8A1_200_S1 = spconvlib::cumm::gemm::main::Simt_f16f16f16f32f32ttt_m64n128k16m32n64k8A1_200_S1::GemmKernel;
using GemmParamsSimt_f16f16f16f32f32tnt_m64n128k16m32n64k8A1_200_S1 = spconvlib::cumm::gemm::main::gpSimt_f16f16f16f32f32tnt_m64n128k16m32n64k8A1_200_S1::GemmParams;
using GemmSimt_f16f16f16f32f32tnt_m64n128k16m32n64k8A1_200_S1 = spconvlib::cumm::gemm::main::Simt_f16f16f16f32f32tnt_m64n128k16m32n64k8A1_200_S1::GemmKernel;
using GemmParamsSimt_f16f16f16f32f32ttt_m64n64k8m32n32k8A1_200_S1 = spconvlib::cumm::gemm::main::gpSimt_f16f16f16f32f32ttt_m64n64k8m32n32k8A1_200_S1::GemmParams;
using GemmSimt_f16f16f16f32f32ttt_m64n64k8m32n32k8A1_200_S1 = spconvlib::cumm::gemm::main::Simt_f16f16f16f32f32ttt_m64n64k8m32n32k8A1_200_S1::GemmKernel;
using GemmParamsSimt_f16f16f16f32f32tnt_m64n64k8m32n32k8A1_200_S1 = spconvlib::cumm::gemm::main::gpSimt_f16f16f16f32f32tnt_m64n64k8m32n32k8A1_200_S1::GemmParams;
using GemmSimt_f16f16f16f32f32tnt_m64n64k8m32n32k8A1_200_S1 = spconvlib::cumm::gemm::main::Simt_f16f16f16f32f32tnt_m64n64k8m32n32k8A1_200_S1::GemmKernel;
void GemmMainUnitTest::matmul_split_Simt_f16f16f16_1(tv::gemm::GemmParams params)   {
  
  params.check_valid();
  auto& algo_desp = params.algo_desp;
  bool found = false;
  auto dacc = tv::DType(algo_desp.dacc);
  auto dcomp = tv::DType(algo_desp.dcomp);
  auto a = params.a;
  auto b = params.b;
  auto c = params.c;
  auto d = params.d;
  if (d.empty()){
      d = c; // TODO fix this
  }
  auto ta = algo_desp.trans_a();
  auto tb = algo_desp.trans_b();
  auto tc = algo_desp.trans_c();
  tv::check_shape(a, {-1, -1});
  tv::check_shape(b, {-1, -1});
  tv::check_shape(c, {-1, -1});
  tv::check_eq_device(a, b, c);
  tv::Tensor a_ten = a;
  tv::Tensor b_ten = b;
  tv::Tensor c_ten = c;
  tv::Tensor d_ten = d;
  auto trans_a = ta;
  auto trans_b = tb;
  auto trans_c = tc;
  if (tc) {
      trans_a = !trans_a;
      trans_b = !trans_b;
      std::swap(trans_a, trans_b);
      std::swap(a_ten, b_ten);
  }
  int split_k_slices = params.split_k_slices;
  auto workspace = params.workspace;
  auto a_inds = params.a_inds;
  auto c_inds = params.c_inds;
  auto b_inds = params.b_inds;
  auto& evtimer = params.timer;
  if (!(algo_desp.split_k_serial() || algo_desp.split_k_parallel()) && split_k_slices > 1){
      TV_ASSERT_RT_ERR("algo don't support splitk but you provide split_k_slices > 1.", split_k_slices);
  }
  int m, n, k, k2;
  constexpr int int_max = std::numeric_limits<int32_t>::max();
  if (algo_desp.shuffle_type == tv::gemm::ShuffleStrideType::kShuffleAC){
      TV_ASSERT_RT_ERR(!trans_a, "a of shuffle AB must be row major");
      if (!a_inds.empty()){
          m = a_inds.dim(0);
      }else{
          m = a.dim(0);
      }
      TV_ASSERT_RT_ERR(int64_t(a.dim(0)) * int64_t(a.dim(1)) * tv::bit_size(algo_desp.dtype_a) / 8 < int_max, 
          "your data exceed int32 range. this will be fixed in cumm + nvrtc (spconv 2.2/2.3).");
      k = a_ten.dim(int(!trans_a));
      k2 = b_ten.dim(int(trans_b));
      n = b_ten.dim(int(!trans_b) );
      if (trans_c){
          tv::check_shape(c_ten, {-1, m});
      }else{
          tv::check_shape(c_ten, {-1, n});
      }
  }else if (algo_desp.shuffle_type == tv::gemm::ShuffleStrideType::kShuffleAB){
      TV_ASSERT_RT_ERR(trans_a && !trans_b, "shuffle AB must be nt, i.e. backward weight");
      m = a_ten.dim(int(trans_a));
      k = a_inds.dim(0);
      k2 = b_inds.dim(0);
      n = b_ten.dim(int(!trans_b) );
      TV_ASSERT_RT_ERR(int64_t(a.dim(0)) * int64_t(a.dim(1)) * tv::bit_size(algo_desp.dtype_a)/ 8 < int_max, 
          "your data exceed int32 range. this will be fixed in cumm + nvrtc (spconv 2.2/2.3).");
      TV_ASSERT_RT_ERR(int64_t(b.dim(0)) * int64_t(b.dim(1)) * tv::bit_size(algo_desp.dtype_b) / 8 < int_max, 
          "your data exceed int32 range. this will be fixed in cumm + nvrtc (spconv 2.2/2.3).");
      if (trans_c){
          tv::check_shape(c_ten, {n, m});
      }else{
          tv::check_shape(c_ten, {m, n});
      }
  }else{
      m = a_ten.dim(int(trans_a));
      k = a_ten.dim(int(!trans_a));
      k2 = b_ten.dim(int(trans_b));
      n = b_ten.dim(int(!trans_b) );
      if (trans_c){
          tv::check_shape(c_ten, {n, m});
      }else{
          tv::check_shape(c_ten, {m, n});
      }
  }
  TV_ASSERT_INVALID_ARG(algo_desp.supported(m, n, k), "this m, n, k isn't supported due to misaligned contiguous dim.")
  TV_ASSERT_INVALID_ARG(k == k2, "error");
  if (d.ndim() == 1){
      TV_ASSERT_RT_ERR(d.dim(0) == n, "d must be a valid bias");
  }
  int workspace_size = algo_desp.query_workspace_size(m, n, k, split_k_slices);
  auto ctx = tv::Context();
  ctx.set_cuda_stream(reinterpret_cast<hipStream_t>(params.stream));
  if (workspace_size > 0){
      if (!workspace.empty()){
          workspace.zero_(ctx);
          TV_ASSERT_RT_ERR(workspace.nbytes() >= workspace_size, 
              "workspace at least", workspace_size, "bytes.");
      }else{
          workspace = tv::empty({workspace_size}, tv::uint8, 0);
          workspace.zero_(ctx);
      }
  }
  void* workspace_ptr = nullptr;
  if (!workspace.empty()){
      workspace_ptr = workspace.raw_data();
  }
  auto& nvrtc_params = params.nvrtc_params;
  if (nvrtc_params.cumodule){
      TV_ASSERT_RT_ERR(nvrtc_params.kernel_name != "", "you must provide name of your kernel");
      tv::gemm::GemmNVRTCParams kernel_params;
      if (algo_desp.shuffle_type == tv::gemm::ShuffleStrideType::kShuffleAC){
          const int* a_ptr = nullptr;
          if (!a_inds.empty()){
              a_ptr = a_inds.data_ptr<const int>();
          }
          TV_ASSERT_RT_ERR(!c_inds.empty(), "c must not empty");
          auto indice_ptr = c_inds.data_ptr<const int>();
          kernel_params = tv::gemm::GemmNVRTCParams{m, n, k, a_ten.const_raw_data(),  b_ten.const_raw_data(),  
              c_ten.raw_data(), d_ten.raw_data(), 
              a_ten.stride(0), b_ten.stride(0), c_ten.stride(0), d.ndim() == 1 ? 0 : d_ten.stride(0),
              a_ptr, indice_ptr, d.ndim() == 1 ? nullptr : indice_ptr, 
              float(params.alpha), float(params.beta), 
              float(params.act_alpha), float(params.act_beta), 
              static_cast<int>(params.act_type),
              split_k_slices, workspace_ptr};
      }else if (algo_desp.shuffle_type == tv::gemm::ShuffleStrideType::kShuffleAB){
          TV_ASSERT_RT_ERR(!a_inds.empty() && !b_inds.empty(), "error");
          kernel_params = tv::gemm::GemmNVRTCParams{m, n, k, a_ten.const_raw_data(),  b_ten.const_raw_data(),  
              c_ten.raw_data(), d_ten.raw_data(), 
              a_ten.stride(0), b_ten.stride(0), c_ten.stride(0), d.ndim() == 1 ? 0 : d_ten.stride(0),
              a_inds.data_ptr<const int>(), b_inds.data_ptr<const int>(), nullptr,
              float(params.alpha), float(params.beta), 
              float(params.act_alpha), float(params.act_beta), 
              static_cast<int>(params.act_type),
              split_k_slices, workspace_ptr};
      }else{
          kernel_params = tv::gemm::GemmNVRTCParams{m, n, k, a_ten.const_raw_data(),  b_ten.const_raw_data(),  
              c_ten.raw_data(), c_ten.raw_data(), 
              a_ten.stride(0), b_ten.stride(0), c_ten.stride(0), d.ndim() == 1 ? 0 : d_ten.stride(0),
              nullptr, nullptr, nullptr,
              float(params.alpha), float(params.beta), 
              float(params.act_alpha), float(params.act_beta), 
              static_cast<int>(params.act_type),
              split_k_slices, workspace_ptr};
      }
      std::string algo_name;
      if (evtimer.enable()){
          algo_name = algo_desp.__repr__();
      }
      auto grid_dims_arr = tv::gemm::get_logical_tile_count(m, n, k, algo_desp.tile_shape[0], algo_desp.tile_shape[1], split_k_slices);
      TV_ASSERT_RT_ERR(grid_dims_arr[0] != 0 && grid_dims_arr[1] != 0 && grid_dims_arr[2] != 0, "unexpected error",
          m, n, k, algo_desp.tile_shape[0], algo_desp.tile_shape[1], split_k_slices);
      dim3 grid_dims;
      grid_dims.x = grid_dims_arr[0];
      grid_dims.y = grid_dims_arr[1];
      grid_dims.z = grid_dims_arr[2];
      hipStream_t stream = reinterpret_cast<hipStream_t>(params.stream);
      auto kernel = nvrtc_params.cumodule->kernel(nvrtc_params.kernel_name);
      if (nvrtc_params.mode == 2){
          tv::CUDAKernelTimerGuard timerguard(algo_name, evtimer, stream);
          std::vector<void*> args{&kernel_params, &grid_dims, &params.stream};
          TV_CUDA_RESULT_CHECK(nvrtc_params.cumodule->cuDrvLaunchKernel(kernel, 1, 1, 1, 
              1, 1, 1, 0, stream, args.data(), 0));
      }else if (nvrtc_params.mode == 3){
          // use kernel-cpu-kernel
          auto init_kernel = nvrtc_params.cumodule->kernel(nvrtc_params.init_kernel_name);
          tv::Tensor temp_data = nvrtc_params.param_storage;
          if (nvrtc_params.param_storage.empty()){
              temp_data = tv::empty({nvrtc_params.param_size}, tv::uint8, 0);
          }else{
              TV_ASSERT_RT_ERR(temp_data.nbytes() >= nvrtc_params.param_size, "your params storage too small");
          }
          void* raw_data_ptr;
          void* temp_data_ptr = temp_data.raw_data();
          tv::Tensor temp_data_cpu = nvrtc_params.param_storage_cpu;
          {
              tv::CUDAKernelTimerGuard timerguard(algo_name + "/init", evtimer, stream);
              std::vector<void*> args{&kernel_params, &temp_data_ptr};
              TV_CUDA_RESULT_CHECK(nvrtc_params.cumodule->cuDrvLaunchKernel(init_kernel, 1, 1, 1, 32, 1, 1, 0, stream, args.data(), 0));
              if (nvrtc_params.param_storage_cpu.empty()){
                  temp_data_cpu = temp_data.cpu(ctx);
              }else{
                  temp_data_cpu.copy_(temp_data, ctx);
              }
              // we must sync here because following kernel launch requires cpu data.
              checkCudaErrors(hipStreamSynchronize(stream));
              raw_data_ptr = temp_data_cpu.raw_data();
          }
          {
              tv::CUDAKernelTimerGuard timerguard(algo_name, evtimer, stream);
              std::vector<void*> args{raw_data_ptr};
              // tv::ssprint(reinterpret_cast<tv::array<int, 4>*>(raw_data_ptr)[0]);
              // tv::ssprint(grid_dims.x, grid_dims.y, grid_dims.z, temp_data.size(), temp_data_cpu.size());
              TV_CUDA_RESULT_CHECK(nvrtc_params.cumodule->cuDrvLaunchKernel(kernel, grid_dims.x, grid_dims.y, grid_dims.z, 
                  nvrtc_params.num_threads, 1, 1, nvrtc_params.smem_size, stream, args.data(), 0));
          }
      }else if (nvrtc_params.mode == 1){
          tv::CUDAKernelTimerGuard timerguard(algo_name, evtimer, stream);
          std::vector<void*> args{&kernel_params};
          TV_CUDA_RESULT_CHECK(nvrtc_params.cumodule->cuDrvLaunchKernel(kernel, grid_dims.x, grid_dims.y, grid_dims.z, 
              nvrtc_params.num_threads, 1, 1, nvrtc_params.smem_size, stream, args.data(), 0));
      }else if (nvrtc_params.mode == 4){
          auto init_kernel = nvrtc_params.cumodule->kernel(nvrtc_params.init_kernel_name);
          tv::Tensor temp_data = nvrtc_params.param_storage;
          if (nvrtc_params.param_storage.empty()){
              temp_data = tv::empty({nvrtc_params.param_size}, tv::uint8, 0);
          }else{
              TV_ASSERT_RT_ERR(temp_data.nbytes() >= nvrtc_params.param_size, "your params storage too small");
          }
          void* temp_data_ptr = temp_data.raw_data();
          {
              tv::CUDAKernelTimerGuard timerguard(algo_name + "/init", evtimer, stream);
              std::vector<void*> args{&kernel_params, &temp_data_ptr};
              TV_CUDA_RESULT_CHECK(nvrtc_params.cumodule->cuDrvLaunchKernel(init_kernel, 1, 1, 1, 32, 1, 1, 0, stream, args.data(), 0));
          }
          {
              tv::CUDAKernelTimerGuard timerguard(algo_name, evtimer, stream);
              auto ptr = nvrtc_params.cumodule->get_global_ptr(nvrtc_params.constant_name);
              auto constant_ten = tv::from_blob(ptr, {nvrtc_params.param_size}, tv::uint8, 0);
              constant_ten.copy_(temp_data, ctx);
              std::vector<void*> args{};
              TV_CUDA_RESULT_CHECK(nvrtc_params.cumodule->cuDrvLaunchKernel(kernel, grid_dims.x, grid_dims.y, grid_dims.z, 
                  nvrtc_params.num_threads, 1, 1, nvrtc_params.smem_size, stream, args.data(), 0));
          }
      }else{
          TV_THROW_RT_ERR("not implemented");
      }
      TV_CHECK_CUDA_ERR_V2(algo_desp.__repr__(), "error with params", a.shape(), b.shape(), c.shape());
      return;
  }
  if (algo_desp.trans_a() == false && algo_desp.trans_b() == false && algo_desp.trans_c() == false){
    if (algo_desp.tile_shape == std::array<int, 3>{128, 128, 8}){
      if (algo_desp.warp_tile_shape == std::array<int, 3>{32, 64, 8}){
        if (algo_desp.num_stage == 2 && algo_desp.dacc == 0 && algo_desp.dcomp == 0){
          if ((params.split_k_slices == 1)){
            if (algo_desp.access_per_vector == 1){
              found = true;
              const int* a_ptr = nullptr;
              if (!a_inds.empty()){
                  a_ptr = a_inds.data_ptr<const int>();
              }
              TV_ASSERT_RT_ERR(!c_inds.empty(), "c must not empty");
              // tv::ssprint(d.ndim() == 1 ? 0 : d_ten.stride(0), (d.ndim() == 1 ? nullptr : c_inds.data_ptr<const int>()) == nullptr, "WTF");
              GemmParamsSimt_f16f16f16f32f32ttt_m128n128k8m32n64k8A1_200_S1 kernel_params(
                  m, n, k, a_ten.data_ptr<const tv::half_t>(), b_ten.data_ptr<const tv::half_t>(),
                  c_ten.data_ptr<tv::half_t>(), d_ten.data_ptr<tv::half_t>(), 
                  a_ten.stride(0), b_ten.stride(0), c_ten.stride(0), d.ndim() == 1 ? 0 : d_ten.stride(0), 
                  a_ptr, c_inds.data_ptr<const int>(), d.ndim() == 1 ? nullptr : c_inds.data_ptr<const int>(), 
                  float(params.alpha), float(params.beta), 
                  float(params.act_alpha), float(params.act_beta),
                  params.act_type,
                  split_k_slices);
              tv::cuda::Launch launcher(kernel_params.grid_dims, dim3(256),
                                          10304, reinterpret_cast<hipStream_t>(params.stream));
              hipError_t result;
              if (10304 >= (48 << 10)) {
                  result = hipFuncSetAttribute(reinterpret_cast<const void*>(Simt_f16f16f16f32f32ttt_m128n128k8m32n64k8A1_200_S1::gemm_kernel),
                                                  hipFuncAttributeMaxDynamicSharedMemorySize,
                                                  10304);
                  TV_ASSERT_RT_ERR(result == hipSuccess, "error");
                  result = hipFuncSetAttribute(reinterpret_cast<const void*>(
                      Simt_f16f16f16f32f32ttt_m128n128k8m32n64k8A1_200_S1::gemm_kernel),
                      hipFuncAttributePreferredSharedMemoryCarveout, 100);
                  TV_ASSERT_RT_ERR(result == hipSuccess, "error");
              }
              {
                  tv::CUDAKernelTimerGuard timerguard("Simt_f16f16f16f32f32ttt_m128n128k8m32n64k8A1_200_S1", evtimer, reinterpret_cast<hipStream_t>(params.stream));
                  launcher(Simt_f16f16f16f32f32ttt_m128n128k8m32n64k8A1_200_S1::gemm_kernel, kernel_params);
              }
              TV_CHECK_CUDA_ERR_V2("Simt_f16f16f16f32f32ttt_m128n128k8m32n64k8A1_200_S1", "error with params", a.shape(), b.shape(), c.shape());
              return;
            }
          }
        }
      }
    }
    if (algo_desp.tile_shape == std::array<int, 3>{32, 64, 32}){
      if (algo_desp.warp_tile_shape == std::array<int, 3>{32, 32, 8}){
        if (algo_desp.num_stage == 2 && algo_desp.dacc == 0 && algo_desp.dcomp == 0){
          if ((params.split_k_slices == 1)){
            if (algo_desp.access_per_vector == 1){
              found = true;
              const int* a_ptr = nullptr;
              if (!a_inds.empty()){
                  a_ptr = a_inds.data_ptr<const int>();
              }
              TV_ASSERT_RT_ERR(!c_inds.empty(), "c must not empty");
              // tv::ssprint(d.ndim() == 1 ? 0 : d_ten.stride(0), (d.ndim() == 1 ? nullptr : c_inds.data_ptr<const int>()) == nullptr, "WTF");
              GemmParamsSimt_f16f16f16f32f32ttt_m32n64k32m32n32k8A1_200_S1 kernel_params(
                  m, n, k, a_ten.data_ptr<const tv::half_t>(), b_ten.data_ptr<const tv::half_t>(),
                  c_ten.data_ptr<tv::half_t>(), d_ten.data_ptr<tv::half_t>(), 
                  a_ten.stride(0), b_ten.stride(0), c_ten.stride(0), d.ndim() == 1 ? 0 : d_ten.stride(0), 
                  a_ptr, c_inds.data_ptr<const int>(), d.ndim() == 1 ? nullptr : c_inds.data_ptr<const int>(), 
                  float(params.alpha), float(params.beta), 
                  float(params.act_alpha), float(params.act_beta),
                  params.act_type,
                  split_k_slices);
              tv::cuda::Launch launcher(kernel_params.grid_dims, dim3(256),
                                          12544, reinterpret_cast<hipStream_t>(params.stream));
              hipError_t result;
              if (12544 >= (48 << 10)) {
                  result = hipFuncSetAttribute(reinterpret_cast<const void*>(Simt_f16f16f16f32f32ttt_m32n64k32m32n32k8A1_200_S1::gemm_kernel),
                                                  hipFuncAttributeMaxDynamicSharedMemorySize,
                                                  12544);
                  TV_ASSERT_RT_ERR(result == hipSuccess, "error");
                  result = hipFuncSetAttribute(reinterpret_cast<const void*>(
                      Simt_f16f16f16f32f32ttt_m32n64k32m32n32k8A1_200_S1::gemm_kernel),
                      hipFuncAttributePreferredSharedMemoryCarveout, 100);
                  TV_ASSERT_RT_ERR(result == hipSuccess, "error");
              }
              {
                  tv::CUDAKernelTimerGuard timerguard("Simt_f16f16f16f32f32ttt_m32n64k32m32n32k8A1_200_S1", evtimer, reinterpret_cast<hipStream_t>(params.stream));
                  launcher(Simt_f16f16f16f32f32ttt_m32n64k32m32n32k8A1_200_S1::gemm_kernel, kernel_params);
              }
              TV_CHECK_CUDA_ERR_V2("Simt_f16f16f16f32f32ttt_m32n64k32m32n32k8A1_200_S1", "error with params", a.shape(), b.shape(), c.shape());
              return;
            }
          }
        }
      }
    }
    if (algo_desp.tile_shape == std::array<int, 3>{32, 32, 32}){
      if (algo_desp.warp_tile_shape == std::array<int, 3>{32, 32, 8}){
        if (algo_desp.num_stage == 2 && algo_desp.dacc == 0 && algo_desp.dcomp == 0){
          if ((params.split_k_slices == 1)){
            if (algo_desp.access_per_vector == 1){
              found = true;
              const int* a_ptr = nullptr;
              if (!a_inds.empty()){
                  a_ptr = a_inds.data_ptr<const int>();
              }
              TV_ASSERT_RT_ERR(!c_inds.empty(), "c must not empty");
              // tv::ssprint(d.ndim() == 1 ? 0 : d_ten.stride(0), (d.ndim() == 1 ? nullptr : c_inds.data_ptr<const int>()) == nullptr, "WTF");
              GemmParamsSimt_f16f16f16f32f32ttt_m32n32k32m32n32k8A1_200_S1 kernel_params(
                  m, n, k, a_ten.data_ptr<const tv::half_t>(), b_ten.data_ptr<const tv::half_t>(),
                  c_ten.data_ptr<tv::half_t>(), d_ten.data_ptr<tv::half_t>(), 
                  a_ten.stride(0), b_ten.stride(0), c_ten.stride(0), d.ndim() == 1 ? 0 : d_ten.stride(0), 
                  a_ptr, c_inds.data_ptr<const int>(), d.ndim() == 1 ? nullptr : c_inds.data_ptr<const int>(), 
                  float(params.alpha), float(params.beta), 
                  float(params.act_alpha), float(params.act_beta),
                  params.act_type,
                  split_k_slices);
              tv::cuda::Launch launcher(kernel_params.grid_dims, dim3(128),
                                          8448, reinterpret_cast<hipStream_t>(params.stream));
              hipError_t result;
              if (8448 >= (48 << 10)) {
                  result = hipFuncSetAttribute(reinterpret_cast<const void*>(Simt_f16f16f16f32f32ttt_m32n32k32m32n32k8A1_200_S1::gemm_kernel),
                                                  hipFuncAttributeMaxDynamicSharedMemorySize,
                                                  8448);
                  TV_ASSERT_RT_ERR(result == hipSuccess, "error");
                  result = hipFuncSetAttribute(reinterpret_cast<const void*>(
                      Simt_f16f16f16f32f32ttt_m32n32k32m32n32k8A1_200_S1::gemm_kernel),
                      hipFuncAttributePreferredSharedMemoryCarveout, 100);
                  TV_ASSERT_RT_ERR(result == hipSuccess, "error");
              }
              {
                  tv::CUDAKernelTimerGuard timerguard("Simt_f16f16f16f32f32ttt_m32n32k32m32n32k8A1_200_S1", evtimer, reinterpret_cast<hipStream_t>(params.stream));
                  launcher(Simt_f16f16f16f32f32ttt_m32n32k32m32n32k8A1_200_S1::gemm_kernel, kernel_params);
              }
              TV_CHECK_CUDA_ERR_V2("Simt_f16f16f16f32f32ttt_m32n32k32m32n32k8A1_200_S1", "error with params", a.shape(), b.shape(), c.shape());
              return;
            }
          }
        }
      }
    }
    if (algo_desp.tile_shape == std::array<int, 3>{64, 128, 16}){
      if (algo_desp.warp_tile_shape == std::array<int, 3>{32, 64, 8}){
        if (algo_desp.num_stage == 2 && algo_desp.dacc == 0 && algo_desp.dcomp == 0){
          if ((params.split_k_slices == 1)){
            if (algo_desp.access_per_vector == 1){
              found = true;
              const int* a_ptr = nullptr;
              if (!a_inds.empty()){
                  a_ptr = a_inds.data_ptr<const int>();
              }
              TV_ASSERT_RT_ERR(!c_inds.empty(), "c must not empty");
              // tv::ssprint(d.ndim() == 1 ? 0 : d_ten.stride(0), (d.ndim() == 1 ? nullptr : c_inds.data_ptr<const int>()) == nullptr, "WTF");
              GemmParamsSimt_f16f16f16f32f32ttt_m64n128k16m32n64k8A1_200_S1 kernel_params(
                  m, n, k, a_ten.data_ptr<const tv::half_t>(), b_ten.data_ptr<const tv::half_t>(),
                  c_ten.data_ptr<tv::half_t>(), d_ten.data_ptr<tv::half_t>(), 
                  a_ten.stride(0), b_ten.stride(0), c_ten.stride(0), d.ndim() == 1 ? 0 : d_ten.stride(0), 
                  a_ptr, c_inds.data_ptr<const int>(), d.ndim() == 1 ? nullptr : c_inds.data_ptr<const int>(), 
                  float(params.alpha), float(params.beta), 
                  float(params.act_alpha), float(params.act_beta),
                  params.act_type,
                  split_k_slices);
              tv::cuda::Launch launcher(kernel_params.grid_dims, dim3(256),
                                          12544, reinterpret_cast<hipStream_t>(params.stream));
              hipError_t result;
              if (12544 >= (48 << 10)) {
                  result = hipFuncSetAttribute(reinterpret_cast<const void*>(Simt_f16f16f16f32f32ttt_m64n128k16m32n64k8A1_200_S1::gemm_kernel),
                                                  hipFuncAttributeMaxDynamicSharedMemorySize,
                                                  12544);
                  TV_ASSERT_RT_ERR(result == hipSuccess, "error");
                  result = hipFuncSetAttribute(reinterpret_cast<const void*>(
                      Simt_f16f16f16f32f32ttt_m64n128k16m32n64k8A1_200_S1::gemm_kernel),
                      hipFuncAttributePreferredSharedMemoryCarveout, 100);
                  TV_ASSERT_RT_ERR(result == hipSuccess, "error");
              }
              {
                  tv::CUDAKernelTimerGuard timerguard("Simt_f16f16f16f32f32ttt_m64n128k16m32n64k8A1_200_S1", evtimer, reinterpret_cast<hipStream_t>(params.stream));
                  launcher(Simt_f16f16f16f32f32ttt_m64n128k16m32n64k8A1_200_S1::gemm_kernel, kernel_params);
              }
              TV_CHECK_CUDA_ERR_V2("Simt_f16f16f16f32f32ttt_m64n128k16m32n64k8A1_200_S1", "error with params", a.shape(), b.shape(), c.shape());
              return;
            }
          }
        }
      }
    }
    if (algo_desp.tile_shape == std::array<int, 3>{64, 64, 8}){
      if (algo_desp.warp_tile_shape == std::array<int, 3>{32, 32, 8}){
        if (algo_desp.num_stage == 2 && algo_desp.dacc == 0 && algo_desp.dcomp == 0){
          if ((params.split_k_slices == 1)){
            if (algo_desp.access_per_vector == 1){
              found = true;
              const int* a_ptr = nullptr;
              if (!a_inds.empty()){
                  a_ptr = a_inds.data_ptr<const int>();
              }
              TV_ASSERT_RT_ERR(!c_inds.empty(), "c must not empty");
              // tv::ssprint(d.ndim() == 1 ? 0 : d_ten.stride(0), (d.ndim() == 1 ? nullptr : c_inds.data_ptr<const int>()) == nullptr, "WTF");
              GemmParamsSimt_f16f16f16f32f32ttt_m64n64k8m32n32k8A1_200_S1 kernel_params(
                  m, n, k, a_ten.data_ptr<const tv::half_t>(), b_ten.data_ptr<const tv::half_t>(),
                  c_ten.data_ptr<tv::half_t>(), d_ten.data_ptr<tv::half_t>(), 
                  a_ten.stride(0), b_ten.stride(0), c_ten.stride(0), d.ndim() == 1 ? 0 : d_ten.stride(0), 
                  a_ptr, c_inds.data_ptr<const int>(), d.ndim() == 1 ? nullptr : c_inds.data_ptr<const int>(), 
                  float(params.alpha), float(params.beta), 
                  float(params.act_alpha), float(params.act_beta),
                  params.act_type,
                  split_k_slices);
              tv::cuda::Launch launcher(kernel_params.grid_dims, dim3(128),
                                          4352, reinterpret_cast<hipStream_t>(params.stream));
              hipError_t result;
              if (4352 >= (48 << 10)) {
                  result = hipFuncSetAttribute(reinterpret_cast<const void*>(Simt_f16f16f16f32f32ttt_m64n64k8m32n32k8A1_200_S1::gemm_kernel),
                                                  hipFuncAttributeMaxDynamicSharedMemorySize,
                                                  4352);
                  TV_ASSERT_RT_ERR(result == hipSuccess, "error");
                  result = hipFuncSetAttribute(reinterpret_cast<const void*>(
                      Simt_f16f16f16f32f32ttt_m64n64k8m32n32k8A1_200_S1::gemm_kernel),
                      hipFuncAttributePreferredSharedMemoryCarveout, 100);
                  TV_ASSERT_RT_ERR(result == hipSuccess, "error");
              }
              {
                  tv::CUDAKernelTimerGuard timerguard("Simt_f16f16f16f32f32ttt_m64n64k8m32n32k8A1_200_S1", evtimer, reinterpret_cast<hipStream_t>(params.stream));
                  launcher(Simt_f16f16f16f32f32ttt_m64n64k8m32n32k8A1_200_S1::gemm_kernel, kernel_params);
              }
              TV_CHECK_CUDA_ERR_V2("Simt_f16f16f16f32f32ttt_m64n64k8m32n32k8A1_200_S1", "error with params", a.shape(), b.shape(), c.shape());
              return;
            }
          }
        }
      }
    }
  }
  if (algo_desp.trans_a() == false && algo_desp.trans_b() == true && algo_desp.trans_c() == false){
    if (algo_desp.tile_shape == std::array<int, 3>{128, 128, 8}){
      if (algo_desp.warp_tile_shape == std::array<int, 3>{32, 64, 8}){
        if (algo_desp.num_stage == 2 && algo_desp.dacc == 0 && algo_desp.dcomp == 0){
          if ((params.split_k_slices == 1)){
            if (algo_desp.access_per_vector == 1){
              found = true;
              const int* a_ptr = nullptr;
              if (!a_inds.empty()){
                  a_ptr = a_inds.data_ptr<const int>();
              }
              TV_ASSERT_RT_ERR(!c_inds.empty(), "c must not empty");
              // tv::ssprint(d.ndim() == 1 ? 0 : d_ten.stride(0), (d.ndim() == 1 ? nullptr : c_inds.data_ptr<const int>()) == nullptr, "WTF");
              GemmParamsSimt_f16f16f16f32f32tnt_m128n128k8m32n64k8A1_200_S1 kernel_params(
                  m, n, k, a_ten.data_ptr<const tv::half_t>(), b_ten.data_ptr<const tv::half_t>(),
                  c_ten.data_ptr<tv::half_t>(), d_ten.data_ptr<tv::half_t>(), 
                  a_ten.stride(0), b_ten.stride(0), c_ten.stride(0), d.ndim() == 1 ? 0 : d_ten.stride(0), 
                  a_ptr, c_inds.data_ptr<const int>(), d.ndim() == 1 ? nullptr : c_inds.data_ptr<const int>(), 
                  float(params.alpha), float(params.beta), 
                  float(params.act_alpha), float(params.act_beta),
                  params.act_type,
                  split_k_slices);
              tv::cuda::Launch launcher(kernel_params.grid_dims, dim3(256),
                                          10304, reinterpret_cast<hipStream_t>(params.stream));
              hipError_t result;
              if (10304 >= (48 << 10)) {
                  result = hipFuncSetAttribute(reinterpret_cast<const void*>(Simt_f16f16f16f32f32tnt_m128n128k8m32n64k8A1_200_S1::gemm_kernel),
                                                  hipFuncAttributeMaxDynamicSharedMemorySize,
                                                  10304);
                  TV_ASSERT_RT_ERR(result == hipSuccess, "error");
                  result = hipFuncSetAttribute(reinterpret_cast<const void*>(
                      Simt_f16f16f16f32f32tnt_m128n128k8m32n64k8A1_200_S1::gemm_kernel),
                      hipFuncAttributePreferredSharedMemoryCarveout, 100);
                  TV_ASSERT_RT_ERR(result == hipSuccess, "error");
              }
              {
                  tv::CUDAKernelTimerGuard timerguard("Simt_f16f16f16f32f32tnt_m128n128k8m32n64k8A1_200_S1", evtimer, reinterpret_cast<hipStream_t>(params.stream));
                  launcher(Simt_f16f16f16f32f32tnt_m128n128k8m32n64k8A1_200_S1::gemm_kernel, kernel_params);
              }
              TV_CHECK_CUDA_ERR_V2("Simt_f16f16f16f32f32tnt_m128n128k8m32n64k8A1_200_S1", "error with params", a.shape(), b.shape(), c.shape());
              return;
            }
          }
        }
      }
    }
    if (algo_desp.tile_shape == std::array<int, 3>{32, 64, 32}){
      if (algo_desp.warp_tile_shape == std::array<int, 3>{32, 32, 8}){
        if (algo_desp.num_stage == 2 && algo_desp.dacc == 0 && algo_desp.dcomp == 0){
          if ((params.split_k_slices == 1)){
            if (algo_desp.access_per_vector == 1){
              found = true;
              const int* a_ptr = nullptr;
              if (!a_inds.empty()){
                  a_ptr = a_inds.data_ptr<const int>();
              }
              TV_ASSERT_RT_ERR(!c_inds.empty(), "c must not empty");
              // tv::ssprint(d.ndim() == 1 ? 0 : d_ten.stride(0), (d.ndim() == 1 ? nullptr : c_inds.data_ptr<const int>()) == nullptr, "WTF");
              GemmParamsSimt_f16f16f16f32f32tnt_m32n64k32m32n32k8A1_200_S1 kernel_params(
                  m, n, k, a_ten.data_ptr<const tv::half_t>(), b_ten.data_ptr<const tv::half_t>(),
                  c_ten.data_ptr<tv::half_t>(), d_ten.data_ptr<tv::half_t>(), 
                  a_ten.stride(0), b_ten.stride(0), c_ten.stride(0), d.ndim() == 1 ? 0 : d_ten.stride(0), 
                  a_ptr, c_inds.data_ptr<const int>(), d.ndim() == 1 ? nullptr : c_inds.data_ptr<const int>(), 
                  float(params.alpha), float(params.beta), 
                  float(params.act_alpha), float(params.act_beta),
                  params.act_type,
                  split_k_slices);
              tv::cuda::Launch launcher(kernel_params.grid_dims, dim3(256),
                                          12800, reinterpret_cast<hipStream_t>(params.stream));
              hipError_t result;
              if (12800 >= (48 << 10)) {
                  result = hipFuncSetAttribute(reinterpret_cast<const void*>(Simt_f16f16f16f32f32tnt_m32n64k32m32n32k8A1_200_S1::gemm_kernel),
                                                  hipFuncAttributeMaxDynamicSharedMemorySize,
                                                  12800);
                  TV_ASSERT_RT_ERR(result == hipSuccess, "error");
                  result = hipFuncSetAttribute(reinterpret_cast<const void*>(
                      Simt_f16f16f16f32f32tnt_m32n64k32m32n32k8A1_200_S1::gemm_kernel),
                      hipFuncAttributePreferredSharedMemoryCarveout, 100);
                  TV_ASSERT_RT_ERR(result == hipSuccess, "error");
              }
              {
                  tv::CUDAKernelTimerGuard timerguard("Simt_f16f16f16f32f32tnt_m32n64k32m32n32k8A1_200_S1", evtimer, reinterpret_cast<hipStream_t>(params.stream));
                  launcher(Simt_f16f16f16f32f32tnt_m32n64k32m32n32k8A1_200_S1::gemm_kernel, kernel_params);
              }
              TV_CHECK_CUDA_ERR_V2("Simt_f16f16f16f32f32tnt_m32n64k32m32n32k8A1_200_S1", "error with params", a.shape(), b.shape(), c.shape());
              return;
            }
          }
        }
      }
    }
    if (algo_desp.tile_shape == std::array<int, 3>{32, 32, 32}){
      if (algo_desp.warp_tile_shape == std::array<int, 3>{32, 32, 8}){
        if (algo_desp.num_stage == 2 && algo_desp.dacc == 0 && algo_desp.dcomp == 0){
          if ((params.split_k_slices == 1)){
            if (algo_desp.access_per_vector == 1){
              found = true;
              const int* a_ptr = nullptr;
              if (!a_inds.empty()){
                  a_ptr = a_inds.data_ptr<const int>();
              }
              TV_ASSERT_RT_ERR(!c_inds.empty(), "c must not empty");
              // tv::ssprint(d.ndim() == 1 ? 0 : d_ten.stride(0), (d.ndim() == 1 ? nullptr : c_inds.data_ptr<const int>()) == nullptr, "WTF");
              GemmParamsSimt_f16f16f16f32f32tnt_m32n32k32m32n32k8A1_200_S1 kernel_params(
                  m, n, k, a_ten.data_ptr<const tv::half_t>(), b_ten.data_ptr<const tv::half_t>(),
                  c_ten.data_ptr<tv::half_t>(), d_ten.data_ptr<tv::half_t>(), 
                  a_ten.stride(0), b_ten.stride(0), c_ten.stride(0), d.ndim() == 1 ? 0 : d_ten.stride(0), 
                  a_ptr, c_inds.data_ptr<const int>(), d.ndim() == 1 ? nullptr : c_inds.data_ptr<const int>(), 
                  float(params.alpha), float(params.beta), 
                  float(params.act_alpha), float(params.act_beta),
                  params.act_type,
                  split_k_slices);
              tv::cuda::Launch launcher(kernel_params.grid_dims, dim3(128),
                                          8704, reinterpret_cast<hipStream_t>(params.stream));
              hipError_t result;
              if (8704 >= (48 << 10)) {
                  result = hipFuncSetAttribute(reinterpret_cast<const void*>(Simt_f16f16f16f32f32tnt_m32n32k32m32n32k8A1_200_S1::gemm_kernel),
                                                  hipFuncAttributeMaxDynamicSharedMemorySize,
                                                  8704);
                  TV_ASSERT_RT_ERR(result == hipSuccess, "error");
                  result = hipFuncSetAttribute(reinterpret_cast<const void*>(
                      Simt_f16f16f16f32f32tnt_m32n32k32m32n32k8A1_200_S1::gemm_kernel),
                      hipFuncAttributePreferredSharedMemoryCarveout, 100);
                  TV_ASSERT_RT_ERR(result == hipSuccess, "error");
              }
              {
                  tv::CUDAKernelTimerGuard timerguard("Simt_f16f16f16f32f32tnt_m32n32k32m32n32k8A1_200_S1", evtimer, reinterpret_cast<hipStream_t>(params.stream));
                  launcher(Simt_f16f16f16f32f32tnt_m32n32k32m32n32k8A1_200_S1::gemm_kernel, kernel_params);
              }
              TV_CHECK_CUDA_ERR_V2("Simt_f16f16f16f32f32tnt_m32n32k32m32n32k8A1_200_S1", "error with params", a.shape(), b.shape(), c.shape());
              return;
            }
          }
        }
      }
    }
    if (algo_desp.tile_shape == std::array<int, 3>{64, 128, 16}){
      if (algo_desp.warp_tile_shape == std::array<int, 3>{32, 64, 8}){
        if (algo_desp.num_stage == 2 && algo_desp.dacc == 0 && algo_desp.dcomp == 0){
          if ((params.split_k_slices == 1)){
            if (algo_desp.access_per_vector == 1){
              found = true;
              const int* a_ptr = nullptr;
              if (!a_inds.empty()){
                  a_ptr = a_inds.data_ptr<const int>();
              }
              TV_ASSERT_RT_ERR(!c_inds.empty(), "c must not empty");
              // tv::ssprint(d.ndim() == 1 ? 0 : d_ten.stride(0), (d.ndim() == 1 ? nullptr : c_inds.data_ptr<const int>()) == nullptr, "WTF");
              GemmParamsSimt_f16f16f16f32f32tnt_m64n128k16m32n64k8A1_200_S1 kernel_params(
                  m, n, k, a_ten.data_ptr<const tv::half_t>(), b_ten.data_ptr<const tv::half_t>(),
                  c_ten.data_ptr<tv::half_t>(), d_ten.data_ptr<tv::half_t>(), 
                  a_ten.stride(0), b_ten.stride(0), c_ten.stride(0), d.ndim() == 1 ? 0 : d_ten.stride(0), 
                  a_ptr, c_inds.data_ptr<const int>(), d.ndim() == 1 ? nullptr : c_inds.data_ptr<const int>(), 
                  float(params.alpha), float(params.beta), 
                  float(params.act_alpha), float(params.act_beta),
                  params.act_type,
                  split_k_slices);
              tv::cuda::Launch launcher(kernel_params.grid_dims, dim3(256),
                                          12800, reinterpret_cast<hipStream_t>(params.stream));
              hipError_t result;
              if (12800 >= (48 << 10)) {
                  result = hipFuncSetAttribute(reinterpret_cast<const void*>(Simt_f16f16f16f32f32tnt_m64n128k16m32n64k8A1_200_S1::gemm_kernel),
                                                  hipFuncAttributeMaxDynamicSharedMemorySize,
                                                  12800);
                  TV_ASSERT_RT_ERR(result == hipSuccess, "error");
                  result = hipFuncSetAttribute(reinterpret_cast<const void*>(
                      Simt_f16f16f16f32f32tnt_m64n128k16m32n64k8A1_200_S1::gemm_kernel),
                      hipFuncAttributePreferredSharedMemoryCarveout, 100);
                  TV_ASSERT_RT_ERR(result == hipSuccess, "error");
              }
              {
                  tv::CUDAKernelTimerGuard timerguard("Simt_f16f16f16f32f32tnt_m64n128k16m32n64k8A1_200_S1", evtimer, reinterpret_cast<hipStream_t>(params.stream));
                  launcher(Simt_f16f16f16f32f32tnt_m64n128k16m32n64k8A1_200_S1::gemm_kernel, kernel_params);
              }
              TV_CHECK_CUDA_ERR_V2("Simt_f16f16f16f32f32tnt_m64n128k16m32n64k8A1_200_S1", "error with params", a.shape(), b.shape(), c.shape());
              return;
            }
          }
        }
      }
    }
    if (algo_desp.tile_shape == std::array<int, 3>{64, 64, 8}){
      if (algo_desp.warp_tile_shape == std::array<int, 3>{32, 32, 8}){
        if (algo_desp.num_stage == 2 && algo_desp.dacc == 0 && algo_desp.dcomp == 0){
          if ((params.split_k_slices == 1)){
            if (algo_desp.access_per_vector == 1){
              found = true;
              const int* a_ptr = nullptr;
              if (!a_inds.empty()){
                  a_ptr = a_inds.data_ptr<const int>();
              }
              TV_ASSERT_RT_ERR(!c_inds.empty(), "c must not empty");
              // tv::ssprint(d.ndim() == 1 ? 0 : d_ten.stride(0), (d.ndim() == 1 ? nullptr : c_inds.data_ptr<const int>()) == nullptr, "WTF");
              GemmParamsSimt_f16f16f16f32f32tnt_m64n64k8m32n32k8A1_200_S1 kernel_params(
                  m, n, k, a_ten.data_ptr<const tv::half_t>(), b_ten.data_ptr<const tv::half_t>(),
                  c_ten.data_ptr<tv::half_t>(), d_ten.data_ptr<tv::half_t>(), 
                  a_ten.stride(0), b_ten.stride(0), c_ten.stride(0), d.ndim() == 1 ? 0 : d_ten.stride(0), 
                  a_ptr, c_inds.data_ptr<const int>(), d.ndim() == 1 ? nullptr : c_inds.data_ptr<const int>(), 
                  float(params.alpha), float(params.beta), 
                  float(params.act_alpha), float(params.act_beta),
                  params.act_type,
                  split_k_slices);
              tv::cuda::Launch launcher(kernel_params.grid_dims, dim3(128),
                                          4608, reinterpret_cast<hipStream_t>(params.stream));
              hipError_t result;
              if (4608 >= (48 << 10)) {
                  result = hipFuncSetAttribute(reinterpret_cast<const void*>(Simt_f16f16f16f32f32tnt_m64n64k8m32n32k8A1_200_S1::gemm_kernel),
                                                  hipFuncAttributeMaxDynamicSharedMemorySize,
                                                  4608);
                  TV_ASSERT_RT_ERR(result == hipSuccess, "error");
                  result = hipFuncSetAttribute(reinterpret_cast<const void*>(
                      Simt_f16f16f16f32f32tnt_m64n64k8m32n32k8A1_200_S1::gemm_kernel),
                      hipFuncAttributePreferredSharedMemoryCarveout, 100);
                  TV_ASSERT_RT_ERR(result == hipSuccess, "error");
              }
              {
                  tv::CUDAKernelTimerGuard timerguard("Simt_f16f16f16f32f32tnt_m64n64k8m32n32k8A1_200_S1", evtimer, reinterpret_cast<hipStream_t>(params.stream));
                  launcher(Simt_f16f16f16f32f32tnt_m64n64k8m32n32k8A1_200_S1::gemm_kernel, kernel_params);
              }
              TV_CHECK_CUDA_ERR_V2("Simt_f16f16f16f32f32tnt_m64n64k8m32n32k8A1_200_S1", "error with params", a.shape(), b.shape(), c.shape());
              return;
            }
          }
        }
      }
    }
  }
  if (!found){
      TV_THROW_INVALID_ARG("Can't Found Algorithm for params:", algo_desp.tile_shape, algo_desp.warp_tile_shape, 
          algo_desp.num_stage, tv::dtype_str(a.dtype()), 
          tv::dtype_str(b.dtype()), tv::dtype_str(c.dtype()), tv::dtype_str(dacc), 
          tv::dtype_str(dcomp), ta, tb, tc, algo_desp.algo, algo_desp.tensorop);
  }
  // return 0;
}
} // namespace main
} // namespace gemm
} // namespace cumm
} // namespace spconvlib