#include <spconvlib/cumm/common/CompileInfo.h>
#include <spconvlib/cumm/common/_CudaInclude.h>
namespace spconvlib {
namespace cumm {
namespace common {
using _CudaInclude = spconvlib::cumm::common::_CudaInclude;
std::tuple<int, int> CompileInfo::get_compiled_cuda_version()   {
  
  #ifdef __CUDACC_VER_MAJOR__
  return std::make_tuple(__CUDACC_VER_MAJOR__, __CUDACC_VER_MINOR__);
  #else
  int ver = CUDA_VERSION; // from hip/hip_runtime.h
  return std::make_tuple(ver / 1000, (ver % 1000) / 10);
  #endif
}
} // namespace common
} // namespace cumm
} // namespace spconvlib