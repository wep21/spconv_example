#include <spconvlib/spconv/csrc/hash/core/HashTable.h>
#include <spconvlib/cumm/common/TensorViewHashKernel.h>
namespace spconvlib {
namespace spconv {
namespace csrc {
namespace hash {
namespace core {
using TensorView = spconvlib::cumm::common::TensorView;
using TslRobinMap = spconvlib::cumm::common::TslRobinMap;
using TensorViewHashKernel = spconvlib::cumm::common::TensorViewHashKernel;
void HashTable::insert(tv::Tensor keys, tv::Tensor values, std::uintptr_t stream)   {
  
  if (!is_cpu){
      int64_t value_after_insert = keys.dim(0) + insert_count_;
      TV_ASSERT_RT_ERR(value_after_insert < keys_data.dim(0), "inserted count exceed maximum hash size");
      insert_count_ += keys.dim(0);
      TV_ASSERT_RT_ERR(keys.dtype() == keys_data.dtype(), "keys dtype not equal to", keys_data.dtype());
  }
  auto N = keys.dim(0);
  if (!values.empty()){
      TV_ASSERT_RT_ERR(values.itemsize() == value_itemsize_, "values itemsize not equal to", value_itemsize_);
      TV_ASSERT_RT_ERR(keys.dim(0) == values.dim(0), "number of key and value must same");
  }
  if (is_cpu){
    {
      bool found = false;
      if (key_itemsize_ == 4 && value_itemsize_ == 4){
        auto& cpu_map = map_4_4;
        auto k_ptr = reinterpret_cast<const uint32_t*>(keys.raw_data());
        if (values.empty()){
            for (size_t i = 0; i < N; ++i){
                cpu_map.insert({k_ptr[i], uint32_t(0)});
            }
        }
        else{
            auto v_ptr = reinterpret_cast<const uint32_t*>(values.raw_data());
            for (size_t i = 0; i < N; ++i){
                cpu_map.insert({k_ptr[i], v_ptr[i]});
            }
        }
        found = true;
      }
      if (key_itemsize_ == 4 && value_itemsize_ == 8){
        auto& cpu_map = map_4_8;
        auto k_ptr = reinterpret_cast<const uint32_t*>(keys.raw_data());
        if (values.empty()){
            for (size_t i = 0; i < N; ++i){
                cpu_map.insert({k_ptr[i], uint64_t(0)});
            }
        }
        else{
            auto v_ptr = reinterpret_cast<const uint64_t*>(values.raw_data());
            for (size_t i = 0; i < N; ++i){
                cpu_map.insert({k_ptr[i], v_ptr[i]});
            }
        }
        found = true;
      }
      if (key_itemsize_ == 8 && value_itemsize_ == 4){
        auto& cpu_map = map_8_4;
        auto k_ptr = reinterpret_cast<const uint64_t*>(keys.raw_data());
        if (values.empty()){
            for (size_t i = 0; i < N; ++i){
                cpu_map.insert({k_ptr[i], uint32_t(0)});
            }
        }
        else{
            auto v_ptr = reinterpret_cast<const uint32_t*>(values.raw_data());
            for (size_t i = 0; i < N; ++i){
                cpu_map.insert({k_ptr[i], v_ptr[i]});
            }
        }
        found = true;
      }
      if (key_itemsize_ == 8 && value_itemsize_ == 8){
        auto& cpu_map = map_8_8;
        auto k_ptr = reinterpret_cast<const uint64_t*>(keys.raw_data());
        if (values.empty()){
            for (size_t i = 0; i < N; ++i){
                cpu_map.insert({k_ptr[i], uint64_t(0)});
            }
        }
        else{
            auto v_ptr = reinterpret_cast<const uint64_t*>(values.raw_data());
            for (size_t i = 0; i < N; ++i){
                cpu_map.insert({k_ptr[i], v_ptr[i]});
            }
        }
        found = true;
      }
      TV_ASSERT_RT_ERR(found, "suitable hash table not found.");
    }
  }
  else{
    auto custream = reinterpret_cast<hipStream_t>(stream);
    if (keys_data.dtype() == tv::DType(1)){
      using K = int32_t;
      K* key_data_ptr = reinterpret_cast<K*>(keys_data.raw_data());
      const K* key_ptr = reinterpret_cast<const K*>(keys.raw_data());
      if (values_data.itemsize() == 4){
        using V = tv::hash::itemsize_to_unsigned_t<4>;
        V* value_data_ptr = reinterpret_cast<V*>(values_data.raw_data());
        const V* value_ptr = reinterpret_cast<const V*>(values.raw_data());
        using table_t =
            tv::hash::LinearHashTableSplit<K, V, tv::hash::Murmur3Hash<K>,
                                        tv::hash::default_empty_key_v<K>, false>;
        tv::cuda::Launch launcher(N, custream);
        table_t table(key_data_ptr, value_data_ptr, keys_data.dim(0));
        launcher(tv::hash::insert_split<table_t>, table, key_ptr, value_ptr, size_t(N));
      }
      else if (values_data.itemsize() == 8){
        using V = tv::hash::itemsize_to_unsigned_t<8>;
        V* value_data_ptr = reinterpret_cast<V*>(values_data.raw_data());
        const V* value_ptr = reinterpret_cast<const V*>(values.raw_data());
        using table_t =
            tv::hash::LinearHashTableSplit<K, V, tv::hash::Murmur3Hash<K>,
                                        tv::hash::default_empty_key_v<K>, false>;
        tv::cuda::Launch launcher(N, custream);
        table_t table(key_data_ptr, value_data_ptr, keys_data.dim(0));
        launcher(tv::hash::insert_split<table_t>, table, key_ptr, value_ptr, size_t(N));
      }
      else{
        TV_THROW_RT_ERR("unknown val values_data.itemsize(), available: [4, 8]")
      }
    }
    else if (keys_data.dtype() == tv::DType(8)){
      using K = int64_t;
      K* key_data_ptr = reinterpret_cast<K*>(keys_data.raw_data());
      const K* key_ptr = reinterpret_cast<const K*>(keys.raw_data());
      if (values_data.itemsize() == 4){
        using V = tv::hash::itemsize_to_unsigned_t<4>;
        V* value_data_ptr = reinterpret_cast<V*>(values_data.raw_data());
        const V* value_ptr = reinterpret_cast<const V*>(values.raw_data());
        using table_t =
            tv::hash::LinearHashTableSplit<K, V, tv::hash::Murmur3Hash<K>,
                                        tv::hash::default_empty_key_v<K>, false>;
        tv::cuda::Launch launcher(N, custream);
        table_t table(key_data_ptr, value_data_ptr, keys_data.dim(0));
        launcher(tv::hash::insert_split<table_t>, table, key_ptr, value_ptr, size_t(N));
      }
      else if (values_data.itemsize() == 8){
        using V = tv::hash::itemsize_to_unsigned_t<8>;
        V* value_data_ptr = reinterpret_cast<V*>(values_data.raw_data());
        const V* value_ptr = reinterpret_cast<const V*>(values.raw_data());
        using table_t =
            tv::hash::LinearHashTableSplit<K, V, tv::hash::Murmur3Hash<K>,
                                        tv::hash::default_empty_key_v<K>, false>;
        tv::cuda::Launch launcher(N, custream);
        table_t table(key_data_ptr, value_data_ptr, keys_data.dim(0));
        launcher(tv::hash::insert_split<table_t>, table, key_ptr, value_ptr, size_t(N));
      }
      else{
        TV_THROW_RT_ERR("unknown val values_data.itemsize(), available: [4, 8]")
      }
    }
    else if (keys_data.dtype() == tv::DType(10)){
      using K = uint32_t;
      K* key_data_ptr = reinterpret_cast<K*>(keys_data.raw_data());
      const K* key_ptr = reinterpret_cast<const K*>(keys.raw_data());
      if (values_data.itemsize() == 4){
        using V = tv::hash::itemsize_to_unsigned_t<4>;
        V* value_data_ptr = reinterpret_cast<V*>(values_data.raw_data());
        const V* value_ptr = reinterpret_cast<const V*>(values.raw_data());
        using table_t =
            tv::hash::LinearHashTableSplit<K, V, tv::hash::Murmur3Hash<K>,
                                        tv::hash::default_empty_key_v<K>, false>;
        tv::cuda::Launch launcher(N, custream);
        table_t table(key_data_ptr, value_data_ptr, keys_data.dim(0));
        launcher(tv::hash::insert_split<table_t>, table, key_ptr, value_ptr, size_t(N));
      }
      else if (values_data.itemsize() == 8){
        using V = tv::hash::itemsize_to_unsigned_t<8>;
        V* value_data_ptr = reinterpret_cast<V*>(values_data.raw_data());
        const V* value_ptr = reinterpret_cast<const V*>(values.raw_data());
        using table_t =
            tv::hash::LinearHashTableSplit<K, V, tv::hash::Murmur3Hash<K>,
                                        tv::hash::default_empty_key_v<K>, false>;
        tv::cuda::Launch launcher(N, custream);
        table_t table(key_data_ptr, value_data_ptr, keys_data.dim(0));
        launcher(tv::hash::insert_split<table_t>, table, key_ptr, value_ptr, size_t(N));
      }
      else{
        TV_THROW_RT_ERR("unknown val values_data.itemsize(), available: [4, 8]")
      }
    }
    else if (keys_data.dtype() == tv::DType(11)){
      using K = uint64_t;
      K* key_data_ptr = reinterpret_cast<K*>(keys_data.raw_data());
      const K* key_ptr = reinterpret_cast<const K*>(keys.raw_data());
      if (values_data.itemsize() == 4){
        using V = tv::hash::itemsize_to_unsigned_t<4>;
        V* value_data_ptr = reinterpret_cast<V*>(values_data.raw_data());
        const V* value_ptr = reinterpret_cast<const V*>(values.raw_data());
        using table_t =
            tv::hash::LinearHashTableSplit<K, V, tv::hash::Murmur3Hash<K>,
                                        tv::hash::default_empty_key_v<K>, false>;
        tv::cuda::Launch launcher(N, custream);
        table_t table(key_data_ptr, value_data_ptr, keys_data.dim(0));
        launcher(tv::hash::insert_split<table_t>, table, key_ptr, value_ptr, size_t(N));
      }
      else if (values_data.itemsize() == 8){
        using V = tv::hash::itemsize_to_unsigned_t<8>;
        V* value_data_ptr = reinterpret_cast<V*>(values_data.raw_data());
        const V* value_ptr = reinterpret_cast<const V*>(values.raw_data());
        using table_t =
            tv::hash::LinearHashTableSplit<K, V, tv::hash::Murmur3Hash<K>,
                                        tv::hash::default_empty_key_v<K>, false>;
        tv::cuda::Launch launcher(N, custream);
        table_t table(key_data_ptr, value_data_ptr, keys_data.dim(0));
        launcher(tv::hash::insert_split<table_t>, table, key_ptr, value_ptr, size_t(N));
      }
      else{
        TV_THROW_RT_ERR("unknown val values_data.itemsize(), available: [4, 8]")
      }
    }
    else{
      TV_THROW_RT_ERR("unknown dtype keys_data.dtype(), available: [int32_t, int64_t, uint32_t, uint64_t]")
    }
  }
}
} // namespace core
} // namespace hash
} // namespace csrc
} // namespace spconv
} // namespace spconvlib