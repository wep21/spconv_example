#include <spconvlib/spconv/csrc/hash/core/HashTable.h>
#include <spconvlib/cumm/common/TensorViewHashKernel.h>
namespace spconvlib {
namespace spconv {
namespace csrc {
namespace hash {
namespace core {
using TensorView = spconvlib::cumm::common::TensorView;
using TslRobinMap = spconvlib::cumm::common::TslRobinMap;
using TensorViewHashKernel = spconvlib::cumm::common::TensorViewHashKernel;
void HashTable::items(tv::Tensor keys, tv::Tensor values, tv::Tensor count, std::uintptr_t stream)   {
  
  auto N = keys.dim(0);
  TV_ASSERT_RT_ERR(keys.itemsize() == key_itemsize_, "keys itemsize not equal to", key_itemsize_);
  TV_ASSERT_RT_ERR(values.itemsize() == value_itemsize_, "values itemsize not equal to", value_itemsize_);
  TV_ASSERT_RT_ERR(N == values.dim(0), "number of key and value must same");
  if (!is_cpu){
      TV_ASSERT_RT_ERR(keys.dtype() == keys_data.dtype(), "keys dtype not equal to", keys_data.dtype());
  }
  if (is_cpu){
    {
      bool found = false;
      if (key_itemsize_ == 4 && value_itemsize_ == 4){
        auto& cpu_map = map_4_4;
        auto k_ptr = reinterpret_cast<uint32_t*>(keys.raw_data());
        auto v_ptr = reinterpret_cast<uint32_t*>(values.raw_data());
        uint32_t index = 0;
        for (auto it = cpu_map.begin(); it != cpu_map.end(); ++it){
            if (index >= N){
                break;
            }
            k_ptr[index] = it->first;
            v_ptr[index] = it->second;
            ++index;
        }
        found = true;
      }
      if (key_itemsize_ == 4 && value_itemsize_ == 8){
        auto& cpu_map = map_4_8;
        auto k_ptr = reinterpret_cast<uint32_t*>(keys.raw_data());
        auto v_ptr = reinterpret_cast<uint64_t*>(values.raw_data());
        uint64_t index = 0;
        for (auto it = cpu_map.begin(); it != cpu_map.end(); ++it){
            if (index >= N){
                break;
            }
            k_ptr[index] = it->first;
            v_ptr[index] = it->second;
            ++index;
        }
        found = true;
      }
      if (key_itemsize_ == 8 && value_itemsize_ == 4){
        auto& cpu_map = map_8_4;
        auto k_ptr = reinterpret_cast<uint64_t*>(keys.raw_data());
        auto v_ptr = reinterpret_cast<uint32_t*>(values.raw_data());
        uint32_t index = 0;
        for (auto it = cpu_map.begin(); it != cpu_map.end(); ++it){
            if (index >= N){
                break;
            }
            k_ptr[index] = it->first;
            v_ptr[index] = it->second;
            ++index;
        }
        found = true;
      }
      if (key_itemsize_ == 8 && value_itemsize_ == 8){
        auto& cpu_map = map_8_8;
        auto k_ptr = reinterpret_cast<uint64_t*>(keys.raw_data());
        auto v_ptr = reinterpret_cast<uint64_t*>(values.raw_data());
        uint64_t index = 0;
        for (auto it = cpu_map.begin(); it != cpu_map.end(); ++it){
            if (index >= N){
                break;
            }
            k_ptr[index] = it->first;
            v_ptr[index] = it->second;
            ++index;
        }
        found = true;
      }
      TV_ASSERT_RT_ERR(found, "suitable hash table not found.");
    }
  }
  else{
    auto custream = reinterpret_cast<hipStream_t>(stream);
    if (keys_data.dtype() == tv::DType(1)){
      using K = int32_t;
      using Kunsigned = tv::hash::itemsize_to_unsigned_t<sizeof(K)>;
      auto count_ptr = count.data_ptr<Kunsigned>();
      K* key_data_ptr = reinterpret_cast<K*>(keys_data.raw_data());
      K* key_ptr = reinterpret_cast<K*>(keys.raw_data());
      if (values_data.itemsize() == 4){
        using V = tv::hash::itemsize_to_unsigned_t<4>;
        V* value_data_ptr = reinterpret_cast<V*>(values_data.raw_data());
        V* value_ptr = reinterpret_cast<V*>(values.raw_data());
        using table_t =
            tv::hash::LinearHashTableSplit<K, V, tv::hash::Murmur3Hash<K>,
                                        tv::hash::default_empty_key_v<K>, false>;
        tv::cuda::Launch launcher(N, custream);
        table_t table(key_data_ptr, value_data_ptr, keys_data.dim(0));
        launcher(tv::hash::iterate_table_split<table_t, Kunsigned>, table, key_ptr, value_ptr, size_t(N), count_ptr);
      }
      else if (values_data.itemsize() == 8){
        using V = tv::hash::itemsize_to_unsigned_t<8>;
        V* value_data_ptr = reinterpret_cast<V*>(values_data.raw_data());
        V* value_ptr = reinterpret_cast<V*>(values.raw_data());
        using table_t =
            tv::hash::LinearHashTableSplit<K, V, tv::hash::Murmur3Hash<K>,
                                        tv::hash::default_empty_key_v<K>, false>;
        tv::cuda::Launch launcher(N, custream);
        table_t table(key_data_ptr, value_data_ptr, keys_data.dim(0));
        launcher(tv::hash::iterate_table_split<table_t, Kunsigned>, table, key_ptr, value_ptr, size_t(N), count_ptr);
      }
      else{
        TV_THROW_RT_ERR("unknown val values_data.itemsize(), available: [4, 8]")
      }
    }
    else if (keys_data.dtype() == tv::DType(8)){
      using K = int64_t;
      using Kunsigned = tv::hash::itemsize_to_unsigned_t<sizeof(K)>;
      auto count_ptr = count.data_ptr<Kunsigned>();
      K* key_data_ptr = reinterpret_cast<K*>(keys_data.raw_data());
      K* key_ptr = reinterpret_cast<K*>(keys.raw_data());
      if (values_data.itemsize() == 4){
        using V = tv::hash::itemsize_to_unsigned_t<4>;
        V* value_data_ptr = reinterpret_cast<V*>(values_data.raw_data());
        V* value_ptr = reinterpret_cast<V*>(values.raw_data());
        using table_t =
            tv::hash::LinearHashTableSplit<K, V, tv::hash::Murmur3Hash<K>,
                                        tv::hash::default_empty_key_v<K>, false>;
        tv::cuda::Launch launcher(N, custream);
        table_t table(key_data_ptr, value_data_ptr, keys_data.dim(0));
        launcher(tv::hash::iterate_table_split<table_t, Kunsigned>, table, key_ptr, value_ptr, size_t(N), count_ptr);
      }
      else if (values_data.itemsize() == 8){
        using V = tv::hash::itemsize_to_unsigned_t<8>;
        V* value_data_ptr = reinterpret_cast<V*>(values_data.raw_data());
        V* value_ptr = reinterpret_cast<V*>(values.raw_data());
        using table_t =
            tv::hash::LinearHashTableSplit<K, V, tv::hash::Murmur3Hash<K>,
                                        tv::hash::default_empty_key_v<K>, false>;
        tv::cuda::Launch launcher(N, custream);
        table_t table(key_data_ptr, value_data_ptr, keys_data.dim(0));
        launcher(tv::hash::iterate_table_split<table_t, Kunsigned>, table, key_ptr, value_ptr, size_t(N), count_ptr);
      }
      else{
        TV_THROW_RT_ERR("unknown val values_data.itemsize(), available: [4, 8]")
      }
    }
    else if (keys_data.dtype() == tv::DType(10)){
      using K = uint32_t;
      using Kunsigned = tv::hash::itemsize_to_unsigned_t<sizeof(K)>;
      auto count_ptr = count.data_ptr<Kunsigned>();
      K* key_data_ptr = reinterpret_cast<K*>(keys_data.raw_data());
      K* key_ptr = reinterpret_cast<K*>(keys.raw_data());
      if (values_data.itemsize() == 4){
        using V = tv::hash::itemsize_to_unsigned_t<4>;
        V* value_data_ptr = reinterpret_cast<V*>(values_data.raw_data());
        V* value_ptr = reinterpret_cast<V*>(values.raw_data());
        using table_t =
            tv::hash::LinearHashTableSplit<K, V, tv::hash::Murmur3Hash<K>,
                                        tv::hash::default_empty_key_v<K>, false>;
        tv::cuda::Launch launcher(N, custream);
        table_t table(key_data_ptr, value_data_ptr, keys_data.dim(0));
        launcher(tv::hash::iterate_table_split<table_t, Kunsigned>, table, key_ptr, value_ptr, size_t(N), count_ptr);
      }
      else if (values_data.itemsize() == 8){
        using V = tv::hash::itemsize_to_unsigned_t<8>;
        V* value_data_ptr = reinterpret_cast<V*>(values_data.raw_data());
        V* value_ptr = reinterpret_cast<V*>(values.raw_data());
        using table_t =
            tv::hash::LinearHashTableSplit<K, V, tv::hash::Murmur3Hash<K>,
                                        tv::hash::default_empty_key_v<K>, false>;
        tv::cuda::Launch launcher(N, custream);
        table_t table(key_data_ptr, value_data_ptr, keys_data.dim(0));
        launcher(tv::hash::iterate_table_split<table_t, Kunsigned>, table, key_ptr, value_ptr, size_t(N), count_ptr);
      }
      else{
        TV_THROW_RT_ERR("unknown val values_data.itemsize(), available: [4, 8]")
      }
    }
    else if (keys_data.dtype() == tv::DType(11)){
      using K = uint64_t;
      using Kunsigned = tv::hash::itemsize_to_unsigned_t<sizeof(K)>;
      auto count_ptr = count.data_ptr<Kunsigned>();
      K* key_data_ptr = reinterpret_cast<K*>(keys_data.raw_data());
      K* key_ptr = reinterpret_cast<K*>(keys.raw_data());
      if (values_data.itemsize() == 4){
        using V = tv::hash::itemsize_to_unsigned_t<4>;
        V* value_data_ptr = reinterpret_cast<V*>(values_data.raw_data());
        V* value_ptr = reinterpret_cast<V*>(values.raw_data());
        using table_t =
            tv::hash::LinearHashTableSplit<K, V, tv::hash::Murmur3Hash<K>,
                                        tv::hash::default_empty_key_v<K>, false>;
        tv::cuda::Launch launcher(N, custream);
        table_t table(key_data_ptr, value_data_ptr, keys_data.dim(0));
        launcher(tv::hash::iterate_table_split<table_t, Kunsigned>, table, key_ptr, value_ptr, size_t(N), count_ptr);
      }
      else if (values_data.itemsize() == 8){
        using V = tv::hash::itemsize_to_unsigned_t<8>;
        V* value_data_ptr = reinterpret_cast<V*>(values_data.raw_data());
        V* value_ptr = reinterpret_cast<V*>(values.raw_data());
        using table_t =
            tv::hash::LinearHashTableSplit<K, V, tv::hash::Murmur3Hash<K>,
                                        tv::hash::default_empty_key_v<K>, false>;
        tv::cuda::Launch launcher(N, custream);
        table_t table(key_data_ptr, value_data_ptr, keys_data.dim(0));
        launcher(tv::hash::iterate_table_split<table_t, Kunsigned>, table, key_ptr, value_ptr, size_t(N), count_ptr);
      }
      else{
        TV_THROW_RT_ERR("unknown val values_data.itemsize(), available: [4, 8]")
      }
    }
    else{
      TV_THROW_RT_ERR("unknown dtype keys_data.dtype(), available: [int32_t, int64_t, uint32_t, uint64_t]")
    }
  }
}
} // namespace core
} // namespace hash
} // namespace csrc
} // namespace spconv
} // namespace spconvlib