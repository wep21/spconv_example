#include <spconvlib/spconv/csrc/hash/core/HashTable.h>
#include <spconvlib/cumm/common/TensorViewHashKernel.h>
#include <spconvlib/spconv/csrc/hash/core/HashTableKernel.h>
namespace spconvlib {
namespace spconv {
namespace csrc {
namespace hash {
namespace core {
using TensorView = spconvlib::cumm::common::TensorView;
using TslRobinMap = spconvlib::cumm::common::TslRobinMap;
using TensorViewHashKernel = spconvlib::cumm::common::TensorViewHashKernel;
using HashTableKernel = spconvlib::spconv::csrc::hash::core::HashTableKernel;
void HashTable::insert_exist_keys(tv::Tensor keys, tv::Tensor values, tv::Tensor is_empty, std::uintptr_t stream)   {
  
  auto N = keys.dim(0);
  TV_ASSERT_RT_ERR(keys.itemsize() == key_itemsize_, "keys itemsize not equal to", key_itemsize_);
  TV_ASSERT_RT_ERR(values.itemsize() == value_itemsize_, "values itemsize not equal to", value_itemsize_);
  TV_ASSERT_RT_ERR(N == values.dim(0) && is_empty.dim(0) == N, "number of key and value must same");
  auto is_empty_ptr = is_empty.data_ptr<uint8_t>();
  if (is_cpu){
    {
      bool found = false;
      if (key_itemsize_ == 4 && value_itemsize_ == 4){
        auto& cpu_map = map_4_4;
        auto k_ptr = reinterpret_cast<uint32_t*>(keys.raw_data());
        auto v_ptr = reinterpret_cast<uint32_t*>(values.raw_data());
        tv::kernel_1d_cpu(keys.device(), N, [&](size_t begin, size_t end, size_t step){
            bool emp;
            for (size_t i = begin; i < end; i += step){
                auto iter = cpu_map.find(k_ptr[i]);
                emp = iter == cpu_map.end();
                if (!emp){
                    iter.value() = v_ptr[i];
                }
                is_empty_ptr[i] = uint8_t(emp);
            }
        });
        found = true;
      }
      if (key_itemsize_ == 4 && value_itemsize_ == 8){
        auto& cpu_map = map_4_8;
        auto k_ptr = reinterpret_cast<uint32_t*>(keys.raw_data());
        auto v_ptr = reinterpret_cast<uint64_t*>(values.raw_data());
        tv::kernel_1d_cpu(keys.device(), N, [&](size_t begin, size_t end, size_t step){
            bool emp;
            for (size_t i = begin; i < end; i += step){
                auto iter = cpu_map.find(k_ptr[i]);
                emp = iter == cpu_map.end();
                if (!emp){
                    iter.value() = v_ptr[i];
                }
                is_empty_ptr[i] = uint8_t(emp);
            }
        });
        found = true;
      }
      if (key_itemsize_ == 8 && value_itemsize_ == 4){
        auto& cpu_map = map_8_4;
        auto k_ptr = reinterpret_cast<uint64_t*>(keys.raw_data());
        auto v_ptr = reinterpret_cast<uint32_t*>(values.raw_data());
        tv::kernel_1d_cpu(keys.device(), N, [&](size_t begin, size_t end, size_t step){
            bool emp;
            for (size_t i = begin; i < end; i += step){
                auto iter = cpu_map.find(k_ptr[i]);
                emp = iter == cpu_map.end();
                if (!emp){
                    iter.value() = v_ptr[i];
                }
                is_empty_ptr[i] = uint8_t(emp);
            }
        });
        found = true;
      }
      if (key_itemsize_ == 8 && value_itemsize_ == 8){
        auto& cpu_map = map_8_8;
        auto k_ptr = reinterpret_cast<uint64_t*>(keys.raw_data());
        auto v_ptr = reinterpret_cast<uint64_t*>(values.raw_data());
        tv::kernel_1d_cpu(keys.device(), N, [&](size_t begin, size_t end, size_t step){
            bool emp;
            for (size_t i = begin; i < end; i += step){
                auto iter = cpu_map.find(k_ptr[i]);
                emp = iter == cpu_map.end();
                if (!emp){
                    iter.value() = v_ptr[i];
                }
                is_empty_ptr[i] = uint8_t(emp);
            }
        });
        found = true;
      }
      TV_ASSERT_RT_ERR(found, "suitable hash table not found.");
    }
  }
  else{
    auto custream = reinterpret_cast<hipStream_t>(stream);
    if (keys_data.dtype() == tv::DType(1)){
      using K = int32_t;
      K* key_data_ptr = reinterpret_cast<K*>(keys_data.raw_data());
      const K* key_ptr = reinterpret_cast<const K*>(keys.raw_data());
      if (values_data.itemsize() == 4){
        using V = tv::hash::itemsize_to_unsigned_t<4>;
        V* value_data_ptr = reinterpret_cast<V*>(values_data.raw_data());
        const V* value_ptr = reinterpret_cast<const V*>(values.raw_data());
        using table_t =
            tv::hash::LinearHashTableSplit<K, V, tv::hash::Murmur3Hash<K>,
                                        tv::hash::default_empty_key_v<K>, false>;
        table_t table(key_data_ptr, value_data_ptr, keys_data.dim(0));
        tv::cuda::Launch launcher(N, custream);
        launcher(insert_exist_keys_kernel<table_t>, table, key_ptr, value_ptr, is_empty_ptr, size_t(N));
      }
      else if (values_data.itemsize() == 8){
        using V = tv::hash::itemsize_to_unsigned_t<8>;
        V* value_data_ptr = reinterpret_cast<V*>(values_data.raw_data());
        const V* value_ptr = reinterpret_cast<const V*>(values.raw_data());
        using table_t =
            tv::hash::LinearHashTableSplit<K, V, tv::hash::Murmur3Hash<K>,
                                        tv::hash::default_empty_key_v<K>, false>;
        table_t table(key_data_ptr, value_data_ptr, keys_data.dim(0));
        tv::cuda::Launch launcher(N, custream);
        launcher(insert_exist_keys_kernel<table_t>, table, key_ptr, value_ptr, is_empty_ptr, size_t(N));
      }
      else{
        TV_THROW_RT_ERR("unknown val values_data.itemsize(), available: [4, 8]")
      }
    }
    else if (keys_data.dtype() == tv::DType(8)){
      using K = int64_t;
      K* key_data_ptr = reinterpret_cast<K*>(keys_data.raw_data());
      const K* key_ptr = reinterpret_cast<const K*>(keys.raw_data());
      if (values_data.itemsize() == 4){
        using V = tv::hash::itemsize_to_unsigned_t<4>;
        V* value_data_ptr = reinterpret_cast<V*>(values_data.raw_data());
        const V* value_ptr = reinterpret_cast<const V*>(values.raw_data());
        using table_t =
            tv::hash::LinearHashTableSplit<K, V, tv::hash::Murmur3Hash<K>,
                                        tv::hash::default_empty_key_v<K>, false>;
        table_t table(key_data_ptr, value_data_ptr, keys_data.dim(0));
        tv::cuda::Launch launcher(N, custream);
        launcher(insert_exist_keys_kernel<table_t>, table, key_ptr, value_ptr, is_empty_ptr, size_t(N));
      }
      else if (values_data.itemsize() == 8){
        using V = tv::hash::itemsize_to_unsigned_t<8>;
        V* value_data_ptr = reinterpret_cast<V*>(values_data.raw_data());
        const V* value_ptr = reinterpret_cast<const V*>(values.raw_data());
        using table_t =
            tv::hash::LinearHashTableSplit<K, V, tv::hash::Murmur3Hash<K>,
                                        tv::hash::default_empty_key_v<K>, false>;
        table_t table(key_data_ptr, value_data_ptr, keys_data.dim(0));
        tv::cuda::Launch launcher(N, custream);
        launcher(insert_exist_keys_kernel<table_t>, table, key_ptr, value_ptr, is_empty_ptr, size_t(N));
      }
      else{
        TV_THROW_RT_ERR("unknown val values_data.itemsize(), available: [4, 8]")
      }
    }
    else if (keys_data.dtype() == tv::DType(10)){
      using K = uint32_t;
      K* key_data_ptr = reinterpret_cast<K*>(keys_data.raw_data());
      const K* key_ptr = reinterpret_cast<const K*>(keys.raw_data());
      if (values_data.itemsize() == 4){
        using V = tv::hash::itemsize_to_unsigned_t<4>;
        V* value_data_ptr = reinterpret_cast<V*>(values_data.raw_data());
        const V* value_ptr = reinterpret_cast<const V*>(values.raw_data());
        using table_t =
            tv::hash::LinearHashTableSplit<K, V, tv::hash::Murmur3Hash<K>,
                                        tv::hash::default_empty_key_v<K>, false>;
        table_t table(key_data_ptr, value_data_ptr, keys_data.dim(0));
        tv::cuda::Launch launcher(N, custream);
        launcher(insert_exist_keys_kernel<table_t>, table, key_ptr, value_ptr, is_empty_ptr, size_t(N));
      }
      else if (values_data.itemsize() == 8){
        using V = tv::hash::itemsize_to_unsigned_t<8>;
        V* value_data_ptr = reinterpret_cast<V*>(values_data.raw_data());
        const V* value_ptr = reinterpret_cast<const V*>(values.raw_data());
        using table_t =
            tv::hash::LinearHashTableSplit<K, V, tv::hash::Murmur3Hash<K>,
                                        tv::hash::default_empty_key_v<K>, false>;
        table_t table(key_data_ptr, value_data_ptr, keys_data.dim(0));
        tv::cuda::Launch launcher(N, custream);
        launcher(insert_exist_keys_kernel<table_t>, table, key_ptr, value_ptr, is_empty_ptr, size_t(N));
      }
      else{
        TV_THROW_RT_ERR("unknown val values_data.itemsize(), available: [4, 8]")
      }
    }
    else if (keys_data.dtype() == tv::DType(11)){
      using K = uint64_t;
      K* key_data_ptr = reinterpret_cast<K*>(keys_data.raw_data());
      const K* key_ptr = reinterpret_cast<const K*>(keys.raw_data());
      if (values_data.itemsize() == 4){
        using V = tv::hash::itemsize_to_unsigned_t<4>;
        V* value_data_ptr = reinterpret_cast<V*>(values_data.raw_data());
        const V* value_ptr = reinterpret_cast<const V*>(values.raw_data());
        using table_t =
            tv::hash::LinearHashTableSplit<K, V, tv::hash::Murmur3Hash<K>,
                                        tv::hash::default_empty_key_v<K>, false>;
        table_t table(key_data_ptr, value_data_ptr, keys_data.dim(0));
        tv::cuda::Launch launcher(N, custream);
        launcher(insert_exist_keys_kernel<table_t>, table, key_ptr, value_ptr, is_empty_ptr, size_t(N));
      }
      else if (values_data.itemsize() == 8){
        using V = tv::hash::itemsize_to_unsigned_t<8>;
        V* value_data_ptr = reinterpret_cast<V*>(values_data.raw_data());
        const V* value_ptr = reinterpret_cast<const V*>(values.raw_data());
        using table_t =
            tv::hash::LinearHashTableSplit<K, V, tv::hash::Murmur3Hash<K>,
                                        tv::hash::default_empty_key_v<K>, false>;
        table_t table(key_data_ptr, value_data_ptr, keys_data.dim(0));
        tv::cuda::Launch launcher(N, custream);
        launcher(insert_exist_keys_kernel<table_t>, table, key_ptr, value_ptr, is_empty_ptr, size_t(N));
      }
      else{
        TV_THROW_RT_ERR("unknown val values_data.itemsize(), available: [4, 8]")
      }
    }
    else{
      TV_THROW_RT_ERR("unknown dtype keys_data.dtype(), available: [int32_t, int64_t, uint32_t, uint64_t]")
    }
  }
}
} // namespace core
} // namespace hash
} // namespace csrc
} // namespace spconv
} // namespace spconvlib