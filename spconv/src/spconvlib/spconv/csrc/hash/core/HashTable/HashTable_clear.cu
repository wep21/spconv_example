#include <spconvlib/spconv/csrc/hash/core/HashTable.h>
#include <spconvlib/cumm/common/TensorViewHashKernel.h>
namespace spconvlib {
namespace spconv {
namespace csrc {
namespace hash {
namespace core {
using TensorView = spconvlib::cumm::common::TensorView;
using TslRobinMap = spconvlib::cumm::common::TslRobinMap;
using TensorViewHashKernel = spconvlib::cumm::common::TensorViewHashKernel;
void HashTable::clear(std::uintptr_t stream)   {
  
  if (is_cpu){
    if (is_cpu){
        map_4_4.clear();
        map_4_8.clear();
        map_8_4.clear();
        map_8_8.clear();
        return;
    }
  }
  else{
    auto custream = reinterpret_cast<hipStream_t>(stream);
    if (keys_data.dtype() == tv::DType(1)){
      using K = int32_t;
      K* key_data_ptr = reinterpret_cast<K*>(keys_data.raw_data());
      if (values_data.itemsize() == 4){
        using V = tv::hash::itemsize_to_unsigned_t<4>;
        V* value_data_ptr = reinterpret_cast<V*>(values_data.raw_data());
        using table_t =
            tv::hash::LinearHashTableSplit<K, V, tv::hash::Murmur3Hash<K>,
                                        tv::hash::default_empty_key_v<K>, false>;
        table_t table(key_data_ptr, value_data_ptr, keys_data.dim(0));
        tv::cuda::Launch launcher(table.size(), custream);
        launcher(tv::hash::clear_map_kernel_split<table_t>, table);
      }
      else if (values_data.itemsize() == 8){
        using V = tv::hash::itemsize_to_unsigned_t<8>;
        V* value_data_ptr = reinterpret_cast<V*>(values_data.raw_data());
        using table_t =
            tv::hash::LinearHashTableSplit<K, V, tv::hash::Murmur3Hash<K>,
                                        tv::hash::default_empty_key_v<K>, false>;
        table_t table(key_data_ptr, value_data_ptr, keys_data.dim(0));
        tv::cuda::Launch launcher(table.size(), custream);
        launcher(tv::hash::clear_map_kernel_split<table_t>, table);
      }
      else{
        TV_THROW_RT_ERR("unknown val values_data.itemsize(), available: [4, 8]")
      }
    }
    else if (keys_data.dtype() == tv::DType(8)){
      using K = int64_t;
      K* key_data_ptr = reinterpret_cast<K*>(keys_data.raw_data());
      if (values_data.itemsize() == 4){
        using V = tv::hash::itemsize_to_unsigned_t<4>;
        V* value_data_ptr = reinterpret_cast<V*>(values_data.raw_data());
        using table_t =
            tv::hash::LinearHashTableSplit<K, V, tv::hash::Murmur3Hash<K>,
                                        tv::hash::default_empty_key_v<K>, false>;
        table_t table(key_data_ptr, value_data_ptr, keys_data.dim(0));
        tv::cuda::Launch launcher(table.size(), custream);
        launcher(tv::hash::clear_map_kernel_split<table_t>, table);
      }
      else if (values_data.itemsize() == 8){
        using V = tv::hash::itemsize_to_unsigned_t<8>;
        V* value_data_ptr = reinterpret_cast<V*>(values_data.raw_data());
        using table_t =
            tv::hash::LinearHashTableSplit<K, V, tv::hash::Murmur3Hash<K>,
                                        tv::hash::default_empty_key_v<K>, false>;
        table_t table(key_data_ptr, value_data_ptr, keys_data.dim(0));
        tv::cuda::Launch launcher(table.size(), custream);
        launcher(tv::hash::clear_map_kernel_split<table_t>, table);
      }
      else{
        TV_THROW_RT_ERR("unknown val values_data.itemsize(), available: [4, 8]")
      }
    }
    else if (keys_data.dtype() == tv::DType(10)){
      using K = uint32_t;
      K* key_data_ptr = reinterpret_cast<K*>(keys_data.raw_data());
      if (values_data.itemsize() == 4){
        using V = tv::hash::itemsize_to_unsigned_t<4>;
        V* value_data_ptr = reinterpret_cast<V*>(values_data.raw_data());
        using table_t =
            tv::hash::LinearHashTableSplit<K, V, tv::hash::Murmur3Hash<K>,
                                        tv::hash::default_empty_key_v<K>, false>;
        table_t table(key_data_ptr, value_data_ptr, keys_data.dim(0));
        tv::cuda::Launch launcher(table.size(), custream);
        launcher(tv::hash::clear_map_kernel_split<table_t>, table);
      }
      else if (values_data.itemsize() == 8){
        using V = tv::hash::itemsize_to_unsigned_t<8>;
        V* value_data_ptr = reinterpret_cast<V*>(values_data.raw_data());
        using table_t =
            tv::hash::LinearHashTableSplit<K, V, tv::hash::Murmur3Hash<K>,
                                        tv::hash::default_empty_key_v<K>, false>;
        table_t table(key_data_ptr, value_data_ptr, keys_data.dim(0));
        tv::cuda::Launch launcher(table.size(), custream);
        launcher(tv::hash::clear_map_kernel_split<table_t>, table);
      }
      else{
        TV_THROW_RT_ERR("unknown val values_data.itemsize(), available: [4, 8]")
      }
    }
    else if (keys_data.dtype() == tv::DType(11)){
      using K = uint64_t;
      K* key_data_ptr = reinterpret_cast<K*>(keys_data.raw_data());
      if (values_data.itemsize() == 4){
        using V = tv::hash::itemsize_to_unsigned_t<4>;
        V* value_data_ptr = reinterpret_cast<V*>(values_data.raw_data());
        using table_t =
            tv::hash::LinearHashTableSplit<K, V, tv::hash::Murmur3Hash<K>,
                                        tv::hash::default_empty_key_v<K>, false>;
        table_t table(key_data_ptr, value_data_ptr, keys_data.dim(0));
        tv::cuda::Launch launcher(table.size(), custream);
        launcher(tv::hash::clear_map_kernel_split<table_t>, table);
      }
      else if (values_data.itemsize() == 8){
        using V = tv::hash::itemsize_to_unsigned_t<8>;
        V* value_data_ptr = reinterpret_cast<V*>(values_data.raw_data());
        using table_t =
            tv::hash::LinearHashTableSplit<K, V, tv::hash::Murmur3Hash<K>,
                                        tv::hash::default_empty_key_v<K>, false>;
        table_t table(key_data_ptr, value_data_ptr, keys_data.dim(0));
        tv::cuda::Launch launcher(table.size(), custream);
        launcher(tv::hash::clear_map_kernel_split<table_t>, table);
      }
      else{
        TV_THROW_RT_ERR("unknown val values_data.itemsize(), available: [4, 8]")
      }
    }
    else{
      TV_THROW_RT_ERR("unknown dtype keys_data.dtype(), available: [int32_t, int64_t, uint32_t, uint64_t]")
    }
  }
}
} // namespace core
} // namespace hash
} // namespace csrc
} // namespace spconv
} // namespace spconvlib