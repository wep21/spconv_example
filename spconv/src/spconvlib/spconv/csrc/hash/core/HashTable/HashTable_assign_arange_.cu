#include <spconvlib/spconv/csrc/hash/core/HashTable.h>
#include <spconvlib/cumm/common/TensorViewHashKernel.h>
namespace spconvlib {
namespace spconv {
namespace csrc {
namespace hash {
namespace core {
using TensorView = spconvlib::cumm::common::TensorView;
using TslRobinMap = spconvlib::cumm::common::TslRobinMap;
using TensorViewHashKernel = spconvlib::cumm::common::TensorViewHashKernel;
void HashTable::assign_arange_(tv::Tensor count, std::uintptr_t stream)   {
  
  if (is_cpu){
    {
      bool found = false;
      if (key_itemsize_ == 4 && value_itemsize_ == 4){
        auto& cpu_map = map_4_4;
        uint32_t index = 0;
        for (auto it = cpu_map.begin(); it != cpu_map.end(); ++it){
            it.value() = index;
            ++index;
        }
        found = true;
      }
      if (key_itemsize_ == 4 && value_itemsize_ == 8){
        auto& cpu_map = map_4_8;
        uint64_t index = 0;
        for (auto it = cpu_map.begin(); it != cpu_map.end(); ++it){
            it.value() = index;
            ++index;
        }
        found = true;
      }
      if (key_itemsize_ == 8 && value_itemsize_ == 4){
        auto& cpu_map = map_8_4;
        uint32_t index = 0;
        for (auto it = cpu_map.begin(); it != cpu_map.end(); ++it){
            it.value() = index;
            ++index;
        }
        found = true;
      }
      if (key_itemsize_ == 8 && value_itemsize_ == 8){
        auto& cpu_map = map_8_8;
        uint64_t index = 0;
        for (auto it = cpu_map.begin(); it != cpu_map.end(); ++it){
            it.value() = index;
            ++index;
        }
        found = true;
      }
      TV_ASSERT_RT_ERR(found, "suitable hash table not found.");
    }
  }
  else{
    TV_ASSERT_RT_ERR(count.device() == 0, "count must be cuda");
    auto custream = reinterpret_cast<hipStream_t>(stream);
    if (keys_data.dtype() == tv::DType(1)){
      using K = int32_t;
      using Kunsigned = tv::hash::itemsize_to_unsigned_t<sizeof(K)>;
      auto count_ptr = count.data_ptr<Kunsigned>();
      K* key_data_ptr = reinterpret_cast<K*>(keys_data.raw_data());
      if (values_data.dtype() == tv::DType(1)){
        using V = int32_t;
        V* value_data_ptr = reinterpret_cast<V*>(values_data.raw_data());
        using table_t =
            tv::hash::LinearHashTableSplit<K, V, tv::hash::Murmur3Hash<K>,
                                        tv::hash::default_empty_key_v<K>, false>;
        table_t table(key_data_ptr, value_data_ptr, keys_data.dim(0));
        tv::cuda::Launch launcher(table.size(), custream);
        launcher(tv::hash::assign_arange_split<table_t, Kunsigned>, table, count_ptr);
      }
      else if (values_data.dtype() == tv::DType(8)){
        using V = int64_t;
        V* value_data_ptr = reinterpret_cast<V*>(values_data.raw_data());
        using table_t =
            tv::hash::LinearHashTableSplit<K, V, tv::hash::Murmur3Hash<K>,
                                        tv::hash::default_empty_key_v<K>, false>;
        table_t table(key_data_ptr, value_data_ptr, keys_data.dim(0));
        tv::cuda::Launch launcher(table.size(), custream);
        launcher(tv::hash::assign_arange_split<table_t, Kunsigned>, table, count_ptr);
      }
      else if (values_data.dtype() == tv::DType(10)){
        using V = uint32_t;
        V* value_data_ptr = reinterpret_cast<V*>(values_data.raw_data());
        using table_t =
            tv::hash::LinearHashTableSplit<K, V, tv::hash::Murmur3Hash<K>,
                                        tv::hash::default_empty_key_v<K>, false>;
        table_t table(key_data_ptr, value_data_ptr, keys_data.dim(0));
        tv::cuda::Launch launcher(table.size(), custream);
        launcher(tv::hash::assign_arange_split<table_t, Kunsigned>, table, count_ptr);
      }
      else if (values_data.dtype() == tv::DType(11)){
        using V = uint64_t;
        V* value_data_ptr = reinterpret_cast<V*>(values_data.raw_data());
        using table_t =
            tv::hash::LinearHashTableSplit<K, V, tv::hash::Murmur3Hash<K>,
                                        tv::hash::default_empty_key_v<K>, false>;
        table_t table(key_data_ptr, value_data_ptr, keys_data.dim(0));
        tv::cuda::Launch launcher(table.size(), custream);
        launcher(tv::hash::assign_arange_split<table_t, Kunsigned>, table, count_ptr);
      }
      else{
        TV_THROW_RT_ERR("unknown dtype values_data.dtype(), available: [int32_t, int64_t, uint32_t, uint64_t]")
      }
    }
    else if (keys_data.dtype() == tv::DType(8)){
      using K = int64_t;
      using Kunsigned = tv::hash::itemsize_to_unsigned_t<sizeof(K)>;
      auto count_ptr = count.data_ptr<Kunsigned>();
      K* key_data_ptr = reinterpret_cast<K*>(keys_data.raw_data());
      if (values_data.dtype() == tv::DType(1)){
        using V = int32_t;
        V* value_data_ptr = reinterpret_cast<V*>(values_data.raw_data());
        using table_t =
            tv::hash::LinearHashTableSplit<K, V, tv::hash::Murmur3Hash<K>,
                                        tv::hash::default_empty_key_v<K>, false>;
        table_t table(key_data_ptr, value_data_ptr, keys_data.dim(0));
        tv::cuda::Launch launcher(table.size(), custream);
        launcher(tv::hash::assign_arange_split<table_t, Kunsigned>, table, count_ptr);
      }
      else if (values_data.dtype() == tv::DType(8)){
        using V = int64_t;
        V* value_data_ptr = reinterpret_cast<V*>(values_data.raw_data());
        using table_t =
            tv::hash::LinearHashTableSplit<K, V, tv::hash::Murmur3Hash<K>,
                                        tv::hash::default_empty_key_v<K>, false>;
        table_t table(key_data_ptr, value_data_ptr, keys_data.dim(0));
        tv::cuda::Launch launcher(table.size(), custream);
        launcher(tv::hash::assign_arange_split<table_t, Kunsigned>, table, count_ptr);
      }
      else if (values_data.dtype() == tv::DType(10)){
        using V = uint32_t;
        V* value_data_ptr = reinterpret_cast<V*>(values_data.raw_data());
        using table_t =
            tv::hash::LinearHashTableSplit<K, V, tv::hash::Murmur3Hash<K>,
                                        tv::hash::default_empty_key_v<K>, false>;
        table_t table(key_data_ptr, value_data_ptr, keys_data.dim(0));
        tv::cuda::Launch launcher(table.size(), custream);
        launcher(tv::hash::assign_arange_split<table_t, Kunsigned>, table, count_ptr);
      }
      else if (values_data.dtype() == tv::DType(11)){
        using V = uint64_t;
        V* value_data_ptr = reinterpret_cast<V*>(values_data.raw_data());
        using table_t =
            tv::hash::LinearHashTableSplit<K, V, tv::hash::Murmur3Hash<K>,
                                        tv::hash::default_empty_key_v<K>, false>;
        table_t table(key_data_ptr, value_data_ptr, keys_data.dim(0));
        tv::cuda::Launch launcher(table.size(), custream);
        launcher(tv::hash::assign_arange_split<table_t, Kunsigned>, table, count_ptr);
      }
      else{
        TV_THROW_RT_ERR("unknown dtype values_data.dtype(), available: [int32_t, int64_t, uint32_t, uint64_t]")
      }
    }
    else if (keys_data.dtype() == tv::DType(10)){
      using K = uint32_t;
      using Kunsigned = tv::hash::itemsize_to_unsigned_t<sizeof(K)>;
      auto count_ptr = count.data_ptr<Kunsigned>();
      K* key_data_ptr = reinterpret_cast<K*>(keys_data.raw_data());
      if (values_data.dtype() == tv::DType(1)){
        using V = int32_t;
        V* value_data_ptr = reinterpret_cast<V*>(values_data.raw_data());
        using table_t =
            tv::hash::LinearHashTableSplit<K, V, tv::hash::Murmur3Hash<K>,
                                        tv::hash::default_empty_key_v<K>, false>;
        table_t table(key_data_ptr, value_data_ptr, keys_data.dim(0));
        tv::cuda::Launch launcher(table.size(), custream);
        launcher(tv::hash::assign_arange_split<table_t, Kunsigned>, table, count_ptr);
      }
      else if (values_data.dtype() == tv::DType(8)){
        using V = int64_t;
        V* value_data_ptr = reinterpret_cast<V*>(values_data.raw_data());
        using table_t =
            tv::hash::LinearHashTableSplit<K, V, tv::hash::Murmur3Hash<K>,
                                        tv::hash::default_empty_key_v<K>, false>;
        table_t table(key_data_ptr, value_data_ptr, keys_data.dim(0));
        tv::cuda::Launch launcher(table.size(), custream);
        launcher(tv::hash::assign_arange_split<table_t, Kunsigned>, table, count_ptr);
      }
      else if (values_data.dtype() == tv::DType(10)){
        using V = uint32_t;
        V* value_data_ptr = reinterpret_cast<V*>(values_data.raw_data());
        using table_t =
            tv::hash::LinearHashTableSplit<K, V, tv::hash::Murmur3Hash<K>,
                                        tv::hash::default_empty_key_v<K>, false>;
        table_t table(key_data_ptr, value_data_ptr, keys_data.dim(0));
        tv::cuda::Launch launcher(table.size(), custream);
        launcher(tv::hash::assign_arange_split<table_t, Kunsigned>, table, count_ptr);
      }
      else if (values_data.dtype() == tv::DType(11)){
        using V = uint64_t;
        V* value_data_ptr = reinterpret_cast<V*>(values_data.raw_data());
        using table_t =
            tv::hash::LinearHashTableSplit<K, V, tv::hash::Murmur3Hash<K>,
                                        tv::hash::default_empty_key_v<K>, false>;
        table_t table(key_data_ptr, value_data_ptr, keys_data.dim(0));
        tv::cuda::Launch launcher(table.size(), custream);
        launcher(tv::hash::assign_arange_split<table_t, Kunsigned>, table, count_ptr);
      }
      else{
        TV_THROW_RT_ERR("unknown dtype values_data.dtype(), available: [int32_t, int64_t, uint32_t, uint64_t]")
      }
    }
    else if (keys_data.dtype() == tv::DType(11)){
      using K = uint64_t;
      using Kunsigned = tv::hash::itemsize_to_unsigned_t<sizeof(K)>;
      auto count_ptr = count.data_ptr<Kunsigned>();
      K* key_data_ptr = reinterpret_cast<K*>(keys_data.raw_data());
      if (values_data.dtype() == tv::DType(1)){
        using V = int32_t;
        V* value_data_ptr = reinterpret_cast<V*>(values_data.raw_data());
        using table_t =
            tv::hash::LinearHashTableSplit<K, V, tv::hash::Murmur3Hash<K>,
                                        tv::hash::default_empty_key_v<K>, false>;
        table_t table(key_data_ptr, value_data_ptr, keys_data.dim(0));
        tv::cuda::Launch launcher(table.size(), custream);
        launcher(tv::hash::assign_arange_split<table_t, Kunsigned>, table, count_ptr);
      }
      else if (values_data.dtype() == tv::DType(8)){
        using V = int64_t;
        V* value_data_ptr = reinterpret_cast<V*>(values_data.raw_data());
        using table_t =
            tv::hash::LinearHashTableSplit<K, V, tv::hash::Murmur3Hash<K>,
                                        tv::hash::default_empty_key_v<K>, false>;
        table_t table(key_data_ptr, value_data_ptr, keys_data.dim(0));
        tv::cuda::Launch launcher(table.size(), custream);
        launcher(tv::hash::assign_arange_split<table_t, Kunsigned>, table, count_ptr);
      }
      else if (values_data.dtype() == tv::DType(10)){
        using V = uint32_t;
        V* value_data_ptr = reinterpret_cast<V*>(values_data.raw_data());
        using table_t =
            tv::hash::LinearHashTableSplit<K, V, tv::hash::Murmur3Hash<K>,
                                        tv::hash::default_empty_key_v<K>, false>;
        table_t table(key_data_ptr, value_data_ptr, keys_data.dim(0));
        tv::cuda::Launch launcher(table.size(), custream);
        launcher(tv::hash::assign_arange_split<table_t, Kunsigned>, table, count_ptr);
      }
      else if (values_data.dtype() == tv::DType(11)){
        using V = uint64_t;
        V* value_data_ptr = reinterpret_cast<V*>(values_data.raw_data());
        using table_t =
            tv::hash::LinearHashTableSplit<K, V, tv::hash::Murmur3Hash<K>,
                                        tv::hash::default_empty_key_v<K>, false>;
        table_t table(key_data_ptr, value_data_ptr, keys_data.dim(0));
        tv::cuda::Launch launcher(table.size(), custream);
        launcher(tv::hash::assign_arange_split<table_t, Kunsigned>, table, count_ptr);
      }
      else{
        TV_THROW_RT_ERR("unknown dtype values_data.dtype(), available: [int32_t, int64_t, uint32_t, uint64_t]")
      }
    }
    else{
      TV_THROW_RT_ERR("unknown dtype keys_data.dtype(), available: [int32_t, int64_t, uint32_t, uint64_t]")
    }
  }
}
} // namespace core
} // namespace hash
} // namespace csrc
} // namespace spconv
} // namespace spconvlib