#include <spconvlib/spconv/csrc/hash/core/HashTable.h>
#include <spconvlib/cumm/common/TensorViewHashKernel.h>
namespace spconvlib {
namespace spconv {
namespace csrc {
namespace hash {
namespace core {
using TensorView = spconvlib::cumm::common::TensorView;
using TslRobinMap = spconvlib::cumm::common::TslRobinMap;
using TensorViewHashKernel = spconvlib::cumm::common::TensorViewHashKernel;
void HashTable::query(tv::Tensor keys, tv::Tensor values, tv::Tensor is_empty, std::uintptr_t stream)   {
  
  auto N = keys.dim(0);
  TV_ASSERT_RT_ERR(keys.itemsize() == key_itemsize_, "keys itemsize not equal to", key_itemsize_);
  TV_ASSERT_RT_ERR(values.itemsize() == value_itemsize_, "values itemsize not equal to", value_itemsize_);
  TV_ASSERT_RT_ERR(N == values.dim(0) && is_empty.dim(0) == N, "number of key and value must same");
  auto is_empty_ptr = is_empty.data_ptr<uint8_t>();
  if (!is_cpu){
      TV_ASSERT_RT_ERR(keys.dtype() == keys_data.dtype(), "keys dtype not equal to", keys_data.dtype());
  }
  if (is_cpu){
    {
      bool found = false;
      if (key_itemsize_ == 4 && value_itemsize_ == 4){
        auto& cpu_map = map_4_4;
        auto k_ptr = reinterpret_cast<uint32_t*>(keys.raw_data());
        auto v_ptr = reinterpret_cast<uint32_t*>(values.raw_data());
        tv::kernel_1d_cpu(keys.device(), N, [&](size_t begin, size_t end, size_t step){
            bool emp;
            for (size_t i = begin; i < end; i += step){
                auto iter = cpu_map.find(k_ptr[i]);
                emp = iter == cpu_map.end();
                if (!emp){
                    v_ptr[i] = iter->second;
                }
                is_empty_ptr[i] = uint8_t(emp);
            }
        });
        found = true;
      }
      if (key_itemsize_ == 4 && value_itemsize_ == 8){
        auto& cpu_map = map_4_8;
        auto k_ptr = reinterpret_cast<uint32_t*>(keys.raw_data());
        auto v_ptr = reinterpret_cast<uint64_t*>(values.raw_data());
        tv::kernel_1d_cpu(keys.device(), N, [&](size_t begin, size_t end, size_t step){
            bool emp;
            for (size_t i = begin; i < end; i += step){
                auto iter = cpu_map.find(k_ptr[i]);
                emp = iter == cpu_map.end();
                if (!emp){
                    v_ptr[i] = iter->second;
                }
                is_empty_ptr[i] = uint8_t(emp);
            }
        });
        found = true;
      }
      if (key_itemsize_ == 8 && value_itemsize_ == 4){
        auto& cpu_map = map_8_4;
        auto k_ptr = reinterpret_cast<uint64_t*>(keys.raw_data());
        auto v_ptr = reinterpret_cast<uint32_t*>(values.raw_data());
        tv::kernel_1d_cpu(keys.device(), N, [&](size_t begin, size_t end, size_t step){
            bool emp;
            for (size_t i = begin; i < end; i += step){
                auto iter = cpu_map.find(k_ptr[i]);
                emp = iter == cpu_map.end();
                if (!emp){
                    v_ptr[i] = iter->second;
                }
                is_empty_ptr[i] = uint8_t(emp);
            }
        });
        found = true;
      }
      if (key_itemsize_ == 8 && value_itemsize_ == 8){
        auto& cpu_map = map_8_8;
        auto k_ptr = reinterpret_cast<uint64_t*>(keys.raw_data());
        auto v_ptr = reinterpret_cast<uint64_t*>(values.raw_data());
        tv::kernel_1d_cpu(keys.device(), N, [&](size_t begin, size_t end, size_t step){
            bool emp;
            for (size_t i = begin; i < end; i += step){
                auto iter = cpu_map.find(k_ptr[i]);
                emp = iter == cpu_map.end();
                if (!emp){
                    v_ptr[i] = iter->second;
                }
                is_empty_ptr[i] = uint8_t(emp);
            }
        });
        found = true;
      }
      TV_ASSERT_RT_ERR(found, "suitable hash table not found.");
    }
  }
  else{
    auto custream = reinterpret_cast<hipStream_t>(stream);
    if (keys_data.dtype() == tv::DType(1)){
      using K = int32_t;
      K* key_data_ptr = reinterpret_cast<K*>(keys_data.raw_data());
      K* key_ptr = reinterpret_cast<K*>(keys.raw_data());
      if (values_data.itemsize() == 4){
        using V = tv::hash::itemsize_to_unsigned_t<4>;
        V* value_data_ptr = reinterpret_cast<V*>(values_data.raw_data());
        V* value_ptr = reinterpret_cast<V*>(values.raw_data());
        using table_t =
            tv::hash::LinearHashTableSplit<K, V, tv::hash::Murmur3Hash<K>,
                                        tv::hash::default_empty_key_v<K>, false>;
        tv::cuda::Launch launcher(N, custream);
        table_t table(key_data_ptr, value_data_ptr, keys_data.dim(0));
        launcher(tv::hash::query_split<table_t>, table, key_ptr, value_ptr, is_empty_ptr, size_t(N));
      }
      else if (values_data.itemsize() == 8){
        using V = tv::hash::itemsize_to_unsigned_t<8>;
        V* value_data_ptr = reinterpret_cast<V*>(values_data.raw_data());
        V* value_ptr = reinterpret_cast<V*>(values.raw_data());
        using table_t =
            tv::hash::LinearHashTableSplit<K, V, tv::hash::Murmur3Hash<K>,
                                        tv::hash::default_empty_key_v<K>, false>;
        tv::cuda::Launch launcher(N, custream);
        table_t table(key_data_ptr, value_data_ptr, keys_data.dim(0));
        launcher(tv::hash::query_split<table_t>, table, key_ptr, value_ptr, is_empty_ptr, size_t(N));
      }
      else{
        TV_THROW_RT_ERR("unknown val values_data.itemsize(), available: [4, 8]")
      }
    }
    else if (keys_data.dtype() == tv::DType(8)){
      using K = int64_t;
      K* key_data_ptr = reinterpret_cast<K*>(keys_data.raw_data());
      K* key_ptr = reinterpret_cast<K*>(keys.raw_data());
      if (values_data.itemsize() == 4){
        using V = tv::hash::itemsize_to_unsigned_t<4>;
        V* value_data_ptr = reinterpret_cast<V*>(values_data.raw_data());
        V* value_ptr = reinterpret_cast<V*>(values.raw_data());
        using table_t =
            tv::hash::LinearHashTableSplit<K, V, tv::hash::Murmur3Hash<K>,
                                        tv::hash::default_empty_key_v<K>, false>;
        tv::cuda::Launch launcher(N, custream);
        table_t table(key_data_ptr, value_data_ptr, keys_data.dim(0));
        launcher(tv::hash::query_split<table_t>, table, key_ptr, value_ptr, is_empty_ptr, size_t(N));
      }
      else if (values_data.itemsize() == 8){
        using V = tv::hash::itemsize_to_unsigned_t<8>;
        V* value_data_ptr = reinterpret_cast<V*>(values_data.raw_data());
        V* value_ptr = reinterpret_cast<V*>(values.raw_data());
        using table_t =
            tv::hash::LinearHashTableSplit<K, V, tv::hash::Murmur3Hash<K>,
                                        tv::hash::default_empty_key_v<K>, false>;
        tv::cuda::Launch launcher(N, custream);
        table_t table(key_data_ptr, value_data_ptr, keys_data.dim(0));
        launcher(tv::hash::query_split<table_t>, table, key_ptr, value_ptr, is_empty_ptr, size_t(N));
      }
      else{
        TV_THROW_RT_ERR("unknown val values_data.itemsize(), available: [4, 8]")
      }
    }
    else if (keys_data.dtype() == tv::DType(10)){
      using K = uint32_t;
      K* key_data_ptr = reinterpret_cast<K*>(keys_data.raw_data());
      K* key_ptr = reinterpret_cast<K*>(keys.raw_data());
      if (values_data.itemsize() == 4){
        using V = tv::hash::itemsize_to_unsigned_t<4>;
        V* value_data_ptr = reinterpret_cast<V*>(values_data.raw_data());
        V* value_ptr = reinterpret_cast<V*>(values.raw_data());
        using table_t =
            tv::hash::LinearHashTableSplit<K, V, tv::hash::Murmur3Hash<K>,
                                        tv::hash::default_empty_key_v<K>, false>;
        tv::cuda::Launch launcher(N, custream);
        table_t table(key_data_ptr, value_data_ptr, keys_data.dim(0));
        launcher(tv::hash::query_split<table_t>, table, key_ptr, value_ptr, is_empty_ptr, size_t(N));
      }
      else if (values_data.itemsize() == 8){
        using V = tv::hash::itemsize_to_unsigned_t<8>;
        V* value_data_ptr = reinterpret_cast<V*>(values_data.raw_data());
        V* value_ptr = reinterpret_cast<V*>(values.raw_data());
        using table_t =
            tv::hash::LinearHashTableSplit<K, V, tv::hash::Murmur3Hash<K>,
                                        tv::hash::default_empty_key_v<K>, false>;
        tv::cuda::Launch launcher(N, custream);
        table_t table(key_data_ptr, value_data_ptr, keys_data.dim(0));
        launcher(tv::hash::query_split<table_t>, table, key_ptr, value_ptr, is_empty_ptr, size_t(N));
      }
      else{
        TV_THROW_RT_ERR("unknown val values_data.itemsize(), available: [4, 8]")
      }
    }
    else if (keys_data.dtype() == tv::DType(11)){
      using K = uint64_t;
      K* key_data_ptr = reinterpret_cast<K*>(keys_data.raw_data());
      K* key_ptr = reinterpret_cast<K*>(keys.raw_data());
      if (values_data.itemsize() == 4){
        using V = tv::hash::itemsize_to_unsigned_t<4>;
        V* value_data_ptr = reinterpret_cast<V*>(values_data.raw_data());
        V* value_ptr = reinterpret_cast<V*>(values.raw_data());
        using table_t =
            tv::hash::LinearHashTableSplit<K, V, tv::hash::Murmur3Hash<K>,
                                        tv::hash::default_empty_key_v<K>, false>;
        tv::cuda::Launch launcher(N, custream);
        table_t table(key_data_ptr, value_data_ptr, keys_data.dim(0));
        launcher(tv::hash::query_split<table_t>, table, key_ptr, value_ptr, is_empty_ptr, size_t(N));
      }
      else if (values_data.itemsize() == 8){
        using V = tv::hash::itemsize_to_unsigned_t<8>;
        V* value_data_ptr = reinterpret_cast<V*>(values_data.raw_data());
        V* value_ptr = reinterpret_cast<V*>(values.raw_data());
        using table_t =
            tv::hash::LinearHashTableSplit<K, V, tv::hash::Murmur3Hash<K>,
                                        tv::hash::default_empty_key_v<K>, false>;
        tv::cuda::Launch launcher(N, custream);
        table_t table(key_data_ptr, value_data_ptr, keys_data.dim(0));
        launcher(tv::hash::query_split<table_t>, table, key_ptr, value_ptr, is_empty_ptr, size_t(N));
      }
      else{
        TV_THROW_RT_ERR("unknown val values_data.itemsize(), available: [4, 8]")
      }
    }
    else{
      TV_THROW_RT_ERR("unknown dtype keys_data.dtype(), available: [int32_t, int64_t, uint32_t, uint64_t]")
    }
  }
}
} // namespace core
} // namespace hash
} // namespace csrc
} // namespace spconv
} // namespace spconvlib