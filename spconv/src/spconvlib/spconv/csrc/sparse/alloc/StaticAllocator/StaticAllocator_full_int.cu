#include <spconvlib/spconv/csrc/sparse/alloc/StaticAllocator.h>
#include <spconvlib/spconv/csrc/sparse/alloc/cudakers/CudaCommonKernel.h>
namespace spconvlib {
namespace spconv {
namespace csrc {
namespace sparse {
namespace alloc {
using TensorView = spconvlib::cumm::common::TensorView;
using ExternalAllocator = spconvlib::spconv::csrc::sparse::alloc::ExternalAllocator;
tv::Tensor StaticAllocator::full_int(std::string name, std::vector<int64_t> shape, int value, int dtype, int device, std::uintptr_t stream, bool is_temp_memory)   {
  
  auto tvctx = tv::Context();
  auto blob = _get_raw_and_check(name, shape, dtype, device, is_temp_memory);
  tvctx.set_cuda_stream(reinterpret_cast<hipStream_t>(stream));
  using ints_t = std::tuple<int32_t, int16_t, int8_t, int64_t, uint32_t, uint64_t, uint16_t, uint8_t>;
  tv::Dispatch<ints_t>()(blob.dtype(), [&](auto I){
      using T = TV_DECLTYPE(I);
      tv::cuda::Launch lanucher_fill(blob.size(), reinterpret_cast<hipStream_t>(stream));
      lanucher_fill(cudakers::fill_kernel<T>, blob.data_ptr<T>(), value, blob.size());
  });
  return blob;
}
} // namespace alloc
} // namespace sparse
} // namespace csrc
} // namespace spconv
} // namespace spconvlib