#include <spconvlib/spconv/csrc/sparse/alloc/StaticAllocator.h>
#include <spconvlib/spconv/csrc/sparse/alloc/cudakers/CudaCommonKernel.h>
namespace spconvlib {
namespace spconv {
namespace csrc {
namespace sparse {
namespace alloc {
using TensorView = spconvlib::cumm::common::TensorView;
using ExternalAllocator = spconvlib::spconv::csrc::sparse::alloc::ExternalAllocator;
tv::Tensor StaticAllocator::full_float(std::string name, std::vector<int64_t> shape, float value, int dtype, int device, std::uintptr_t stream, bool is_temp_memory)   {
  
  auto tvctx = tv::Context();
  auto blob = _get_raw_and_check(name, shape, dtype, device, is_temp_memory);
  tvctx.set_cuda_stream(reinterpret_cast<hipStream_t>(stream));
  using dtypes_t = std::tuple<float, double>;
  tv::Dispatch<dtypes_t>()(blob.dtype(), [&](auto I){
      using T = TV_DECLTYPE(I);
      tv::cuda::Launch lanucher_fill(blob.size(), reinterpret_cast<hipStream_t>(stream));
      lanucher_fill(cudakers::fill_kernel<T>, blob.data_ptr<T>(), value, blob.size());
  });
  return blob;
}
} // namespace alloc
} // namespace sparse
} // namespace csrc
} // namespace spconv
} // namespace spconvlib