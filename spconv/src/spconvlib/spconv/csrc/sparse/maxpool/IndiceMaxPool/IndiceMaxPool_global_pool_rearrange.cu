#include <spconvlib/spconv/csrc/sparse/maxpool/IndiceMaxPool.h>
namespace spconvlib {
namespace spconv {
namespace csrc {
namespace sparse {
namespace maxpool {
using TensorViewKernel = spconvlib::cumm::common::TensorViewKernel;
using TensorView = spconvlib::cumm::common::TensorView;
using GemmBasic = spconvlib::cumm::common::GemmBasic;
using LaunchUtils = spconvlib::spconv::csrc::utils::launch::LaunchUtils;
void IndiceMaxPool::global_pool_rearrange(tv::Tensor out_indices, tv::Tensor coords, tv::Tensor counts, std::uintptr_t stream)   {
  
  auto nhot = coords.dim(0);
  auto cudastream = reinterpret_cast<hipStream_t>(stream);
  tv::cuda::Launch launcher = tv::cuda::Launch(nhot, cudastream);
  launcher(global_pool_rearrange_kernel, out_indices.data_ptr<int>(), 
      coords.data_ptr<const int>(), counts.data_ptr<int>(), nhot, 
      coords.stride(0));
  TV_CHECK_CUDA_ERR_V2("global_pool_feature_rearrange failed!!!");
}
} // namespace maxpool
} // namespace sparse
} // namespace csrc
} // namespace spconv
} // namespace spconvlib