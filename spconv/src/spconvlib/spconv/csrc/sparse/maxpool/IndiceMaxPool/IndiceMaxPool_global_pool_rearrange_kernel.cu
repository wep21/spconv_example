#include "hip/hip_runtime.h"
#include <spconvlib/spconv/csrc/sparse/maxpool/IndiceMaxPool.h>
namespace spconvlib {
namespace spconv {
namespace csrc {
namespace sparse {
namespace maxpool {
using TensorViewKernel = spconvlib::cumm::common::TensorViewKernel;
using TensorView = spconvlib::cumm::common::TensorView;
using GemmBasic = spconvlib::cumm::common::GemmBasic;
using LaunchUtils = spconvlib::spconv::csrc::utils::launch::LaunchUtils;
__global__ void global_pool_rearrange_kernel(int* out_indices, const int* coords, int* counts, int num_indices, int indices_stride)   {
  
  for (int i : tv::KernelLoopX<int>(num_indices)) {
      int batch_idx = coords[i * indices_stride];
      if (batch_idx >= 0){
          auto old = atomicAdd(counts + batch_idx, 1);
          out_indices[batch_idx * num_indices + old] = i;
      }
  }
}
} // namespace maxpool
} // namespace sparse
} // namespace csrc
} // namespace spconv
} // namespace spconvlib