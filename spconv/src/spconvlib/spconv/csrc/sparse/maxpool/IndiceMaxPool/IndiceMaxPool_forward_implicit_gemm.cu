#include "hip/hip_runtime.h"
#include <spconvlib/spconv/csrc/sparse/maxpool/IndiceMaxPool.h>
namespace spconvlib {
namespace spconv {
namespace csrc {
namespace sparse {
namespace maxpool {
using TensorViewKernel = spconvlib::cumm::common::TensorViewKernel;
using TensorView = spconvlib::cumm::common::TensorView;
using GemmBasic = spconvlib::cumm::common::GemmBasic;
using LaunchUtils = spconvlib::spconv::csrc::utils::launch::LaunchUtils;
void IndiceMaxPool::forward_implicit_gemm(tv::Tensor out, tv::Tensor in, tv::Tensor inds, std::uintptr_t stream)   {
  
  auto nhot = out.dim(0);
  tv::check_shape(inds, {-1, nhot});
  tv::check_shape(in, {-1, out.dim(1)});
  auto cudastream = reinterpret_cast<hipStream_t>(stream);
  tv::dispatch<float, double, tv::half_t, tv::bfloat16_t, int8_t>(out.dtype(), [&](auto I){
      using T = TV_DECLTYPE(I);
      auto launchdims = LaunchUtils::get_blocks_threads_of_2d_tensor(nhot, out.dim(1));
      int num_blocks_X = std::get<0>(launchdims);
      int num_blocks_Y = std::get<1>(launchdims);
      dim3 blocks;
      dim3 threads(std::get<2>(launchdims), std::get<3>(launchdims));
      if (num_blocks_Y > kMaxGridYZDim){
          blocks = dim3(num_blocks_X * num_blocks_Y);
      }else{
          blocks = dim3(num_blocks_X, num_blocks_Y);
      }
      tv::cuda::Launch launcher = tv::cuda::Launch(blocks, threads, cudastream);
      T lowest = std::numeric_limits<T>::lowest();
      lowest = T(0);
      tv::dispatch_int<0, 1>(int(num_blocks_Y > kMaxGridYZDim), [&](auto I2){
          constexpr bool OneDim = TV_DECLTYPE(I2)::value == 1;
          launcher(forward_implicit_gemm_kernel<T, OneDim>, out.data_ptr<T>(), in.data_ptr<const T>(),
              inds.data_ptr<const int>(), out.dim(1), inds.dim(0), inds.dim(1), lowest, 
              num_blocks_X, num_blocks_Y);
      });
      TV_CHECK_CUDA_ERR_V2("max pool fwd failed!!!");
  });
}
} // namespace maxpool
} // namespace sparse
} // namespace csrc
} // namespace spconv
} // namespace spconvlib