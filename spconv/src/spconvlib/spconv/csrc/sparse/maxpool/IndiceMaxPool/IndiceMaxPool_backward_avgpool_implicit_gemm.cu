#include <spconvlib/spconv/csrc/sparse/maxpool/IndiceMaxPool.h>
namespace spconvlib {
namespace spconv {
namespace csrc {
namespace sparse {
namespace maxpool {
using TensorViewKernel = spconvlib::cumm::common::TensorViewKernel;
using TensorView = spconvlib::cumm::common::TensorView;
using GemmBasic = spconvlib::cumm::common::GemmBasic;
using LaunchUtils = spconvlib::spconv::csrc::utils::launch::LaunchUtils;
void IndiceMaxPool::backward_avgpool_implicit_gemm(tv::Tensor dout, tv::Tensor din, tv::Tensor inds, tv::Tensor count_out, std::uintptr_t stream)   {
  
  auto nhot = din.dim(0);
  TV_ASSERT_RT_ERR(!count_out.empty(), "count out must not empty")
  tv::check_shape(inds, {-1, nhot});
  tv::check_shape(din, {-1, dout.dim(1)});
  auto cudastream = reinterpret_cast<hipStream_t>(stream);
  tv::dispatch<float, double, tv::half_t, tv::bfloat16_t>(dout.dtype(), [&](auto I){
      using T = TV_DECLTYPE(I);
      auto launchdims = LaunchUtils::get_blocks_threads_of_2d_tensor(nhot, dout.dim(1));
      int num_blocks_X = std::get<0>(launchdims);
      int num_blocks_Y = std::get<1>(launchdims);
      dim3 blocks;
      dim3 threads(std::get<2>(launchdims), std::get<3>(launchdims));
      if (num_blocks_Y > kMaxGridYZDim){
          blocks = dim3(num_blocks_X * num_blocks_Y);
      }else{
          blocks = dim3(num_blocks_X, num_blocks_Y);
      }
      tv::cuda::Launch launcher = tv::cuda::Launch(blocks, threads, cudastream);
      tv::dispatch_int<0, 1>(int(num_blocks_Y > kMaxGridYZDim), [&](auto I2){
          constexpr bool OneDim = TV_DECLTYPE(I2)::value == 1;
          launcher(backward_avgpool_implicit_gemm_kernel<T, OneDim>, 
              dout.data_ptr<const T>(), din.data_ptr<T>(),
              inds.data_ptr<const int>(), count_out.data_ptr<const int>(),
              dout.dim(1), inds.dim(0), inds.dim(1),
              num_blocks_X, num_blocks_Y);
      });
      TV_CHECK_CUDA_ERR_V2("avg pool bwd failed!!!");
  });
}
} // namespace maxpool
} // namespace sparse
} // namespace csrc
} // namespace spconv
} // namespace spconvlib