#include <spconvlib/spconv/csrc/sparse/inference/InferenceOps.h>
#include <spconvlib/spconv/csrc/sparse/inference/ker/InferenceOpsKernel.h>
namespace spconvlib {
namespace spconv {
namespace csrc {
namespace sparse {
namespace inference {
using TensorView = spconvlib::cumm::common::TensorView;
using LaunchUtils = spconvlib::spconv::csrc::utils::launch::LaunchUtils;
void InferenceOps::bias_add_act_inplace(tv::Tensor out, tv::Tensor bias, tv::gemm::Activation act_type, float alpha, float beta, std::uintptr_t stream)   {
  
  auto nhot = out.dim(0);
  auto cudastream = reinterpret_cast<hipStream_t>(stream);
  TV_ASSERT_RT_ERR(bias.dim(0) == out.dim(1), "error");
  tv::dispatch<float, tv::half_t, tv::bfloat16_t>(out.dtype(), [&](auto I){
      using T = TV_DECLTYPE(I);
      auto launchdims = LaunchUtils::get_blocks_threads_of_2d_tensor(nhot, out.dim(1));
      int num_blocks_X = std::get<0>(launchdims);
      int num_blocks_Y = std::get<1>(launchdims);
      dim3 blocks;
      dim3 threads(std::get<2>(launchdims), std::get<3>(launchdims));
      if (num_blocks_Y > kMaxGridYZDim){
          blocks = dim3(num_blocks_X * num_blocks_Y);
      }else{
          blocks = dim3(num_blocks_X, num_blocks_Y);
      }
      tv::cuda::Launch launcher = tv::cuda::Launch(blocks, threads, cudastream);
      tv::dispatch_int<0, 1>(int(num_blocks_Y > kMaxGridYZDim), [&](auto I2){
          constexpr bool OneDim = TV_DECLTYPE(I2)::value == 1;
          if (act_type == tv::gemm::Activation::kNone){
              launcher(ker::bias_add_inplace_kernel<T, OneDim>, out.data_ptr<T>(), bias.data_ptr<const T>(),
                  nhot, out.dim(1), num_blocks_X, num_blocks_Y);
          }else{
              launcher(ker::bias_add_act_inplace_kernel<T, OneDim>, out.data_ptr<T>(), bias.data_ptr<const T>(),
                  act_type, T(alpha), T(beta), nhot, out.dim(1), num_blocks_X, num_blocks_Y);
          }
      });
      TV_CHECK_CUDA_ERR_V2("bias add act failed!!!");
  });
}
} // namespace inference
} // namespace sparse
} // namespace csrc
} // namespace spconv
} // namespace spconvlib