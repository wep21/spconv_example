#include <spconvlib/spconv/csrc/sparse/inference/InferenceOps.h>
#include <spconvlib/spconv/csrc/sparse/inference/ker/InferenceOpsKernel.h>
namespace spconvlib {
namespace spconv {
namespace csrc {
namespace sparse {
namespace inference {
using TensorView = spconvlib::cumm::common::TensorView;
using LaunchUtils = spconvlib::spconv::csrc::utils::launch::LaunchUtils;
void InferenceOps::activation_inplace(tv::Tensor out, tv::gemm::Activation act_type, float alpha, float beta, std::uintptr_t stream)   {
  
  auto nhot = out.size();
  auto cudastream = reinterpret_cast<hipStream_t>(stream);
  tv::cuda::Launch launcher = tv::cuda::Launch(nhot, cudastream);
  tv::dispatch<float, tv::half_t, tv::bfloat16_t>(out.dtype(), [&](auto I){
      using T = TV_DECLTYPE(I);
      launcher(ker::activation_inplace_kernel<T>, out.data_ptr<T>(), act_type, T(alpha), T(beta),
          nhot);
      TV_CHECK_CUDA_ERR_V2("bias add act failed!!!");
  });
}
} // namespace inference
} // namespace sparse
} // namespace csrc
} // namespace spconv
} // namespace spconvlib