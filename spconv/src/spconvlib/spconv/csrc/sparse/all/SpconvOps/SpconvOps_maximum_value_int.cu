#include "hip/hip_runtime.h"
#include <spconvlib/spconv/csrc/sparse/all/SpconvOps.h>
#include <spconvlib/spconv/csrc/sparse/all/cudakers/CudaCommonKernel.h>
namespace spconvlib {
namespace spconv {
namespace csrc {
namespace sparse {
namespace all {
using ThrustCustomAllocatorV2 = spconvlib::spconv::csrc::sparse::all::ThrustCustomAllocatorV2;
using ExternalAllocator = spconvlib::spconv::csrc::sparse::alloc::ExternalAllocator;
using GemmBasicHost = spconvlib::cumm::common::GemmBasicHost;
using ThrustAllocator = spconvlib::spconv::csrc::sparse::alloc::ThrustAllocator;
using Point2Voxel1DCPU = spconvlib::spconv::csrc::sparse::all::ops_cpu1d::Point2VoxelCPU;
using SpconvIndicesCPU1D = spconvlib::spconv::csrc::sparse::all::ops_cpu1d::SparseConvIndicesCPU;
using Point2Voxel1D = spconvlib::spconv::csrc::sparse::all::ops1d::Point2Voxel;
using Point2Voxel2DCPU = spconvlib::spconv::csrc::sparse::all::ops_cpu2d::Point2VoxelCPU;
using SpconvIndicesCPU2D = spconvlib::spconv::csrc::sparse::all::ops_cpu2d::SparseConvIndicesCPU;
using Point2Voxel2D = spconvlib::spconv::csrc::sparse::all::ops2d::Point2Voxel;
using Point2Voxel3DCPU = spconvlib::spconv::csrc::sparse::all::ops_cpu3d::Point2VoxelCPU;
using SpconvIndicesCPU3D = spconvlib::spconv::csrc::sparse::all::ops_cpu3d::SparseConvIndicesCPU;
using Point2Voxel3D = spconvlib::spconv::csrc::sparse::all::ops3d::Point2Voxel;
using Point2Voxel4DCPU = spconvlib::spconv::csrc::sparse::all::ops_cpu4d::Point2VoxelCPU;
using SpconvIndicesCPU4D = spconvlib::spconv::csrc::sparse::all::ops_cpu4d::SparseConvIndicesCPU;
using Point2Voxel4D = spconvlib::spconv::csrc::sparse::all::ops4d::Point2Voxel;
void SpconvOps::maximum_value_int(tv::Tensor data, int value, std::uintptr_t stream_int)   {
  
  auto size = data.size();
  using ints_t = std::tuple<int32_t, int16_t, int8_t, int64_t, uint32_t, uint64_t, uint16_t, uint8_t>;
  tv::Dispatch<ints_t>()(data.dtype(), [&](auto I){
    using T = TV_DECLTYPE(I);
    auto ptr = data.data_ptr<T>();
    if (data.is_cpu()){
      for (int i = 0; i < size; ++i){
          ptr[i] = std::max(ptr[i], T(value));
      }
    }
    else{
      tv::cuda::Launch lanucher(size, reinterpret_cast<hipStream_t>(stream_int));
      lanucher(cudakers::maximum_value_kernel<T>, ptr, value, size);
    }
  });
}
} // namespace all
} // namespace sparse
} // namespace csrc
} // namespace spconv
} // namespace spconvlib