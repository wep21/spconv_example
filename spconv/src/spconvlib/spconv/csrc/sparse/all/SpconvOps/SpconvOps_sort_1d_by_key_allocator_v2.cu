#include <spconvlib/spconv/csrc/sparse/all/SpconvOps.h>
#include <spconvlib/spconv/csrc/sparse/all/CustomThrustLib.h>
#include <spconvlib/cumm/common/TensorViewKernel.h>
#include <spconvlib/spconv/csrc/sparse/all/cudakers/CudaCommonKernel.h>
namespace spconvlib {
namespace spconv {
namespace csrc {
namespace sparse {
namespace all {
using ThrustCustomAllocatorV2 = spconvlib::spconv::csrc::sparse::all::ThrustCustomAllocatorV2;
using ExternalAllocator = spconvlib::spconv::csrc::sparse::alloc::ExternalAllocator;
using GemmBasicHost = spconvlib::cumm::common::GemmBasicHost;
using ThrustAllocator = spconvlib::spconv::csrc::sparse::alloc::ThrustAllocator;
using Point2Voxel1DCPU = spconvlib::spconv::csrc::sparse::all::ops_cpu1d::Point2VoxelCPU;
using SpconvIndicesCPU1D = spconvlib::spconv::csrc::sparse::all::ops_cpu1d::SparseConvIndicesCPU;
using Point2Voxel1D = spconvlib::spconv::csrc::sparse::all::ops1d::Point2Voxel;
using Point2Voxel2DCPU = spconvlib::spconv::csrc::sparse::all::ops_cpu2d::Point2VoxelCPU;
using SpconvIndicesCPU2D = spconvlib::spconv::csrc::sparse::all::ops_cpu2d::SparseConvIndicesCPU;
using Point2Voxel2D = spconvlib::spconv::csrc::sparse::all::ops2d::Point2Voxel;
using Point2Voxel3DCPU = spconvlib::spconv::csrc::sparse::all::ops_cpu3d::Point2VoxelCPU;
using SpconvIndicesCPU3D = spconvlib::spconv::csrc::sparse::all::ops_cpu3d::SparseConvIndicesCPU;
using Point2Voxel3D = spconvlib::spconv::csrc::sparse::all::ops3d::Point2Voxel;
using Point2Voxel4DCPU = spconvlib::spconv::csrc::sparse::all::ops_cpu4d::Point2VoxelCPU;
using SpconvIndicesCPU4D = spconvlib::spconv::csrc::sparse::all::ops_cpu4d::SparseConvIndicesCPU;
using Point2Voxel4D = spconvlib::spconv::csrc::sparse::all::ops4d::Point2Voxel;
using CustomThrustLib = spconvlib::spconv::csrc::sparse::all::CustomThrustLib;
using TensorViewKernel = spconvlib::cumm::common::TensorViewKernel;
tv::Tensor SpconvOps::sort_1d_by_key_allocator_v2(tv::Tensor data, ThrustAllocator& allocator, tv::Tensor indices, std::uintptr_t stream, int mask_count, bool do_sort)   {
  
  hipStream_t stream_cu = reinterpret_cast<hipStream_t>(stream);
  if (indices.empty()){
      indices = tv::empty({data.dim(0)}, tv::int32, 0);
  }
  tv::cuda::Launch launcher(data.dim(0), stream_cu);
  launcher(cudakers::arange_kernel<int32_t>, indices.data_ptr<int32_t>(), indices.dim(0));
  if (!do_sort){
      return indices;
  }
  // auto timer = tv::CUDATimer();
  if (data.dtype() == tv::DType(1)){
    using T_ = int32_t;
    tv::dispatch_int<1, 2, 3, 4>(mask_count, [&](auto IV){
        constexpr int I = TV_DECLTYPE(IV)::value;
        // we can't use thrust::tuple in mp_repeat_c directly because
        // thrust tuple actually has fixed size template arguments.
        using T = tv::mp_rename<tv::mp_repeat_c<tv::mp_list<T_>, I>, thrust::tuple>;
        thrust::device_ptr<T> ptr_tr(reinterpret_cast<T*>(data.data_ptr<T_>()));
        thrust::device_ptr<int32_t> ptr_k(indices.data_ptr<int32_t>());
        auto thrust_ctx = thrust::cuda::par.on(stream_cu);
        auto ctx2 = thrust::cuda::par(allocator).on(stream_cu);
        thrust::sort_by_key(ctx2, ptr_tr, ptr_tr + data.dim(0), ptr_k);
    });
  }
  else if (data.dtype() == tv::DType(8)){
    using T_ = int64_t;
    tv::dispatch_int<1, 2, 3, 4>(mask_count, [&](auto IV){
        constexpr int I = TV_DECLTYPE(IV)::value;
        // we can't use thrust::tuple in mp_repeat_c directly because
        // thrust tuple actually has fixed size template arguments.
        using T = tv::mp_rename<tv::mp_repeat_c<tv::mp_list<T_>, I>, thrust::tuple>;
        thrust::device_ptr<T> ptr_tr(reinterpret_cast<T*>(data.data_ptr<T_>()));
        thrust::device_ptr<int32_t> ptr_k(indices.data_ptr<int32_t>());
        auto thrust_ctx = thrust::cuda::par.on(stream_cu);
        auto ctx2 = thrust::cuda::par(allocator).on(stream_cu);
        thrust::sort_by_key(ctx2, ptr_tr, ptr_tr + data.dim(0), ptr_k);
    });
  }
  else if (data.dtype() == tv::DType(10)){
    using T_ = uint32_t;
    tv::dispatch_int<1, 2, 3, 4>(mask_count, [&](auto IV){
        constexpr int I = TV_DECLTYPE(IV)::value;
        // we can't use thrust::tuple in mp_repeat_c directly because
        // thrust tuple actually has fixed size template arguments.
        using T = tv::mp_rename<tv::mp_repeat_c<tv::mp_list<T_>, I>, thrust::tuple>;
        thrust::device_ptr<T> ptr_tr(reinterpret_cast<T*>(data.data_ptr<T_>()));
        thrust::device_ptr<int32_t> ptr_k(indices.data_ptr<int32_t>());
        auto thrust_ctx = thrust::cuda::par.on(stream_cu);
        auto ctx2 = thrust::cuda::par(allocator).on(stream_cu);
        thrust::sort_by_key(ctx2, ptr_tr, ptr_tr + data.dim(0), ptr_k);
    });
  }
  else if (data.dtype() == tv::DType(11)){
    using T_ = uint64_t;
    tv::dispatch_int<1, 2, 3, 4>(mask_count, [&](auto IV){
        constexpr int I = TV_DECLTYPE(IV)::value;
        // we can't use thrust::tuple in mp_repeat_c directly because
        // thrust tuple actually has fixed size template arguments.
        using T = tv::mp_rename<tv::mp_repeat_c<tv::mp_list<T_>, I>, thrust::tuple>;
        thrust::device_ptr<T> ptr_tr(reinterpret_cast<T*>(data.data_ptr<T_>()));
        thrust::device_ptr<int32_t> ptr_k(indices.data_ptr<int32_t>());
        auto thrust_ctx = thrust::cuda::par.on(stream_cu);
        auto ctx2 = thrust::cuda::par(allocator).on(stream_cu);
        thrust::sort_by_key(ctx2, ptr_tr, ptr_tr + data.dim(0), ptr_k);
    });
  }
  else{
    TV_THROW_RT_ERR("unknown dtype data.dtype(), available: [int32_t, int64_t, uint32_t, uint64_t]")
  }
  // tv::ssprint("SORT BY KEY TIME", data.dim(0), timer.report() / 1000.0);
  return indices;
}
} // namespace all
} // namespace sparse
} // namespace csrc
} // namespace spconv
} // namespace spconvlib