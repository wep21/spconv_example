#include "hip/hip_runtime.h"
#include <spconvlib/spconv/csrc/sparse/all/SpconvOps.h>
#include <spconvlib/spconv/csrc/sparse/all/CustomThrustLib.h>
#include <spconvlib/cumm/common/TensorViewKernel.h>
#include <spconvlib/spconv/csrc/sparse/all/cudakers/CudaCommonKernel.h>

        template <typename T> struct SmallOrEqualTo {
            TV_HOST_DEVICE_INLINE T operator()(const T &x, const T &y) const {
                return x < y;
            }
        };
        template <typename T> __global__ void mask_input(T* inp, T mask, int size){
            for (int i : tv::KernelLoopX<int>(size)){
                inp[i] &= mask;
            }
        }
        
namespace spconvlib {
namespace spconv {
namespace csrc {
namespace sparse {
namespace all {
using ThrustCustomAllocatorV2 = spconvlib::spconv::csrc::sparse::all::ThrustCustomAllocatorV2;
using ExternalAllocator = spconvlib::spconv::csrc::sparse::alloc::ExternalAllocator;
using GemmBasicHost = spconvlib::cumm::common::GemmBasicHost;
using ThrustAllocator = spconvlib::spconv::csrc::sparse::alloc::ThrustAllocator;
using Point2Voxel1DCPU = spconvlib::spconv::csrc::sparse::all::ops_cpu1d::Point2VoxelCPU;
using SpconvIndicesCPU1D = spconvlib::spconv::csrc::sparse::all::ops_cpu1d::SparseConvIndicesCPU;
using Point2Voxel1D = spconvlib::spconv::csrc::sparse::all::ops1d::Point2Voxel;
using Point2Voxel2DCPU = spconvlib::spconv::csrc::sparse::all::ops_cpu2d::Point2VoxelCPU;
using SpconvIndicesCPU2D = spconvlib::spconv::csrc::sparse::all::ops_cpu2d::SparseConvIndicesCPU;
using Point2Voxel2D = spconvlib::spconv::csrc::sparse::all::ops2d::Point2Voxel;
using Point2Voxel3DCPU = spconvlib::spconv::csrc::sparse::all::ops_cpu3d::Point2VoxelCPU;
using SpconvIndicesCPU3D = spconvlib::spconv::csrc::sparse::all::ops_cpu3d::SparseConvIndicesCPU;
using Point2Voxel3D = spconvlib::spconv::csrc::sparse::all::ops3d::Point2Voxel;
using Point2Voxel4DCPU = spconvlib::spconv::csrc::sparse::all::ops_cpu4d::Point2VoxelCPU;
using SpconvIndicesCPU4D = spconvlib::spconv::csrc::sparse::all::ops_cpu4d::SparseConvIndicesCPU;
using Point2Voxel4D = spconvlib::spconv::csrc::sparse::all::ops4d::Point2Voxel;
using CustomThrustLib = spconvlib::spconv::csrc::sparse::all::CustomThrustLib;
using TensorViewKernel = spconvlib::cumm::common::TensorViewKernel;
tv::Tensor SpconvOps::sort_1d_by_key_allocator_v2(tv::Tensor data, ThrustAllocator& allocator, tv::Tensor indices, std::uintptr_t stream)   {
  
  hipStream_t stream_cu = reinterpret_cast<hipStream_t>(stream);
  if (indices.empty()){
      indices = tv::empty({data.dim(0)}, tv::int32, 0);
  }
  tv::cuda::Launch launcher(data.dim(0), stream_cu);
  launcher(cudakers::arange_kernel<int32_t>, indices.data_ptr<int32_t>(), indices.dim(0));
  // auto timer = tv::CUDATimer();
  tv::dispatch<int32_t, uint32_t, int64_t, uint64_t>(data.dtype(), [&](auto I){
      using T = TV_DECLTYPE(I);
      thrust::device_ptr<T> ptr_tr(data.data_ptr<T>());
      thrust::device_ptr<int32_t> ptr_k(indices.data_ptr<int32_t>());
      auto thrust_ctx = thrust::cuda::par.on(stream_cu);
      auto ctx2 = thrust::cuda::par(allocator).on(stream_cu);
      thrust::sort_by_key(ctx2, ptr_tr, ptr_tr + data.dim(0), ptr_k);
  });
  // tv::ssprint("SORT BY KEY TIME", data.dim(0), timer.report() / 1000.0);
  return indices;
}
} // namespace all
} // namespace sparse
} // namespace csrc
} // namespace spconv
} // namespace spconvlib