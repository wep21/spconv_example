#include "hip/hip_runtime.h"
#include <spconvlib/spconv/csrc/sparse/all/SpconvOps.h>
#include <spconvlib/spconv/csrc/sparse/maxpool/IndiceMaxPool.h>
#include <spconvlib/spconv/csrc/sparse/maxpool/IndiceMaxPoolCPU.h>
namespace spconvlib {
namespace spconv {
namespace csrc {
namespace sparse {
namespace all {
using ThrustCustomAllocatorV2 = spconvlib::spconv::csrc::sparse::all::ThrustCustomAllocatorV2;
using ExternalAllocator = spconvlib::spconv::csrc::sparse::alloc::ExternalAllocator;
using GemmBasicHost = spconvlib::cumm::common::GemmBasicHost;
using ThrustAllocator = spconvlib::spconv::csrc::sparse::alloc::ThrustAllocator;
using Point2Voxel1DCPU = spconvlib::spconv::csrc::sparse::all::ops_cpu1d::Point2VoxelCPU;
using SpconvIndicesCPU1D = spconvlib::spconv::csrc::sparse::all::ops_cpu1d::SparseConvIndicesCPU;
using Point2Voxel1D = spconvlib::spconv::csrc::sparse::all::ops1d::Point2Voxel;
using Point2Voxel2DCPU = spconvlib::spconv::csrc::sparse::all::ops_cpu2d::Point2VoxelCPU;
using SpconvIndicesCPU2D = spconvlib::spconv::csrc::sparse::all::ops_cpu2d::SparseConvIndicesCPU;
using Point2Voxel2D = spconvlib::spconv::csrc::sparse::all::ops2d::Point2Voxel;
using Point2Voxel3DCPU = spconvlib::spconv::csrc::sparse::all::ops_cpu3d::Point2VoxelCPU;
using SpconvIndicesCPU3D = spconvlib::spconv::csrc::sparse::all::ops_cpu3d::SparseConvIndicesCPU;
using Point2Voxel3D = spconvlib::spconv::csrc::sparse::all::ops3d::Point2Voxel;
using Point2Voxel4DCPU = spconvlib::spconv::csrc::sparse::all::ops_cpu4d::Point2VoxelCPU;
using SpconvIndicesCPU4D = spconvlib::spconv::csrc::sparse::all::ops_cpu4d::SparseConvIndicesCPU;
using Point2Voxel4D = spconvlib::spconv::csrc::sparse::all::ops4d::Point2Voxel;
using IndiceMaxPool = spconvlib::spconv::csrc::sparse::maxpool::IndiceMaxPool;
using IndiceMaxPoolCPU = spconvlib::spconv::csrc::sparse::maxpool::IndiceMaxPoolCPU;
void SpconvOps::indice_maxpool(tv::Tensor out_features, tv::Tensor features, tv::Tensor indice_pairs, tv::Tensor indice_pair_num, int num_activate_out, std::uintptr_t stream)   {
  
  tv::check_shape(out_features, {-1, features.dim(1)});
  auto indice_pair_num_cpu = indice_pair_num.cpu();
  auto indice_pair_num_cpu_ptr = indice_pair_num_cpu.data_ptr<int>();
  for (int i = 0; i < indice_pair_num.dim(0); ++i){
    int nhot = indice_pair_num_cpu_ptr[i];
    nhot = std::min(nhot, int(indice_pairs.dim(2)));
    if (nhot <= 0){
        continue;
    }
    auto inp_indices = indice_pairs[0][i].slice_first_axis(0, nhot);
    auto out_indices = indice_pairs[1][i].slice_first_axis(0, nhot);
    if (features.is_cpu()){
        IndiceMaxPoolCPU::forward(out_features, features, out_indices, inp_indices);
    }
    else{
      IndiceMaxPool::forward(out_features, features, out_indices, inp_indices, stream);
    }
  }
}
} // namespace all
} // namespace sparse
} // namespace csrc
} // namespace spconv
} // namespace spconvlib