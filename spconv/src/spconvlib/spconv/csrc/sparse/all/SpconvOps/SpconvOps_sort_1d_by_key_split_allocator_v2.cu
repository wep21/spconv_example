#include "hip/hip_runtime.h"
#include <spconvlib/spconv/csrc/sparse/all/SpconvOps.h>
#include <spconvlib/spconv/csrc/sparse/all/CustomThrustLib.h>
#include <spconvlib/cumm/common/TensorViewKernel.h>
#include <spconvlib/spconv/csrc/sparse/all/cudakers/CudaCommonKernel.h>

        template <typename T> struct MaskedElementComp {
            T mask_;
            TV_HOST_DEVICE_INLINE T operator()(const T &x, const T &y) const {
                return (x & mask_) < (y & mask_);
            }
        };
        template <typename T> __global__ void mask_input(T* inp, T mask, int size){
            for (int i : tv::KernelLoopX<int>(size)){
                inp[i] &= mask;
            }
        }
        
namespace spconvlib {
namespace spconv {
namespace csrc {
namespace sparse {
namespace all {
using ThrustCustomAllocatorV2 = spconvlib::spconv::csrc::sparse::all::ThrustCustomAllocatorV2;
using ExternalAllocator = spconvlib::spconv::csrc::sparse::alloc::ExternalAllocator;
using GemmBasicHost = spconvlib::cumm::common::GemmBasicHost;
using ThrustAllocator = spconvlib::spconv::csrc::sparse::alloc::ThrustAllocator;
using Point2Voxel1DCPU = spconvlib::spconv::csrc::sparse::all::ops_cpu1d::Point2VoxelCPU;
using SpconvIndicesCPU1D = spconvlib::spconv::csrc::sparse::all::ops_cpu1d::SparseConvIndicesCPU;
using Point2Voxel1D = spconvlib::spconv::csrc::sparse::all::ops1d::Point2Voxel;
using Point2Voxel2DCPU = spconvlib::spconv::csrc::sparse::all::ops_cpu2d::Point2VoxelCPU;
using SpconvIndicesCPU2D = spconvlib::spconv::csrc::sparse::all::ops_cpu2d::SparseConvIndicesCPU;
using Point2Voxel2D = spconvlib::spconv::csrc::sparse::all::ops2d::Point2Voxel;
using Point2Voxel3DCPU = spconvlib::spconv::csrc::sparse::all::ops_cpu3d::Point2VoxelCPU;
using SpconvIndicesCPU3D = spconvlib::spconv::csrc::sparse::all::ops_cpu3d::SparseConvIndicesCPU;
using Point2Voxel3D = spconvlib::spconv::csrc::sparse::all::ops3d::Point2Voxel;
using Point2Voxel4DCPU = spconvlib::spconv::csrc::sparse::all::ops_cpu4d::Point2VoxelCPU;
using SpconvIndicesCPU4D = spconvlib::spconv::csrc::sparse::all::ops_cpu4d::SparseConvIndicesCPU;
using Point2Voxel4D = spconvlib::spconv::csrc::sparse::all::ops4d::Point2Voxel;
using CustomThrustLib = spconvlib::spconv::csrc::sparse::all::CustomThrustLib;
using TensorViewKernel = spconvlib::cumm::common::TensorViewKernel;
tv::Tensor SpconvOps::sort_1d_by_key_split_allocator_v2(tv::Tensor data, ThrustAllocator& allocator, tv::Tensor mask, tv::Tensor indices, std::uintptr_t stream, bool mask_output)   {
  
  hipStream_t stream_cu = reinterpret_cast<hipStream_t>(stream);
  // auto timer = tv::CudaContextTimer<>();
  if (indices.empty()){
      indices = tv::empty({data.dim(0)}, tv::int32, 0);
  }
  tv::cuda::Launch launcher(data.dim(0), stream_cu);
  launcher(cudakers::arange_kernel<int32_t>, indices.data_ptr<int32_t>(), indices.dim(0));
  tv::dispatch<int32_t, uint32_t, int64_t, uint64_t>(data.dtype(), [&](auto I){
      using T = TV_DECLTYPE(I);
      auto masks_ptr = mask.data_ptr<T>();
      MaskedElementComp<T> op_comp{masks_ptr[0]};
      thrust::device_ptr<T> ptr_tr(data.data_ptr<T>());
      thrust::device_ptr<int32_t> ptr_k(indices.data_ptr<int32_t>());
      // auto thrust_ctx = thrust::cuda::par.on(stream_cu);
      auto ctx2 = thrust::cuda::par(allocator).on(stream_cu);
      thrust::sort_by_key(ctx2, ptr_tr, ptr_tr + data.dim(0), ptr_k, op_comp);
      if (mask_output){
          launcher(mask_input<T>, data.data_ptr<T>(), masks_ptr[0], data.dim(0));
      }
  });
  // tv::ssprint("SORT_BY_KEY_MASKED", timer.report() / 1000.0);
  return indices;
}
} // namespace all
} // namespace sparse
} // namespace csrc
} // namespace spconv
} // namespace spconvlib