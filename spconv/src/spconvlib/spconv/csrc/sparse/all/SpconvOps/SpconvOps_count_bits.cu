#include "hip/hip_runtime.h"
#include <spconvlib/spconv/csrc/sparse/all/SpconvOps.h>
#include <spconvlib/cumm/common/TensorViewKernel.h>

        __global__ void count_bits_kernel_64(const uint64_t* data, int32_t* out, int size){
            for (int i : tv::KernelLoopX<int>(size)){
                out[i] = __popcll(reinterpret_cast<const unsigned long long*>(data)[i]);
            }
        }
        __global__ void count_bits_kernel(const uint32_t* data, int32_t* out, int size){
            for (int i : tv::KernelLoopX<int>(size)){
                out[i] = __popc(data[i]);
            }
        }

        int numberOfSetBits(uint32_t i)
        {
            // https://stackoverflow.com/questions/109023/how-to-count-the-number-of-set-bits-in-a-32-bit-integer
            // Java: use int, and use >>> instead of >>. Or use Integer.bitCount()
            // C or C++: use uint32_t
            i = i - ((i >> 1) & 0x55555555);        // add pairs of bits
            i = (i & 0x33333333) + ((i >> 2) & 0x33333333);  // quads
            i = (i + (i >> 4)) & 0x0F0F0F0F;        // groups of 8
            return (i * 0x01010101) >> 24;          // horizontal sum of bytes
        }

        int numberOfSetBits(uint64_t i)
        {
            return numberOfSetBits(uint32_t(i)) + numberOfSetBits(uint32_t(i >> 32));
        }
        
namespace spconvlib {
namespace spconv {
namespace csrc {
namespace sparse {
namespace all {
using ThrustCustomAllocatorV2 = spconvlib::spconv::csrc::sparse::all::ThrustCustomAllocatorV2;
using ExternalAllocator = spconvlib::spconv::csrc::sparse::alloc::ExternalAllocator;
using GemmBasicHost = spconvlib::cumm::common::GemmBasicHost;
using ThrustAllocator = spconvlib::spconv::csrc::sparse::alloc::ThrustAllocator;
using Point2Voxel1DCPU = spconvlib::spconv::csrc::sparse::all::ops_cpu1d::Point2VoxelCPU;
using SpconvIndicesCPU1D = spconvlib::spconv::csrc::sparse::all::ops_cpu1d::SparseConvIndicesCPU;
using Point2Voxel1D = spconvlib::spconv::csrc::sparse::all::ops1d::Point2Voxel;
using Point2Voxel2DCPU = spconvlib::spconv::csrc::sparse::all::ops_cpu2d::Point2VoxelCPU;
using SpconvIndicesCPU2D = spconvlib::spconv::csrc::sparse::all::ops_cpu2d::SparseConvIndicesCPU;
using Point2Voxel2D = spconvlib::spconv::csrc::sparse::all::ops2d::Point2Voxel;
using Point2Voxel3DCPU = spconvlib::spconv::csrc::sparse::all::ops_cpu3d::Point2VoxelCPU;
using SpconvIndicesCPU3D = spconvlib::spconv::csrc::sparse::all::ops_cpu3d::SparseConvIndicesCPU;
using Point2Voxel3D = spconvlib::spconv::csrc::sparse::all::ops3d::Point2Voxel;
using Point2Voxel4DCPU = spconvlib::spconv::csrc::sparse::all::ops_cpu4d::Point2VoxelCPU;
using SpconvIndicesCPU4D = spconvlib::spconv::csrc::sparse::all::ops_cpu4d::SparseConvIndicesCPU;
using Point2Voxel4D = spconvlib::spconv::csrc::sparse::all::ops4d::Point2Voxel;
using TensorViewKernel = spconvlib::cumm::common::TensorViewKernel;
tv::Tensor SpconvOps::count_bits(tv::Tensor a)   {
  
  tv::Tensor res(a.shape(), tv::int32, a.device());
  tv::dispatch<uint32_t, uint64_t>(a.dtype(), [&](auto I){
      auto res_ptr = res.data_ptr<int>();
      using T = TV_DECLTYPE(I);
      auto a_ptr = a.data_ptr<const T>();
      if (a.device() == -1){
          for (int i = 0; i < a.size(); ++i){
              res_ptr[i] = numberOfSetBits(a_ptr[i]);
          }
      }else{
          tv::cuda::Launch launcher(a.size());
          tv::if_constexpr<std::is_same<T, uint64_t>::value>([=](auto _)mutable{
              launcher(_(count_bits_kernel_64), a_ptr, res_ptr, int(a.size()));
          }, [=](auto _)mutable{
              launcher(_(count_bits_kernel), a_ptr, res_ptr, int(a.size()));
          });
      }
  });
  return res;
}
} // namespace all
} // namespace sparse
} // namespace csrc
} // namespace spconv
} // namespace spconvlib