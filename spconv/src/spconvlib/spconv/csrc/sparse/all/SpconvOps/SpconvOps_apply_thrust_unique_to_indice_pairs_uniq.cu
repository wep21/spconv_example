#include <spconvlib/spconv/csrc/sparse/all/SpconvOps.h>
#include <spconvlib/spconv/csrc/sparse/all/CustomThrustLib.h>
namespace spconvlib {
namespace spconv {
namespace csrc {
namespace sparse {
namespace all {
using ThrustCustomAllocatorV2 = spconvlib::spconv::csrc::sparse::all::ThrustCustomAllocatorV2;
using ExternalAllocator = spconvlib::spconv::csrc::sparse::alloc::ExternalAllocator;
using GemmBasicHost = spconvlib::cumm::common::GemmBasicHost;
using ThrustAllocator = spconvlib::spconv::csrc::sparse::alloc::ThrustAllocator;
using Point2Voxel1DCPU = spconvlib::spconv::csrc::sparse::all::ops_cpu1d::Point2VoxelCPU;
using SpconvIndicesCPU1D = spconvlib::spconv::csrc::sparse::all::ops_cpu1d::SparseConvIndicesCPU;
using Point2Voxel1D = spconvlib::spconv::csrc::sparse::all::ops1d::Point2Voxel;
using Point2Voxel2DCPU = spconvlib::spconv::csrc::sparse::all::ops_cpu2d::Point2VoxelCPU;
using SpconvIndicesCPU2D = spconvlib::spconv::csrc::sparse::all::ops_cpu2d::SparseConvIndicesCPU;
using Point2Voxel2D = spconvlib::spconv::csrc::sparse::all::ops2d::Point2Voxel;
using Point2Voxel3DCPU = spconvlib::spconv::csrc::sparse::all::ops_cpu3d::Point2VoxelCPU;
using SpconvIndicesCPU3D = spconvlib::spconv::csrc::sparse::all::ops_cpu3d::SparseConvIndicesCPU;
using Point2Voxel3D = spconvlib::spconv::csrc::sparse::all::ops3d::Point2Voxel;
using Point2Voxel4DCPU = spconvlib::spconv::csrc::sparse::all::ops_cpu4d::Point2VoxelCPU;
using SpconvIndicesCPU4D = spconvlib::spconv::csrc::sparse::all::ops_cpu4d::SparseConvIndicesCPU;
using Point2Voxel4D = spconvlib::spconv::csrc::sparse::all::ops4d::Point2Voxel;
using CustomThrustLib = spconvlib::spconv::csrc::sparse::all::CustomThrustLib;
int SpconvOps::apply_thrust_unique_to_indice_pairs_uniq(tv::Tensor data, ThrustAllocator& allocator, std::uintptr_t stream_int)   {
  
  int num_out_act = 0;
  int uniq_size = data.dim(0);
  tv::dispatch<int32_t, int64_t>(data.dtype(), [&](auto I){
      using T = TV_DECLTYPE(I);
      thrust::device_ptr<T> ptr_tr(data.data_ptr<T>());
      auto thrust_ctx = thrust::cuda::par(allocator).on(reinterpret_cast<hipStream_t>(stream_int));
      thrust::sort(thrust_ctx, ptr_tr, ptr_tr + uniq_size);
      auto new_end = thrust::unique(thrust_ctx, ptr_tr, ptr_tr + uniq_size);
      num_out_act = new_end - ptr_tr - 1;
  });
  return num_out_act;
}
} // namespace all
} // namespace sparse
} // namespace csrc
} // namespace spconv
} // namespace spconvlib