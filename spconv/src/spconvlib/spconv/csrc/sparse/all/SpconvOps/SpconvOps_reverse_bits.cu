#include "hip/hip_runtime.h"
#include <spconvlib/spconv/csrc/sparse/all/SpconvOps.h>
#include <spconvlib/cumm/common/TensorViewKernel.h>

        __global__ void reverse_bits_kernel_64(const uint64_t* data, uint64_t* out, int size){
            for (int i : tv::KernelLoopX<int>(size)){
                out[i] = __brevll(reinterpret_cast<const unsigned long long*>(data)[i]);
            }
        }

        __global__ void reverse_bits_kernel(const uint32_t* data, uint32_t* out, int size){
            for (int i : tv::KernelLoopX<int>(size)){
                out[i] = __brev(data[i]);
            }
        }

        uint32_t reverse(uint32_t x)
        {
            x = ((x >> 1) & 0x55555555u) | ((x & 0x55555555u) << 1);
            x = ((x >> 2) & 0x33333333u) | ((x & 0x33333333u) << 2);
            x = ((x >> 4) & 0x0f0f0f0fu) | ((x & 0x0f0f0f0fu) << 4);
            x = ((x >> 8) & 0x00ff00ffu) | ((x & 0x00ff00ffu) << 8);
            x = ((x >> 16) & 0xffffu) | ((x & 0xffffu) << 16);
            return x;
        }

        int reverse(uint64_t i)
        {
            return (reverse(uint32_t(i)) << 32) | reverse(uint32_t(i >> 32));
        }
        
namespace spconvlib {
namespace spconv {
namespace csrc {
namespace sparse {
namespace all {
using ThrustCustomAllocatorV2 = spconvlib::spconv::csrc::sparse::all::ThrustCustomAllocatorV2;
using ExternalAllocator = spconvlib::spconv::csrc::sparse::alloc::ExternalAllocator;
using GemmBasicHost = spconvlib::cumm::common::GemmBasicHost;
using ThrustAllocator = spconvlib::spconv::csrc::sparse::alloc::ThrustAllocator;
using Point2Voxel1DCPU = spconvlib::spconv::csrc::sparse::all::ops_cpu1d::Point2VoxelCPU;
using SpconvIndicesCPU1D = spconvlib::spconv::csrc::sparse::all::ops_cpu1d::SparseConvIndicesCPU;
using Point2Voxel1D = spconvlib::spconv::csrc::sparse::all::ops1d::Point2Voxel;
using Point2Voxel2DCPU = spconvlib::spconv::csrc::sparse::all::ops_cpu2d::Point2VoxelCPU;
using SpconvIndicesCPU2D = spconvlib::spconv::csrc::sparse::all::ops_cpu2d::SparseConvIndicesCPU;
using Point2Voxel2D = spconvlib::spconv::csrc::sparse::all::ops2d::Point2Voxel;
using Point2Voxel3DCPU = spconvlib::spconv::csrc::sparse::all::ops_cpu3d::Point2VoxelCPU;
using SpconvIndicesCPU3D = spconvlib::spconv::csrc::sparse::all::ops_cpu3d::SparseConvIndicesCPU;
using Point2Voxel3D = spconvlib::spconv::csrc::sparse::all::ops3d::Point2Voxel;
using Point2Voxel4DCPU = spconvlib::spconv::csrc::sparse::all::ops_cpu4d::Point2VoxelCPU;
using SpconvIndicesCPU4D = spconvlib::spconv::csrc::sparse::all::ops_cpu4d::SparseConvIndicesCPU;
using Point2Voxel4D = spconvlib::spconv::csrc::sparse::all::ops4d::Point2Voxel;
using TensorViewKernel = spconvlib::cumm::common::TensorViewKernel;
tv::Tensor SpconvOps::reverse_bits(tv::Tensor a)   {
  
  tv::Tensor res(a.shape(), a.dtype(), a.device());
  tv::dispatch<uint32_t, uint64_t>(a.dtype(), [&](auto I){
      using T = TV_DECLTYPE(I);
      auto res_ptr = res.data_ptr<T>();
      auto a_ptr = a.data_ptr<const T>();
      if (a.device() == -1){
          for (int i = 0; i < a.size(); ++i){
              res_ptr[i] = reverse(a_ptr[i]);
          }
      }else{
          tv::cuda::Launch launcher(a.size());
          tv::if_constexpr<std::is_same<T, uint64_t>::value>([=](auto _)mutable{
              launcher(_(reverse_bits_kernel_64), a_ptr, res_ptr, int(a.size()));
          }, [=](auto _)mutable{
              launcher(_(reverse_bits_kernel), a_ptr, res_ptr, int(a.size()));
          });
      }
  });
  return res;
}
} // namespace all
} // namespace sparse
} // namespace csrc
} // namespace spconv
} // namespace spconvlib