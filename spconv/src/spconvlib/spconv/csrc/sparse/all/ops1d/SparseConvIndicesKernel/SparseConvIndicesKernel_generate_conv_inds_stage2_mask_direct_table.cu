#include <spconvlib/spconv/csrc/sparse/all/ops1d/SparseConvIndicesKernel.h>
namespace spconvlib {
namespace spconv {
namespace csrc {
namespace sparse {
namespace all {
namespace ops1d {
using TensorView = spconvlib::cumm::common::TensorView;
using TensorViewKernel = spconvlib::cumm::common::TensorViewKernel;
using TensorViewHashKernel = spconvlib::cumm::common::TensorViewHashKernel;
using ConvLocIter = spconvlib::spconv::csrc::sparse::all::ops1d::spinds::ConvOutLocIter;
using ConvProblem = spconvlib::spconv::csrc::sparse::all::ops_cpu1d::spinds::ConvProblem;
using ConvLocIter64 = spconvlib::spconv::csrc::sparse::all::ops1d::spinds64::ConvOutLocIter;
int SparseConvIndicesKernel::generate_conv_inds_stage2_mask_direct_table(tv::Tensor indices, tv::Tensor hashdata_k, tv::Tensor hashdata_v, tv::Tensor indice_pairs_fwd, tv::Tensor indice_pairs_bwd, tv::Tensor indice_pairs_uniq, tv::Tensor indice_pairs_uniq_before_sort, tv::Tensor out_inds, tv::Tensor mask_fwd, tv::Tensor mask_bwd, int num_out_act, int batch_size, tv::array<int, 1> output_dims, tv::array<int, 1> input_dims, tv::array<int, 1> ksize, tv::array<int, 1> stride, tv::array<int, 1> padding, tv::array<int, 1> dilation, bool transposed, std::uintptr_t stream_int)   {
  
  auto custream = reinterpret_cast<hipStream_t>(stream_int);
  // TODO stream
  // TODO handle num input == 0
  int kv = ksize.op<tv::arrayops::prod>();
  // indice_pairs_bwd: [kv, num_act_in]  or empty
  // indice_pairs_fwd: [kv, num_act_out]
  auto ctx = tv::Context();
  ctx.set_cuda_stream(custream);
  int num_act_in = indices.dim(0);
  int num_act_out = num_out_act;
  TV_ASSERT_RT_ERR(hashdata_v.dtype() == tv::int32, "error");
  // out_inds: [num_out_act, 2]
  // auto timer = tv::CudaContextTimer<>();
  if (!indice_pairs_bwd.empty()){
      tv::check_shape(indice_pairs_bwd, {kv, num_act_in});
  }
  tv::check_shape(indice_pairs_fwd, {kv, num_act_out});
  tv::check_shape(out_inds, {num_out_act, 2});
  tv::cuda::Launch launcher_num_act_in(num_act_in, custream);
  launcher_num_act_in.blocks.y = kv;
  tv::cuda::Launch launcher_num_act_in_no_y(num_act_in, custream);
  ConvProblem problem(batch_size, 1, 1, input_dims, output_dims, ksize, padding, stride, dilation);
  tv::cuda::Launch lanucher_build_hash(num_out_act, custream);
  bool use_int32 = problem.check_npq_not_overflow();
  // TODO handle invalid num_out_act
  tv::dispatch<int32_t, int64_t>(hashdata_k.dtype(), [&](auto I){
    using V = int32_t;
    using K = TV_DECLTYPE(I);
    using table_t =
        tv::hash::LinearHashTableSplit<K, V, tv::hash::Murmur3Hash<K>,
                                    tv::hash::default_empty_key_v<K>, false>;
    TV_ASSERT_RT_ERR(hashdata_k.dim(0) >= num_out_act, "hash size not enough");
    table_t hash = table_t(hashdata_k.data_ptr<K>(), hashdata_v.data_ptr<V>(), hashdata_k.dim(0));
    if (!mask_bwd.empty()){
        launcher_num_act_in(calc_conv_indices_stage2_mask<table_t, true>, hash, 
            indice_pairs_fwd.data_ptr<int>(), indice_pairs_bwd.data_ptr<int>(), 
            indice_pairs_uniq_before_sort.data_ptr<K>(),
            mask_fwd.data_ptr<uint32_t>(), mask_bwd.data_ptr<uint32_t>(),
            num_act_in, indice_pairs_fwd.dim(1));
        launcher_num_act_in_no_y(calc_conv_indices_stage2_mask_output, 
            indice_pairs_bwd.data_ptr<int>(), 
            mask_bwd.data_ptr<uint32_t>(),
            num_act_in, kv);
        if (mask_fwd.dim(0) == 2){
            mask_fwd[1].copy_(mask_fwd[0], ctx);
        }
        if (mask_bwd.dim(0) == 2){
            mask_bwd[1].copy_(mask_bwd[0], ctx);
        }
    }else{
        launcher_num_act_in(calc_conv_indices_stage2_inference_mask<table_t, true>, hash, 
            indice_pairs_fwd.data_ptr<int>(), indice_pairs_bwd.data_ptr<int>(), 
            indice_pairs_uniq_before_sort.data_ptr<K>(),
            mask_fwd.data_ptr<uint32_t>(),
            num_act_in, indice_pairs_fwd.dim(1));
        if (mask_fwd.dim(0) == 2){
            mask_fwd[1].copy_(mask_fwd[0], ctx);
        }
    }
  });
  return num_out_act;
}
} // namespace ops1d
} // namespace all
} // namespace sparse
} // namespace csrc
} // namespace spconv
} // namespace spconvlib