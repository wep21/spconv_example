#include "hip/hip_runtime.h"
#include <spconvlib/spconv/csrc/sparse/all/ops1d/kernel/Point2VoxelKernel.h>
namespace spconvlib {
namespace spconv {
namespace csrc {
namespace sparse {
namespace all {
namespace ops1d {
namespace kernel {
using TensorView = spconvlib::cumm::common::TensorView;
using TensorViewHashKernel = spconvlib::cumm::common::TensorViewHashKernel;
using Layout = spconvlib::spconv::csrc::sparse::all::ops1d::layout_ns::TensorGeneric;
__global__ void limit_num_per_voxel_value(int * num_per_voxel, int num_voxels, int num_points_per_voxel)   {
  
  for (int i : tv::KernelLoopX<int>(num_voxels)){
      int count = min(num_points_per_voxel, num_per_voxel[i]);
      num_per_voxel[i] = count;
  }
}
} // namespace kernel
} // namespace ops1d
} // namespace all
} // namespace sparse
} // namespace csrc
} // namespace spconv
} // namespace spconvlib