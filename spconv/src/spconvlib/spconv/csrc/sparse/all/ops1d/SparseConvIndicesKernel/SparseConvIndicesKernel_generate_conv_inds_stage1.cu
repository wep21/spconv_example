#include "hip/hip_runtime.h"
#include <spconvlib/spconv/csrc/sparse/all/ops1d/SparseConvIndicesKernel.h>
namespace spconvlib {
namespace spconv {
namespace csrc {
namespace sparse {
namespace all {
namespace ops1d {
using TensorView = spconvlib::cumm::common::TensorView;
using TensorViewKernel = spconvlib::cumm::common::TensorViewKernel;
using TensorViewHashKernel = spconvlib::cumm::common::TensorViewHashKernel;
using ConvLocIter = spconvlib::spconv::csrc::sparse::all::ops1d::spinds::ConvOutLocIter;
using ConvProblem = spconvlib::spconv::csrc::sparse::all::ops_cpu1d::spinds::ConvProblem;
using ConvLocIter64 = spconvlib::spconv::csrc::sparse::all::ops1d::spinds64::ConvOutLocIter;
void SparseConvIndicesKernel::generate_conv_inds_stage1(tv::Tensor indices, tv::Tensor indice_pairs, tv::Tensor indice_pairs_uniq, tv::Tensor indice_num_per_loc, int batch_size, tv::array<int, 1> output_dims, tv::array<int, 1> input_dims, tv::array<int, 1> ksize, tv::array<int, 1> stride, tv::array<int, 1> padding, tv::array<int, 1> dilation, bool transposed, std::uintptr_t stream_int)   {
  
  // TODO stream
  // TODO handle num input == 0
  int kv = ksize.op<tv::arrayops::prod>();
  TV_ASSERT_RT_ERR(kv == indice_pairs.dim(1), "error");
  // indice_pairs: [2, kv, num_act_in]
  // indice_pairs_uniq: [num_act_in * kv + 1]
  tv::check_shape(indice_pairs, {2, kv, -1});
  // TV_ASSERT_RT_ERR(indice_pairs.dim(-1) == indices.dim(0), "error");
  tv::check_shape(indice_num_per_loc, {kv});
  int64_t uniq_size = indice_pairs.size() / 2 + 1;
  TV_ASSERT_RT_ERR(indice_pairs_uniq.dim(0) >= uniq_size, "error");
  TV_ASSERT_RT_ERR(indice_num_per_loc.dim(0) == kv, "error");
  tv::cuda::Launch launcher_num_act_in(indices.dim(0), reinterpret_cast<hipStream_t>(stream_int));
  // tv::cuda::Launch launcher_num_act_in_2(indices.dim(0));
  launcher_num_act_in.blocks.y = kv;
  ConvProblem problem(batch_size, 1, 1, input_dims, output_dims, ksize, padding, stride, dilation);
  bool use_int32 = problem.check_npq_not_overflow();
  tv::cuda::Launch launcher_clean_uniq(uniq_size, reinterpret_cast<hipStream_t>(stream_int));
  if (int(use_int32) == 0){
    ConvLocIter64 loc_iter(problem);
    tv::dispatch<int32_t, int64_t>(indice_pairs_uniq.dtype(), [&](auto I){
        using T = TV_DECLTYPE(I);
        TV_ASSERT_RT_ERR(input_dims.op<tv::arrayops::prod>() < std::numeric_limits<T>::max(), 
            "kernel volume must smaller than max value of T");
        launcher_clean_uniq(clean_indices_uniq<T>, indice_pairs_uniq.data_ptr<T>(), uniq_size);
        launcher_num_act_in(calc_conv_indices_stage1<T, ConvLocIter64>, loc_iter, indices.data_ptr<const int>(), 
            indice_pairs.data_ptr<int32_t>(), 
            indice_pairs_uniq.data_ptr<T>(), indice_num_per_loc.data_ptr<int>(), indices.dim(0),
            indice_pairs.dim(2), kv, transposed);
    });
  }
  else if (int(use_int32) == 1){
    ConvLocIter loc_iter(problem);
    tv::dispatch<int32_t, int64_t>(indice_pairs_uniq.dtype(), [&](auto I){
        using T = TV_DECLTYPE(I);
        TV_ASSERT_RT_ERR(input_dims.op<tv::arrayops::prod>() < std::numeric_limits<T>::max(), 
            "kernel volume must smaller than max value of T");
        launcher_clean_uniq(clean_indices_uniq<T>, indice_pairs_uniq.data_ptr<T>(), uniq_size);
        launcher_num_act_in(calc_conv_indices_stage1<T, ConvLocIter>, loc_iter, indices.data_ptr<const int>(), 
            indice_pairs.data_ptr<int32_t>(), 
            indice_pairs_uniq.data_ptr<T>(), indice_num_per_loc.data_ptr<int>(), indices.dim(0),
            indice_pairs.dim(2), kv, transposed);
    });
  }
  else{
    TV_THROW_RT_ERR("unknown val int(use_int32), available: [0, 1]")
  }
}
} // namespace ops1d
} // namespace all
} // namespace sparse
} // namespace csrc
} // namespace spconv
} // namespace spconvlib