#include "hip/hip_runtime.h"
#include <spconvlib/spconv/csrc/sparse/all/ops2d/SparseConvIndicesKernel.h>
namespace spconvlib {
namespace spconv {
namespace csrc {
namespace sparse {
namespace all {
namespace ops2d {
using TensorView = spconvlib::cumm::common::TensorView;
using TensorViewKernel = spconvlib::cumm::common::TensorViewKernel;
using TensorViewHashKernel = spconvlib::cumm::common::TensorViewHashKernel;
using ConvLocIter = spconvlib::spconv::csrc::sparse::all::ops2d::spinds::ConvOutLocIter;
using ConvProblem = spconvlib::spconv::csrc::sparse::all::ops_cpu2d::spinds::ConvProblem;
using ConvLocIter64 = spconvlib::spconv::csrc::sparse::all::ops2d::spinds64::ConvOutLocIter;
void SparseConvIndicesKernel::generate_conv_inds_mask_stage1(tv::Tensor indices, tv::Tensor indice_pairs_bwd, tv::Tensor indice_pairs_uniq, tv::Tensor indice_num_per_loc, int batch_size, tv::array<int, 2> output_dims, tv::array<int, 2> input_dims, tv::array<int, 2> ksize, tv::array<int, 2> stride, tv::array<int, 2> padding, tv::array<int, 2> dilation, bool transposed, std::uintptr_t stream_int)   {
  
  // TODO stream
  // TODO handle num input == 0
  int kv = ksize.op<tv::arrayops::prod>();
  int num_act_in = indices.dim(0);
  // indice_pairs_bwd: [kv, num_act_in] or empty
  // indice_pairs_uniq: [kv * num_act_in + 1]
  if (!indice_pairs_bwd.empty()){
      tv::check_shape(indice_pairs_bwd, {kv, num_act_in});
  }
  tv::check_shape(indice_num_per_loc, {kv});
  int64_t uniq_size = kv * num_act_in + 1;
  TV_ASSERT_RT_ERR(indice_pairs_uniq.dim(0) == uniq_size, "error");
  tv::cuda::Launch launcher_num_act_in(indices.dim(0), reinterpret_cast<hipStream_t>(stream_int));
  // tv::cuda::Launch launcher_num_act_in_2(indices.dim(0));
  launcher_num_act_in.blocks.y = kv;
  ConvProblem problem(batch_size, 1, 1, input_dims, output_dims, ksize, padding, stride, dilation);
  bool use_int32 = problem.check_npq_not_overflow();
  tv::cuda::Launch launcher_clean_uniq(uniq_size, reinterpret_cast<hipStream_t>(stream_int));
  if (int(use_int32) == 0){
    ConvLocIter64 loc_iter(problem);
    tv::dispatch<int32_t, int64_t>(indice_pairs_uniq.dtype(), [&](auto I){
        using T = TV_DECLTYPE(I);
        TV_ASSERT_RT_ERR(input_dims.op<tv::arrayops::prod>() < std::numeric_limits<T>::max(), 
            "kernel volume must smaller than max value of T");
        launcher_clean_uniq(clean_indices_uniq<T>, indice_pairs_uniq.data_ptr<T>(), uniq_size);
        launcher_num_act_in(calc_conv_indices_stage1_mask<T, ConvLocIter64>, loc_iter, indices.data_ptr<const int>(), 
            indice_pairs_bwd.data_ptr<int32_t>(), 
            indice_pairs_uniq.data_ptr<T>(), indice_num_per_loc.data_ptr<int>(), indices.dim(0),
            kv, transposed);
    });
  }
  else if (int(use_int32) == 1){
    ConvLocIter loc_iter(problem);
    tv::dispatch<int32_t, int64_t>(indice_pairs_uniq.dtype(), [&](auto I){
        using T = TV_DECLTYPE(I);
        TV_ASSERT_RT_ERR(input_dims.op<tv::arrayops::prod>() < std::numeric_limits<T>::max(), 
            "kernel volume must smaller than max value of T");
        launcher_clean_uniq(clean_indices_uniq<T>, indice_pairs_uniq.data_ptr<T>(), uniq_size);
        launcher_num_act_in(calc_conv_indices_stage1_mask<T, ConvLocIter>, loc_iter, indices.data_ptr<const int>(), 
            indice_pairs_bwd.data_ptr<int32_t>(), 
            indice_pairs_uniq.data_ptr<T>(), indice_num_per_loc.data_ptr<int>(), indices.dim(0),
            kv, transposed);
    });
  }
  else{
    TV_THROW_RT_ERR("unknown val int(use_int32), available: [0, 1]")
  }
}
} // namespace ops2d
} // namespace all
} // namespace sparse
} // namespace csrc
} // namespace spconv
} // namespace spconvlib