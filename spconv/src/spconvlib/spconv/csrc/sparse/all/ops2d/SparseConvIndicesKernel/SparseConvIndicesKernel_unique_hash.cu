#include "hip/hip_runtime.h"
#include <spconvlib/spconv/csrc/sparse/all/ops2d/SparseConvIndicesKernel.h>
namespace spconvlib {
namespace spconv {
namespace csrc {
namespace sparse {
namespace all {
namespace ops2d {
using TensorView = spconvlib::cumm::common::TensorView;
using TensorViewKernel = spconvlib::cumm::common::TensorViewKernel;
using TensorViewHashKernel = spconvlib::cumm::common::TensorViewHashKernel;
using ConvLocIter = spconvlib::spconv::csrc::sparse::all::ops2d::spinds::ConvOutLocIter;
using ConvProblem = spconvlib::spconv::csrc::sparse::all::ops_cpu2d::spinds::ConvProblem;
using ConvLocIter64 = spconvlib::spconv::csrc::sparse::all::ops2d::spinds64::ConvOutLocIter;
int SparseConvIndicesKernel::unique_hash(tv::Tensor hashdata_k, tv::Tensor hashdata_v, tv::Tensor uniq_cnt, tv::Tensor out_indices_offset, int num_out_bound, std::uintptr_t stream_int)   {
  
  auto custream = reinterpret_cast<hipStream_t>(stream_int);
  tv::cuda::Launch lanucher_build_hash(hashdata_k.size(), custream);
  auto tvctx = tv::Context();
  tvctx.set_cuda_stream(reinterpret_cast<hipStream_t>(stream_int));
  if (num_out_bound <= 0){
      num_out_bound = out_indices_offset.dim(0);
  }
  tv::dispatch<int32_t, int64_t>(hashdata_k.dtype(), [&](auto I){
      using V = int32_t;
      using K = TV_DECLTYPE(I);
      using table_t =
          tv::hash::LinearHashTableSplit<K, V, tv::hash::Murmur3Hash<K>,
                                      tv::hash::default_empty_key_v<K>, false>;
      table_t table = table_t(hashdata_k.data_ptr<K>(), hashdata_v.data_ptr<V>(), hashdata_k.dim(0));
      lanucher_build_hash(arange_hash_table<table_t>, table, 
          out_indices_offset.data_ptr<K>(),
          uniq_cnt.data_ptr<int>(), num_out_bound);
  });
  auto uniq_cnt_cpu = uniq_cnt.cpu(tvctx);
  return std::min(uniq_cnt_cpu.data_ptr<int>()[0], num_out_bound);
}
} // namespace ops2d
} // namespace all
} // namespace sparse
} // namespace csrc
} // namespace spconv
} // namespace spconvlib