#include "hip/hip_runtime.h"
#include <spconvlib/spconv/csrc/sparse/all/ops3d/SparseConvIndicesKernel.h>
namespace spconvlib {
namespace spconv {
namespace csrc {
namespace sparse {
namespace all {
namespace ops3d {
using TensorView = spconvlib::cumm::common::TensorView;
using TensorViewKernel = spconvlib::cumm::common::TensorViewKernel;
using TensorViewHashKernel = spconvlib::cumm::common::TensorViewHashKernel;
using ConvLocIter = spconvlib::spconv::csrc::sparse::all::ops3d::spinds::ConvOutLocIter;
using ConvProblem = spconvlib::spconv::csrc::sparse::all::ops_cpu3d::spinds::ConvProblem;
using ConvLocIter64 = spconvlib::spconv::csrc::sparse::all::ops3d::spinds64::ConvOutLocIter;
int SparseConvIndicesKernel::unique_and_assign_output_direct_hash(tv::Tensor hashdata_k, tv::Tensor hashdata_v, tv::Tensor uniq_cnt, tv::Tensor out_inds, int num_out_bound, int batch_size, tv::array<int, 3> output_dims, tv::array<int, 3> input_dims, tv::array<int, 3> ksize, tv::array<int, 3> stride, tv::array<int, 3> padding, tv::array<int, 3> dilation, std::uintptr_t stream_int)   {
  
  auto custream = reinterpret_cast<hipStream_t>(stream_int);
  tv::cuda::Launch lanucher_build_hash(hashdata_k.size(), custream);
  ConvProblem problem(batch_size, 1, 1, input_dims, output_dims, ksize, padding, stride, dilation);
  bool use_int32 = problem.check_npq_not_overflow();
  auto tvctx = tv::Context();
  tvctx.set_cuda_stream(reinterpret_cast<hipStream_t>(stream_int));
  if (num_out_bound <= 0){
      num_out_bound = hashdata_k.size();
  }
  if (int(use_int32) == 0){
    ConvLocIter64 loc_iter(problem);
    tv::dispatch<int32_t, int64_t>(hashdata_k.dtype(), [&](auto I){
        using V = int32_t;
        using K = TV_DECLTYPE(I);
        using table_t =
            tv::hash::LinearHashTableSplit<K, V, tv::hash::Murmur3Hash<K>,
                                        tv::hash::default_empty_key_v<K>, false>;
        table_t table = table_t(hashdata_k.data_ptr<K>(), hashdata_v.data_ptr<V>(), hashdata_k.dim(0));
        lanucher_build_hash(arange_hash_table_and_assign_out<table_t, std::decay_t<decltype(loc_iter.layout_npq)>>, table, 
            out_inds.data_ptr<int>(), uniq_cnt.data_ptr<int>(), num_out_bound,
            loc_iter.layout_npq);
    });
  }
  else if (int(use_int32) == 1){
    ConvLocIter loc_iter(problem);
    tv::dispatch<int32_t, int64_t>(hashdata_k.dtype(), [&](auto I){
        using V = int32_t;
        using K = TV_DECLTYPE(I);
        using table_t =
            tv::hash::LinearHashTableSplit<K, V, tv::hash::Murmur3Hash<K>,
                                        tv::hash::default_empty_key_v<K>, false>;
        table_t table = table_t(hashdata_k.data_ptr<K>(), hashdata_v.data_ptr<V>(), hashdata_k.dim(0));
        lanucher_build_hash(arange_hash_table_and_assign_out<table_t, std::decay_t<decltype(loc_iter.layout_npq)>>, table, 
            out_inds.data_ptr<int>(), uniq_cnt.data_ptr<int>(), num_out_bound,
            loc_iter.layout_npq);
    });
  }
  else{
    TV_THROW_RT_ERR("unknown val int(use_int32), available: [0, 1]")
  }
  auto uniq_cnt_cpu = uniq_cnt.cpu(tvctx);
  return std::min(uniq_cnt_cpu.data_ptr<int>()[0], num_out_bound);
}
} // namespace ops3d
} // namespace all
} // namespace sparse
} // namespace csrc
} // namespace spconv
} // namespace spconvlib