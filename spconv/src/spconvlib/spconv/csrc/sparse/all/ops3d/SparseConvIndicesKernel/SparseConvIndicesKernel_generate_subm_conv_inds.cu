#include "hip/hip_runtime.h"
#include <spconvlib/spconv/csrc/sparse/all/ops3d/SparseConvIndicesKernel.h>
namespace spconvlib {
namespace spconv {
namespace csrc {
namespace sparse {
namespace all {
namespace ops3d {
using TensorView = spconvlib::cumm::common::TensorView;
using TensorViewKernel = spconvlib::cumm::common::TensorViewKernel;
using TensorViewHashKernel = spconvlib::cumm::common::TensorViewHashKernel;
using ConvLocIter = spconvlib::spconv::csrc::sparse::all::ops3d::spinds::ConvOutLocIter;
using ConvProblem = spconvlib::spconv::csrc::sparse::all::ops_cpu3d::spinds::ConvProblem;
using ConvLocIter64 = spconvlib::spconv::csrc::sparse::all::ops3d::spinds64::ConvOutLocIter;
int SparseConvIndicesKernel::generate_subm_conv_inds(tv::Tensor indices, tv::Tensor hashdata_k, tv::Tensor hashdata_v, tv::Tensor indice_pairs, tv::Tensor out_inds, tv::Tensor indice_num_per_loc, int batch_size, tv::array<int, 3> input_dims, tv::array<int, 3> ksize, tv::array<int, 3> dilation, tv::Tensor indice_pair_mask, bool is_train, std::uintptr_t stream_int)   {
  
  int num_act_in_real = indices.dim(0);
  auto custream = reinterpret_cast<hipStream_t>(stream_int);
  auto ctx = tv::Context();
  ctx.set_cuda_stream(custream);
  if (!indice_pair_mask.empty()){
      TV_ASSERT_INVALID_ARG(ksize.op<tv::arrayops::prod>() <= 32, "for now only support 32bit mask");
  }
  // TODO stream
  // TODO handle num input == 0
  tv::array<int, 3> stride, padding;
  for (int i = 0; i < 3; ++i){
      TV_ASSERT_RT_ERR(ksize[i] % 2 == 1, "subm only support odd ksize");
      stride[i] = 1;
      padding[i] = (ksize[i] / 2) * dilation[i];
  }
  int kv = ksize.op<tv::arrayops::prod>();
  TV_ASSERT_RT_ERR(kv == indice_pairs.dim(1), "error");
  // indice_pairs: [1 or 2, kv, num_act_in] if mask else [2, kv, num_act_in]
  // out_inds: [MaxSize, 4]
  TV_ASSERT_RT_ERR(indice_num_per_loc.dim(0) == kv, "error");
  tv::cuda::Launch launcher_num_act_in(num_act_in_real, custream);
  launcher_num_act_in.blocks.y = (kv / 2) + 1;
  // launcher_num_act_in.blocks.y = kv;
  ConvProblem problem(batch_size, 1, 1, input_dims, input_dims, ksize, padding, stride, dilation);
  bool use_int32 = problem.check_npq_not_overflow();
  tv::cuda::Launch lanucher_build_hash(num_act_in_real, custream);
  if (int(use_int32) == 0){
        ConvLocIter64 loc_iter(problem);
        tv::dispatch<int32_t, int64_t>(hashdata_k.dtype(), [&](auto I){
            using V = int32_t;
            using K = TV_DECLTYPE(I);
            TV_ASSERT_RT_ERR(input_dims.op<tv::arrayops::prod>() < std::numeric_limits<K>::max(), 
                "kernel volume must smaller than max value of K");
            using table_t =
                tv::hash::LinearHashTableSplit<K, V, tv::hash::Murmur3Hash<K>,
                                            tv::hash::default_empty_key_v<K>, false>;
            TV_ASSERT_RT_ERR(hashdata_k.dim(0) >= num_act_in_real, "hash size not enough");
            table_t hash = table_t(hashdata_k.data_ptr<K>(), hashdata_v.data_ptr<V>(), hashdata_k.dim(0));
            tv::hash::clear_map_split(hash, custream);
            lanucher_build_hash(build_subm_conv_hash_table<table_t, std::decay_t<decltype(loc_iter.layout_npq)>>, hash, indices.data_ptr<const int>(),
                loc_iter.layout_npq, num_act_in_real);
            if (!indice_pair_mask.empty()){
                TV_ASSERT_RT_ERR(indice_pairs.ndim() == 3, "error");
                TV_ASSERT_RT_ERR(indice_pairs.dim(0) == (is_train ? 2 : 1), "error");
                TV_ASSERT_INVALID_ARG(indice_pair_mask.ndim() == 2, "error");
                // indice_pair_mask: [mask_split_count, num_act_in]
                if (indice_pair_mask.dim(0) == 2){
                    auto mask_0 = indice_pair_mask[0].slice_first_axis(0, num_act_in_real);
                    auto mask_1 = indice_pair_mask[1].slice_first_axis(0, num_act_in_real);
                    tv::cuda::Launch lanucher_fill(num_act_in_real, custream);
                    lanucher_fill(cudakers::fill_kernel<uint32_t>, mask_0.data_ptr<uint32_t>(), (1 << (kv / 2)), indices.dim(0));
                    mask_1.zero_(ctx);
                    auto kernel = &calc_subm_conv_indices_split_mask<table_t, ConvLocIter64>;
                    launcher_num_act_in(kernel, loc_iter, hash,  
                        indices.data_ptr<const int>(), indice_pairs.data_ptr<int>(), 
                        mask_0.data_ptr<uint32_t>(), mask_1.data_ptr<uint32_t>(), 
                        indices.dim(0), indice_pairs.dim(2), kv, is_train);
                }else{
                    // indice_pair_mask: [1, num_act_in]
                    tv::cuda::Launch lanucher_fill(num_act_in_real, custream);
                    lanucher_fill(cudakers::fill_kernel<uint32_t>, indice_pair_mask.data_ptr<uint32_t>(), (1 << (kv / 2)), indices.dim(0));
                    TV_ASSERT_RT_ERR(indice_pair_mask.dim(0) == 1, "error");
                    launcher_num_act_in(calc_subm_conv_indices_mask<table_t, ConvLocIter64>, loc_iter, hash, 
                        indices.data_ptr<const int>(), indice_pairs.data_ptr<int>(), 
                        indice_pair_mask.data_ptr<uint32_t>(), indices.dim(0), indice_pairs.dim(2), kv, is_train);
                }
            }else{
                TV_ASSERT_RT_ERR(indice_pairs.ndim() == 3, "error");
                TV_ASSERT_RT_ERR(indice_pairs.dim(0) == 2, "error");
                launcher_num_act_in(calc_subm_conv_indices<table_t, ConvLocIter64>, loc_iter, hash, indices.data_ptr<const int>(), 
                    indice_pairs.data_ptr<int>(), 
                    indice_num_per_loc.data_ptr<int>(), indices.dim(0), indice_pairs.dim(2), kv);
            }
        });
    return indices.dim(0);
  }
  else if (int(use_int32) == 1){
        ConvLocIter loc_iter(problem);
        tv::dispatch<int32_t, int64_t>(hashdata_k.dtype(), [&](auto I){
            using V = int32_t;
            using K = TV_DECLTYPE(I);
            TV_ASSERT_RT_ERR(input_dims.op<tv::arrayops::prod>() < std::numeric_limits<K>::max(), 
                "kernel volume must smaller than max value of K");
            using table_t =
                tv::hash::LinearHashTableSplit<K, V, tv::hash::Murmur3Hash<K>,
                                            tv::hash::default_empty_key_v<K>, false>;
            TV_ASSERT_RT_ERR(hashdata_k.dim(0) >= num_act_in_real, "hash size not enough");
            table_t hash = table_t(hashdata_k.data_ptr<K>(), hashdata_v.data_ptr<V>(), hashdata_k.dim(0));
            tv::hash::clear_map_split(hash, custream);
            lanucher_build_hash(build_subm_conv_hash_table<table_t, std::decay_t<decltype(loc_iter.layout_npq)>>, hash, indices.data_ptr<const int>(),
                loc_iter.layout_npq, num_act_in_real);
            if (!indice_pair_mask.empty()){
                TV_ASSERT_RT_ERR(indice_pairs.ndim() == 3, "error");
                TV_ASSERT_RT_ERR(indice_pairs.dim(0) == (is_train ? 2 : 1), "error");
                TV_ASSERT_INVALID_ARG(indice_pair_mask.ndim() == 2, "error");
                // indice_pair_mask: [mask_split_count, num_act_in]
                if (indice_pair_mask.dim(0) == 2){
                    auto mask_0 = indice_pair_mask[0].slice_first_axis(0, num_act_in_real);
                    auto mask_1 = indice_pair_mask[1].slice_first_axis(0, num_act_in_real);
                    tv::cuda::Launch lanucher_fill(num_act_in_real, custream);
                    lanucher_fill(cudakers::fill_kernel<uint32_t>, mask_0.data_ptr<uint32_t>(), (1 << (kv / 2)), indices.dim(0));
                    mask_1.zero_(ctx);
                    auto kernel = &calc_subm_conv_indices_split_mask<table_t, ConvLocIter>;
                    launcher_num_act_in(kernel, loc_iter, hash,  
                        indices.data_ptr<const int>(), indice_pairs.data_ptr<int>(), 
                        mask_0.data_ptr<uint32_t>(), mask_1.data_ptr<uint32_t>(), 
                        indices.dim(0), indice_pairs.dim(2), kv, is_train);
                }else{
                    // indice_pair_mask: [1, num_act_in]
                    tv::cuda::Launch lanucher_fill(num_act_in_real, custream);
                    lanucher_fill(cudakers::fill_kernel<uint32_t>, indice_pair_mask.data_ptr<uint32_t>(), (1 << (kv / 2)), indices.dim(0));
                    TV_ASSERT_RT_ERR(indice_pair_mask.dim(0) == 1, "error");
                    launcher_num_act_in(calc_subm_conv_indices_mask<table_t, ConvLocIter>, loc_iter, hash, 
                        indices.data_ptr<const int>(), indice_pairs.data_ptr<int>(), 
                        indice_pair_mask.data_ptr<uint32_t>(), indices.dim(0), indice_pairs.dim(2), kv, is_train);
                }
            }else{
                TV_ASSERT_RT_ERR(indice_pairs.ndim() == 3, "error");
                TV_ASSERT_RT_ERR(indice_pairs.dim(0) == 2, "error");
                launcher_num_act_in(calc_subm_conv_indices<table_t, ConvLocIter>, loc_iter, hash, indices.data_ptr<const int>(), 
                    indice_pairs.data_ptr<int>(), 
                    indice_num_per_loc.data_ptr<int>(), indices.dim(0), indice_pairs.dim(2), kv);
            }
        });
    return indices.dim(0);
  }
  else{
    TV_THROW_RT_ERR("unknown val int(use_int32), available: [0, 1]")
  }
}
} // namespace ops3d
} // namespace all
} // namespace sparse
} // namespace csrc
} // namespace spconv
} // namespace spconvlib