#include "hip/hip_runtime.h"
#include <spconvlib/spconv/csrc/sparse/all/ops3d/kernel/Point2VoxelKernel.h>
namespace spconvlib {
namespace spconv {
namespace csrc {
namespace sparse {
namespace all {
namespace ops3d {
namespace kernel {
using TensorView = spconvlib::cumm::common::TensorView;
using TensorViewHashKernel = spconvlib::cumm::common::TensorViewHashKernel;
using Layout = spconvlib::spconv::csrc::sparse::all::ops3d::layout_ns::TensorGeneric;
__global__ void voxel_empty_fill_mean(float * voxels, int * num_per_voxel, int num_voxels, int num_points_per_voxel, int num_voxel_features)   {
  
  int voxel_stride = num_points_per_voxel * num_voxel_features;
  for (int i : tv::KernelLoopX<int>(num_voxels)){
      int count = min(num_points_per_voxel, num_per_voxel[i]);
      num_per_voxel[i] = count;
      for (int j = 0; j < num_voxel_features; ++j){
          auto voxel_ptr = voxels + i * voxel_stride + j;
          float sum_val = 0;
          for (int k = 0; k < count; ++k){
              sum_val += voxel_ptr[0];
              voxel_ptr += num_voxel_features;
          }
          sum_val = count == 0 ? 0 : sum_val / count;
          for (int k = count; k < num_points_per_voxel; ++k){
              voxel_ptr[0] = sum_val;
              voxel_ptr += num_voxel_features;
          }
      }
  }
}
} // namespace kernel
} // namespace ops3d
} // namespace all
} // namespace sparse
} // namespace csrc
} // namespace spconv
} // namespace spconvlib