#include <spconvlib/spconv/csrc/sparse/all/ops3d/SparseConvIndicesKernel.h>
namespace spconvlib {
namespace spconv {
namespace csrc {
namespace sparse {
namespace all {
namespace ops3d {
using TensorView = spconvlib::cumm::common::TensorView;
using TensorViewKernel = spconvlib::cumm::common::TensorViewKernel;
using TensorViewHashKernel = spconvlib::cumm::common::TensorViewHashKernel;
using ConvLocIter = spconvlib::spconv::csrc::sparse::all::ops3d::spinds::ConvOutLocIter;
using ConvProblem = spconvlib::spconv::csrc::sparse::all::ops_cpu3d::spinds::ConvProblem;
using ConvLocIter64 = spconvlib::spconv::csrc::sparse::all::ops3d::spinds64::ConvOutLocIter;
int SparseConvIndicesKernel::generate_conv_inds_stage2(tv::Tensor indices, tv::Tensor hashdata_k, tv::Tensor hashdata_v, tv::Tensor indice_pairs, tv::Tensor indice_pairs_uniq, tv::Tensor indice_pairs_uniq_before_sort, tv::Tensor out_inds, tv::Tensor indice_num_per_loc, int num_out_act, int batch_size, tv::array<int, 3> output_dims, tv::array<int, 3> input_dims, tv::array<int, 3> ksize, tv::array<int, 3> stride, tv::array<int, 3> padding, tv::array<int, 3> dilation, bool transposed, std::uintptr_t stream_int, bool use_bound_algo)   {
  
  auto custream = reinterpret_cast<hipStream_t>(stream_int);
  // use_bound_algo = true;
  // TODO stream
  // TODO handle num input == 0
  int kv = ksize.op<tv::arrayops::prod>();
  TV_ASSERT_RT_ERR(kv == indice_pairs.dim(1), "error");
  TV_ASSERT_RT_ERR(hashdata_k.dtype() == indice_pairs_uniq.dtype(), "error");
  TV_ASSERT_RT_ERR(hashdata_v.dtype() == tv::int32, "error");
  auto ctx = tv::Context();
  ctx.set_cuda_stream(custream);
  // indice_pairs: [2, kv, num_act_in_bounded]
  // indice_pairs_uniq: [indice_pairs.size() / 2 + 1]
  // out_inds: [MaxSize, 4]
  // auto timer = tv::CudaContextTimer<>();
  int64_t uniq_size = indice_pairs.size() / 2 + 1;
  TV_ASSERT_RT_ERR(indice_pairs_uniq.dim(0) >= num_out_act, "error");
  TV_ASSERT_RT_ERR(out_inds.dim(0) >= num_out_act && out_inds.dim(1) == 4, "error");
  tv::cuda::Launch launcher_num_act_in(indices.dim(0), custream);
  launcher_num_act_in.blocks.y = kv;
  ConvProblem problem(batch_size, 1, 1, input_dims, output_dims, ksize, padding, stride, dilation);
  bool use_int32 = problem.check_npq_not_overflow();
  // TODO handle invalid num_out_act
  indice_pairs_uniq = indice_pairs_uniq.slice_first_axis(0, num_out_act);
  tv::cuda::Launch lanucher_build_hash(num_out_act, custream);
  tv::dispatch<int32_t, int64_t>(hashdata_k.dtype(), [&](auto I){
    using V = int32_t;
    using K = TV_DECLTYPE(I);
    using table_t =
        tv::hash::LinearHashTableSplit<K, V, tv::hash::Murmur3Hash<K>,
                                    tv::hash::default_empty_key_v<K>, false>;
    TV_ASSERT_RT_ERR(hashdata_k.dim(0) >= num_out_act, "hash size not enough");
    table_t hash = table_t(hashdata_k.data_ptr<K>(), hashdata_v.data_ptr<V>(), hashdata_k.dim(0));
    tv::hash::clear_map_split(hash, custream);
    // hash.clear(custream);
    if (int(use_int32) == 0){
      ConvLocIter64 loc_iter(problem);
      lanucher_build_hash(build_conv_hash_table<table_t, std::decay_t<decltype(loc_iter.layout_npq)>>, hash, 
          out_inds.data_ptr<int>(), indice_pairs_uniq.data_ptr<const K>(), 
          loc_iter.layout_npq, num_out_act);
    }
    else if (int(use_int32) == 1){
      ConvLocIter loc_iter(problem);
      lanucher_build_hash(build_conv_hash_table<table_t, std::decay_t<decltype(loc_iter.layout_npq)>>, hash, 
          out_inds.data_ptr<int>(), indice_pairs_uniq.data_ptr<const K>(), 
          loc_iter.layout_npq, num_out_act);
    }
    else{
      TV_THROW_RT_ERR("unknown val int(use_int32), available: [0, 1]")
    }
    if (!use_bound_algo){
        launcher_num_act_in(calc_conv_indices_stage2<table_t>, hash, 
            indice_pairs_uniq_before_sort.data_ptr<const K>(),
            indice_pairs[1].data_ptr<int>(), 
            indices.dim(0), 
            indice_pairs.dim(2));
    }else{
        indice_num_per_loc.zero_(ctx);
        // copy previous pair in to indice_pairs_uniq
        // we need to ensure size of indice_pairs_uniq larger than pair in
        TV_ASSERT_RT_ERR(true, "error");
        tv::Tensor indice_pairs_in_temp = tv::from_blob(indice_pairs_uniq.raw_data(), {indice_pairs.dim(1), indice_pairs.dim(2)}, 
            indice_pairs.dtype(), indice_pairs.device());
        indice_pairs_in_temp.copy_(indice_pairs[0].view(-1), ctx);
        launcher_num_act_in(calc_conv_indices_stage2_bounded<table_t>, hash, 
            indice_pairs_uniq_before_sort.data_ptr<const K>(),
            indice_pairs_in_temp.data_ptr<const int>(),
            indice_pairs[0].data_ptr<int>(), 
            indice_pairs[1].data_ptr<int>(), 
            indice_num_per_loc.data_ptr<int>(),
            indices.dim(0), 
            indice_pairs.dim(2));
    }
  });
  return num_out_act;
}
} // namespace ops3d
} // namespace all
} // namespace sparse
} // namespace csrc
} // namespace spconv
} // namespace spconvlib