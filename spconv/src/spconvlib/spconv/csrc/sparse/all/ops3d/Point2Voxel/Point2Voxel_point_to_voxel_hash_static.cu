#include "hip/hip_runtime.h"
#include <spconvlib/spconv/csrc/sparse/all/ops3d/Point2Voxel.h>
#include <spconvlib/spconv/csrc/sparse/all/ops3d/kernel/Point2VoxelKernel.h>
namespace spconvlib {
namespace spconv {
namespace csrc {
namespace sparse {
namespace all {
namespace ops3d {
using TensorView = spconvlib::cumm::common::TensorView;
using Point2VoxelCommon = spconvlib::spconv::csrc::sparse::all::ops3d::p2v_c::Point2VoxelCommon;
using Layout = spconvlib::spconv::csrc::sparse::all::ops3d::layout_ns::TensorGeneric;
std::tuple<tv::Tensor, tv::Tensor, tv::Tensor> Point2Voxel::point_to_voxel_hash_static(tv::Tensor points, tv::Tensor voxels, tv::Tensor indices, tv::Tensor num_per_voxel, tv::Tensor hashdata, tv::Tensor point_indice_data, tv::Tensor points_voxel_id, std::array<float, 3> vsize, std::array<int, 3> grid_size, std::array<int64_t, 3> grid_stride, std::array<float, 6> coors_range, bool clear_voxels, bool empty_mean, std::uintptr_t stream_int)   {
  
  auto vsize_tv = Point2VoxelCommon::array2tvarray(vsize);
  auto grid_size_tv = Point2VoxelCommon::array2tvarray(grid_size);
  auto grid_stride_tv = Point2VoxelCommon::array2tvarray(grid_stride);
  auto coors_range_tv = Point2VoxelCommon::array2tvarray(coors_range);
  auto custream = reinterpret_cast<hipStream_t>(stream_int);
  auto ctx = tv::Context();
  ctx.set_cuda_stream(custream);
  TV_ASSERT_INVALID_ARG(points.ndim() == 2 && points.dim(1) >= 3, "error");
  using V = int64_t;
  using KeyType = int64_t;
  constexpr KeyType kEmptyKey = std::numeric_limits<KeyType>::max();
  if (clear_voxels){
      voxels.zero_(ctx);
  }
  using table_t =
      tv::hash::LinearHashTable<KeyType, V, tv::hash::Murmur3Hash<KeyType>,
                                  kEmptyKey, false>;
  using pair_t = typename table_t::value_type;
  // int64_t expected_hash_data_num = int64_t(tv::hash::align_to_power2(points.dim(0) * 2));
  int64_t expected_hash_data_num = points.dim(0) * 2;
  TV_ASSERT_RT_ERR(hashdata.dim(0) >= expected_hash_data_num, "hash table too small")
  TV_ASSERT_RT_ERR(point_indice_data.dim(0) >= points.dim(0), "point_indice_data too small")
  num_per_voxel.zero_(ctx);
  table_t hash = table_t(hashdata.data_ptr<pair_t>(), expected_hash_data_num);
  tv::hash::clear_map(hash, custream);
  auto launcher = tv::cuda::Launch(points.dim(0), custream);
  launcher(kernel::build_hash_table<table_t>, hash, points.data_ptr<const float>(),
          point_indice_data.data_ptr<int64_t>(),
          points.dim(1), vsize_tv, coors_range_tv, grid_size_tv, grid_stride_tv, points.dim(0));
  auto table_launcher = tv::cuda::Launch(hash.size(), custream);
  tv::Tensor count = tv::zeros({1}, tv::int32, 0);
  Layout layout = Layout::from_shape(grid_size_tv);
  table_launcher(kernel::assign_table<table_t>, hash, indices.data_ptr<int>(),
                  count.data_ptr<int>(),
                  layout, voxels.dim(0));
  auto count_cpu = count.cpu();
  int count_val = count_cpu.item<int32_t>();
  count_val = count_val > voxels.dim(0) ? voxels.dim(0) : count_val;
  launcher(kernel::generate_voxel<table_t>, hash, points.data_ptr<const float>(),
          point_indice_data.data_ptr<const int64_t>(), voxels.data_ptr<float>(),
          num_per_voxel.data_ptr<int>(), points_voxel_id.data_ptr<int64_t>(), points.dim(1), voxels.dim(1), 
          voxels.dim(0), vsize_tv, coors_range_tv,
          grid_size_tv, grid_stride_tv, points.dim(0));
  auto voxel_launcher = tv::cuda::Launch(count_val, custream);
  if (empty_mean){
      launcher(kernel::voxel_empty_fill_mean, voxels.data_ptr<float>(),
              num_per_voxel.data_ptr<int>(), count_val, 
              voxels.dim(1), voxels.dim(2));
  }else{
      launcher(kernel::limit_num_per_voxel_value, num_per_voxel.data_ptr<int>(), count_val, 
              voxels.dim(1));
  }
  return std::make_tuple(voxels.slice_first_axis(0, count_val), 
      indices.slice_first_axis(0, count_val), 
      num_per_voxel.slice_first_axis(0, count_val));
}
} // namespace ops3d
} // namespace all
} // namespace sparse
} // namespace csrc
} // namespace spconv
} // namespace spconvlib