#include <spconvlib/spconv/csrc/sparse/all/ops4d/SparseConvIndicesKernel.h>
namespace spconvlib {
namespace spconv {
namespace csrc {
namespace sparse {
namespace all {
namespace ops4d {
using TensorView = spconvlib::cumm::common::TensorView;
using TensorViewKernel = spconvlib::cumm::common::TensorViewKernel;
using TensorViewHashKernel = spconvlib::cumm::common::TensorViewHashKernel;
using ConvLocIter = spconvlib::spconv::csrc::sparse::all::ops4d::spinds::ConvOutLocIter;
using ConvProblem = spconvlib::spconv::csrc::sparse::all::ops_cpu4d::spinds::ConvProblem;
using ConvLocIter64 = spconvlib::spconv::csrc::sparse::all::ops4d::spinds64::ConvOutLocIter;
int SparseConvIndicesKernel::generate_conv_inds_stage2_mask(tv::Tensor indices, tv::Tensor hashdata_k, tv::Tensor hashdata_v, tv::Tensor indice_pairs_fwd, tv::Tensor indice_pairs_bwd, tv::Tensor indice_pairs_uniq, tv::Tensor indice_pairs_uniq_before_sort, tv::Tensor out_inds, tv::Tensor mask_fwd, tv::Tensor mask_bwd, int num_out_act, int batch_size, tv::array<int, 4> output_dims, tv::array<int, 4> input_dims, tv::array<int, 4> ksize, tv::array<int, 4> stride, tv::array<int, 4> padding, tv::array<int, 4> dilation, bool transposed, std::uintptr_t stream_int)   {
  
  auto custream = reinterpret_cast<hipStream_t>(stream_int);
  // TODO stream
  // TODO handle num input == 0
  int kv = ksize.op<tv::arrayops::prod>();
  // indice_pairs_bwd: [kv, num_act_in]  or empty
  // indice_pairs_fwd: [kv, num_act_out]
  auto ctx = tv::Context();
  ctx.set_cuda_stream(custream);
  int num_act_in = indices.dim(0);
  int num_act_out = num_out_act;
  TV_ASSERT_RT_ERR(hashdata_k.dtype() == indice_pairs_uniq.dtype(), "error");
  TV_ASSERT_RT_ERR(hashdata_v.dtype() == tv::int32, "error");
  // out_inds: [num_out_act, 5]
  // auto timer = tv::CudaContextTimer<>();
  if (!indice_pairs_bwd.empty()){
      tv::check_shape(indice_pairs_bwd, {kv, num_act_in});
  }
  tv::check_shape(indice_pairs_fwd, {kv, num_act_out});
  tv::check_shape(out_inds, {num_out_act, 5});
  tv::cuda::Launch launcher_num_act_in(num_act_in, custream);
  launcher_num_act_in.blocks.y = kv;
  tv::cuda::Launch launcher_num_act_in_no_y(num_act_in, custream);
  ConvProblem problem(batch_size, 1, 1, input_dims, output_dims, ksize, padding, stride, dilation);
  tv::cuda::Launch lanucher_build_hash(num_out_act, custream);
  bool use_int32 = problem.check_npq_not_overflow();
  // TODO handle invalid num_out_act
  indice_pairs_uniq = indice_pairs_uniq.slice_first_axis(0, num_out_act);
  tv::dispatch<int32_t, int64_t>(hashdata_k.dtype(), [&](auto I){
    using V = int32_t;
    using K = TV_DECLTYPE(I);
    using table_t =
        tv::hash::LinearHashTableSplit<K, V, tv::hash::Murmur3Hash<K>,
                                    tv::hash::default_empty_key_v<K>, false>;
    TV_ASSERT_RT_ERR(hashdata_k.dim(0) >= num_out_act, "hash size not enough");
    table_t hash = table_t(hashdata_k.data_ptr<K>(), hashdata_v.data_ptr<V>(), hashdata_k.dim(0));
    tv::hash::clear_map_split(hash, custream);
    if (int(use_int32) == 0){
      ConvLocIter64 loc_iter(problem);
      lanucher_build_hash(build_conv_hash_table<table_t, std::decay_t<decltype(loc_iter.layout_npq)>>, hash, 
          out_inds.data_ptr<int>(), indice_pairs_uniq.data_ptr<const K>(), 
          loc_iter.layout_npq, num_out_act);
    }
    else if (int(use_int32) == 1){
      ConvLocIter loc_iter(problem);
      lanucher_build_hash(build_conv_hash_table<table_t, std::decay_t<decltype(loc_iter.layout_npq)>>, hash, 
          out_inds.data_ptr<int>(), indice_pairs_uniq.data_ptr<const K>(), 
          loc_iter.layout_npq, num_out_act);
    }
    else{
      TV_THROW_RT_ERR("unknown val int(use_int32), available: [0, 1]")
    }
    if (!mask_bwd.empty()){
        launcher_num_act_in(calc_conv_indices_stage2_mask<table_t, false>, hash, 
            indice_pairs_fwd.data_ptr<int>(), indice_pairs_bwd.data_ptr<int>(), 
            indice_pairs_uniq_before_sort.data_ptr<K>(),
            mask_fwd.data_ptr<uint32_t>(), mask_bwd.data_ptr<uint32_t>(),
            num_act_in, indice_pairs_fwd.dim(1));
        launcher_num_act_in_no_y(calc_conv_indices_stage2_mask_output, 
            indice_pairs_bwd.data_ptr<int>(), 
            mask_bwd.data_ptr<uint32_t>(),
            num_act_in, kv);
        if (mask_fwd.dim(0) == 2){
            mask_fwd[1].copy_(mask_fwd[0], ctx);
        }
        if (mask_bwd.dim(0) == 2){
            mask_bwd[1].copy_(mask_bwd[0], ctx);
        }
    }else{
        launcher_num_act_in(calc_conv_indices_stage2_inference_mask<table_t, false>, hash, 
            indice_pairs_fwd.data_ptr<int>(), indice_pairs_bwd.data_ptr<int>(), 
            indice_pairs_uniq_before_sort.data_ptr<K>(),
            mask_fwd.data_ptr<uint32_t>(),
            num_act_in, indice_pairs_fwd.dim(1));
        if (mask_fwd.dim(0) == 2){
            mask_fwd[1].copy_(mask_fwd[0], ctx);
        }
    }
  });
  return num_out_act;
}
} // namespace ops4d
} // namespace all
} // namespace sparse
} // namespace csrc
} // namespace spconv
} // namespace spconvlib