#include "hip/hip_runtime.h"
#include <spconvlib/spconv/csrc/sparse/all/ops4d/SparseConvIndicesKernel.h>
namespace spconvlib {
namespace spconv {
namespace csrc {
namespace sparse {
namespace all {
namespace ops4d {
using TensorView = spconvlib::cumm::common::TensorView;
using TensorViewKernel = spconvlib::cumm::common::TensorViewKernel;
using TensorViewHashKernel = spconvlib::cumm::common::TensorViewHashKernel;
using ConvLocIter = spconvlib::spconv::csrc::sparse::all::ops4d::spinds::ConvOutLocIter;
using ConvProblem = spconvlib::spconv::csrc::sparse::all::ops_cpu4d::spinds::ConvProblem;
using ConvLocIter64 = spconvlib::spconv::csrc::sparse::all::ops4d::spinds64::ConvOutLocIter;
void SparseConvIndicesKernel::generate_conv_inds_mask_stage1_direct_table(tv::Tensor indices, tv::Tensor hashdata_k, tv::Tensor hashdata_v, tv::Tensor indice_pairs_bwd, tv::Tensor indice_pairs_uniq, tv::Tensor indice_num_per_loc, int batch_size, tv::array<int, 4> output_dims, tv::array<int, 4> input_dims, tv::array<int, 4> ksize, tv::array<int, 4> stride, tv::array<int, 4> padding, tv::array<int, 4> dilation, bool transposed, std::uintptr_t stream_int)   {
  
  // TODO stream
  // TODO handle num input == 0
  int kv = ksize.op<tv::arrayops::prod>();
  int num_act_in = indices.dim(0);
  // indice_pairs_bwd: [kv, num_act_in] or empty
  // indice_pairs_uniq: [kv * num_act_in + 1]
  if (!indice_pairs_bwd.empty()){
      tv::check_shape(indice_pairs_bwd, {kv, num_act_in});
  }
  tv::check_shape(indice_num_per_loc, {kv});
  int64_t uniq_size = kv * num_act_in + 1;
  TV_ASSERT_RT_ERR(indice_pairs_uniq.dim(0) == uniq_size, "error");
  tv::cuda::Launch launcher_num_act_in(indices.dim(0), reinterpret_cast<hipStream_t>(stream_int));
  // tv::cuda::Launch launcher_num_act_in_2(indices.dim(0));
  launcher_num_act_in.blocks.y = kv;
  ConvProblem problem(batch_size, 1, 1, input_dims, output_dims, ksize, padding, stride, dilation);
  tv::cuda::Launch launcher_clean_uniq(uniq_size, reinterpret_cast<hipStream_t>(stream_int));
  bool use_int32 = problem.check_npq_not_overflow();
  tv::dispatch<int32_t, int64_t>(indice_pairs_uniq.dtype(), [&](auto I){
    using V = int32_t;
    using K = TV_DECLTYPE(I);
    using table_t =
        tv::hash::LinearHashTableSplit<K, V, tv::hash::Murmur3Hash<K>,
                                    tv::hash::default_empty_key_v<K>, false>;
    table_t table = table_t(hashdata_k.data_ptr<K>(), hashdata_v.data_ptr<V>(), hashdata_k.dim(0));
    tv::hash::clear_map_split(table, reinterpret_cast<hipStream_t>(stream_int));
    using T = TV_DECLTYPE(I);
    TV_ASSERT_RT_ERR(input_dims.op<tv::arrayops::prod>() < std::numeric_limits<T>::max(), 
        "kernel volume must smaller than max value of T");
    launcher_clean_uniq(clean_indices_uniq<T>, indice_pairs_uniq.data_ptr<T>(), uniq_size);
    if (int(use_int32) == 0){
      ConvLocIter64 loc_iter(problem);
      launcher_num_act_in(calc_conv_indices_stage1_mask_direct_table<T, table_t, ConvLocIter64>, table, 
          loc_iter, indices.data_ptr<const int>(), 
          indice_pairs_bwd.data_ptr<int32_t>(), 
          indice_pairs_uniq.data_ptr<T>(), indice_num_per_loc.data_ptr<int>(), 
          indices.dim(0),
          kv, transposed);
    }
    else if (int(use_int32) == 1){
      ConvLocIter loc_iter(problem);
      launcher_num_act_in(calc_conv_indices_stage1_mask_direct_table<T, table_t, ConvLocIter>, table, 
          loc_iter, indices.data_ptr<const int>(), 
          indice_pairs_bwd.data_ptr<int32_t>(), 
          indice_pairs_uniq.data_ptr<T>(), indice_num_per_loc.data_ptr<int>(), 
          indices.dim(0),
          kv, transposed);
    }
    else{
      TV_THROW_RT_ERR("unknown val int(use_int32), available: [0, 1]")
    }
  });
}
} // namespace ops4d
} // namespace all
} // namespace sparse
} // namespace csrc
} // namespace spconv
} // namespace spconvlib