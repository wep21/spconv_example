#include <spconvlib/spconv/csrc/sparse/all/ops4d/SparseConvIndicesKernel.h>
namespace spconvlib {
namespace spconv {
namespace csrc {
namespace sparse {
namespace all {
namespace ops4d {
using TensorView = spconvlib::cumm::common::TensorView;
using TensorViewKernel = spconvlib::cumm::common::TensorViewKernel;
using TensorViewHashKernel = spconvlib::cumm::common::TensorViewHashKernel;
using ConvLocIter = spconvlib::spconv::csrc::sparse::all::ops4d::spinds::ConvOutLocIter;
using ConvProblem = spconvlib::spconv::csrc::sparse::all::ops_cpu4d::spinds::ConvProblem;
using ConvLocIter64 = spconvlib::spconv::csrc::sparse::all::ops4d::spinds64::ConvOutLocIter;
void SparseConvIndicesKernel::assign_output_direct_hash(tv::Tensor out_indices_offset, tv::Tensor out_inds, int batch_size, tv::array<int, 4> output_dims, tv::array<int, 4> input_dims, tv::array<int, 4> ksize, tv::array<int, 4> stride, tv::array<int, 4> padding, tv::array<int, 4> dilation, std::uintptr_t stream_int)   {
  
  auto custream = reinterpret_cast<hipStream_t>(stream_int);
  tv::cuda::Launch lanucher_build_hash(out_inds.dim(0), custream);
  TV_ASSERT_RT_ERR(out_indices_offset.dim(0) >= out_inds.dim(0), "error");
  ConvProblem problem(batch_size, 1, 1, input_dims, output_dims, ksize, padding, stride, dilation);
  bool use_int32 = problem.check_npq_not_overflow();
  auto tvctx = tv::Context();
  tvctx.set_cuda_stream(reinterpret_cast<hipStream_t>(stream_int));
  if (int(use_int32) == 0){
    ConvLocIter64 loc_iter(problem);
    tv::dispatch<int32_t, int64_t>(out_indices_offset.dtype(), [&](auto I){
        using K = TV_DECLTYPE(I);
        lanucher_build_hash(assign_out_indices<K, std::decay_t<decltype(loc_iter.layout_npq)>>, out_inds.data_ptr<int>(),
            out_indices_offset.data_ptr<const K>(),
            loc_iter.layout_npq, out_inds.dim(0));
    });
  }
  else if (int(use_int32) == 1){
    ConvLocIter loc_iter(problem);
    tv::dispatch<int32_t, int64_t>(out_indices_offset.dtype(), [&](auto I){
        using K = TV_DECLTYPE(I);
        lanucher_build_hash(assign_out_indices<K, std::decay_t<decltype(loc_iter.layout_npq)>>, out_inds.data_ptr<int>(),
            out_indices_offset.data_ptr<const K>(),
            loc_iter.layout_npq, out_inds.dim(0));
    });
  }
  else{
    TV_THROW_RT_ERR("unknown val int(use_int32), available: [0, 1]")
  }
}
} // namespace ops4d
} // namespace all
} // namespace sparse
} // namespace csrc
} // namespace spconv
} // namespace spconvlib