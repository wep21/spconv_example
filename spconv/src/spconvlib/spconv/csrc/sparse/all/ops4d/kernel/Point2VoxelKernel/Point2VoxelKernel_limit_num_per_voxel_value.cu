#include "hip/hip_runtime.h"
#include <spconvlib/spconv/csrc/sparse/all/ops4d/kernel/Point2VoxelKernel.h>
namespace spconvlib {
namespace spconv {
namespace csrc {
namespace sparse {
namespace all {
namespace ops4d {
namespace kernel {
using TensorView = spconvlib::cumm::common::TensorView;
using TensorViewHashKernel = spconvlib::cumm::common::TensorViewHashKernel;
using Layout = spconvlib::spconv::csrc::sparse::all::ops4d::layout_ns::TensorGeneric;
__global__ void limit_num_per_voxel_value(int * num_per_voxel, int num_voxels, int num_points_per_voxel)   {
  
  for (int i : tv::KernelLoopX<int>(num_voxels)){
      int count = min(num_points_per_voxel, num_per_voxel[i]);
      num_per_voxel[i] = count;
  }
}
} // namespace kernel
} // namespace ops4d
} // namespace all
} // namespace sparse
} // namespace csrc
} // namespace spconv
} // namespace spconvlib