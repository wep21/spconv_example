#include <spconvlib/spconv/csrc/sparse/all/ops4d/SparseConvIndicesKernel.h>
#include <spconvlib/cumm/common/ThrustLib.h>
namespace spconvlib {
namespace spconv {
namespace csrc {
namespace sparse {
namespace all {
namespace ops4d {
using TensorView = spconvlib::cumm::common::TensorView;
using TensorViewKernel = spconvlib::cumm::common::TensorViewKernel;
using TensorViewHashKernel = spconvlib::cumm::common::TensorViewHashKernel;
using ConvLocIter = spconvlib::spconv::csrc::sparse::all::ops4d::spinds::ConvOutLocIter;
using ConvProblem = spconvlib::spconv::csrc::sparse::all::ops_cpu4d::spinds::ConvProblem;
using ConvLocIter64 = spconvlib::spconv::csrc::sparse::all::ops4d::spinds64::ConvOutLocIter;
using ThrustLib = spconvlib::cumm::common::ThrustLib;
int SparseConvIndicesKernel::generate_conv_inds_stage1_5(tv::Tensor indice_pairs_uniq, int64_t uniq_size, std::uintptr_t stream_int)   {
  
  int num_out_act = 0;
  tv::dispatch<int32_t, int64_t>(indice_pairs_uniq.dtype(), [&](auto I){
      using T = TV_DECLTYPE(I);
      thrust::device_ptr<T> ptr_tr(indice_pairs_uniq.data_ptr<T>());
      auto thrust_ctx = thrust::cuda::par.on(reinterpret_cast<hipStream_t>(stream_int));
      thrust::sort(thrust_ctx, ptr_tr, ptr_tr + uniq_size);
      auto new_end = thrust::unique(thrust_ctx, ptr_tr, ptr_tr + uniq_size);
      num_out_act = new_end - ptr_tr - 1;
  });
  return num_out_act;
}
} // namespace ops4d
} // namespace all
} // namespace sparse
} // namespace csrc
} // namespace spconv
} // namespace spconvlib