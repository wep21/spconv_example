#include "hip/hip_runtime.h"
#include <spconvlib/spconv/csrc/sparse/all/ops4d/SparseConvIndicesKernel.h>
namespace spconvlib {
namespace spconv {
namespace csrc {
namespace sparse {
namespace all {
namespace ops4d {
using TensorView = spconvlib::cumm::common::TensorView;
using TensorViewKernel = spconvlib::cumm::common::TensorViewKernel;
using TensorViewHashKernel = spconvlib::cumm::common::TensorViewHashKernel;
using ConvLocIter = spconvlib::spconv::csrc::sparse::all::ops4d::spinds::ConvOutLocIter;
using ConvProblem = spconvlib::spconv::csrc::sparse::all::ops_cpu4d::spinds::ConvProblem;
using ConvLocIter64 = spconvlib::spconv::csrc::sparse::all::ops4d::spinds64::ConvOutLocIter;
__global__ void calc_conv_indices_stage2_mask_output(int* indice_pairs_bwd, uint32_t* mask_bwd, int num_indices_in, int kv, int mask_int_count)   {
  
  for (int input_index : tv::KernelLoopX<int>(num_indices_in)) {
      for (int mask_offset = 0; mask_offset < mask_int_count; ++mask_offset){
          uint32_t mask = 0;
          for (int filter_offset = mask_offset * 32; filter_offset < mask_offset * 32 +  32 && filter_offset < kv; ++filter_offset){
              auto val = indice_pairs_bwd[filter_offset * num_indices_in + input_index];
              mask |= (val != -1) << (filter_offset % 32);
          }
          mask_bwd[input_index * mask_int_count + mask_offset] = mask;
      }
  }
}
} // namespace ops4d
} // namespace all
} // namespace sparse
} // namespace csrc
} // namespace spconv
} // namespace spconvlib