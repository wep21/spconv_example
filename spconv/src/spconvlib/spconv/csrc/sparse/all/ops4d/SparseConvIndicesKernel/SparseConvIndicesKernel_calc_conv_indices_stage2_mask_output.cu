#include "hip/hip_runtime.h"
#include <spconvlib/spconv/csrc/sparse/all/ops4d/SparseConvIndicesKernel.h>
namespace spconvlib {
namespace spconv {
namespace csrc {
namespace sparse {
namespace all {
namespace ops4d {
using TensorView = spconvlib::cumm::common::TensorView;
using TensorViewKernel = spconvlib::cumm::common::TensorViewKernel;
using TensorViewHashKernel = spconvlib::cumm::common::TensorViewHashKernel;
using ConvLocIter = spconvlib::spconv::csrc::sparse::all::ops4d::spinds::ConvOutLocIter;
using ConvProblem = spconvlib::spconv::csrc::sparse::all::ops_cpu4d::spinds::ConvProblem;
using ConvLocIter64 = spconvlib::spconv::csrc::sparse::all::ops4d::spinds64::ConvOutLocIter;
__global__ void calc_conv_indices_stage2_mask_output(int* indice_pairs_bwd, uint32_t* mask_bwd, int num_indices_in, int kv)   {
  
  for (int input_index : tv::KernelLoopX<int>(num_indices_in)) {
      uint32_t mask = 0;
      for (int filter_offset = 0; filter_offset < kv; ++filter_offset){
          auto val = indice_pairs_bwd[filter_offset * num_indices_in + input_index];
          mask |= (val != -1) << filter_offset;
      }
      mask_bwd[input_index] = mask;
  }
}
} // namespace ops4d
} // namespace all
} // namespace sparse
} // namespace csrc
} // namespace spconv
} // namespace spconvlib